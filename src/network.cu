#include "hip/hip_runtime.h"
﻿#include <network.h>
#include <kernelDefs.h>
#include <utilFunc.h>
#include <neuroFunc.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <ostream>
#include <vector>
#include <ctime>
#include <cstdio>
#include <assert.h>
//parameter definitions
//    _hostParams.array[0] = number of neurons
//    _hostparams.array[1] = number of weights per individual
//    _hostParams.array[2] = size of individual
//    _hostParams.array[3] = number of input neurons per individual
//    _hostParams.array[4] = number of hidden neurons per individual
//    _hostParams.array[5] = number of memory neurons per individual
//    _hostParams.array[6] = number of memoryGate In nuerons per individual
//    _hostParams.array[7] = number of memoryGate Out neurons per individual
//    _hostParams.array[8] = number of memoryGate Forget neurons per individual
//    _hostParams.array[9] = number of output neurons per individual
//    _hostParams.array[10] = number of individuals in stream
//    _hostParams.array[11] = weights offset
//    _hostParams.array[12] = input offset
//    _hostParams.array[13] = hidden neurons offset
//    _hostParams.array[14] = short term memory neurons offset
//    _hostParams.array[15] = memoryIn neurons offset
//    _hostParams.array[16] = memoryOut neurons offset
//    _hostParams.array[17] = memoryForget neurons offset
//    _hostParams.array[18] = output neurons offset
//    _hostParams.array[19] = fitness offset
//    _hostParams.array[20] = community magnitude offset
//    _hostParams.array[21] = when offset
//    _hostParams.array[22] = howCertain offset
//    _hostParams.array[23] = number of sites
//    _hostParams.array[24] = sample rate
//    _hostParams.array[26] = number of orders
//    _hostParams.array[27] = number of long term memory neurons per individual
//_hostParams.array[28] = long term memory neurons offset


NetworkGenetic::NetworkGenetic(){
    _hostParams.array = new int[30];
    _hostParams.size = 30;
}
NetworkGenetic::~NetworkGenetic(){
    delete[] _hostParams.array;
}

void NetworkGenetic::generateWeights(){
    int blockSize = 512; // number of blocks in the grid
    int gridSize=(_hostParams.array[10]*_hostParams.array[1])/blockSize; //number of weights in stream/blocksize
    size_t global_offset=0;
    size_t device_offset=0;
    std::cerr<<"generating weights.. "<<std::endl;
    std::cerr<<"grid size: "<<gridSize<<std::endl;
    for(int n=0; n<_numOfStreams; n++){//fill the host first.
        if(n%2==0 && n !=0)
            device_offset =0;
        size_t seed;
        FILE *fp;
        fp = std::fopen("/dev/urandom", "r");
        size_t chk;
        chk =std::fread(&seed, 4, 1, fp);
        if(chk <1){std::cerr<<"couldn't read /dev/urandom"<<std::endl; exit(1);}
        std::fclose(fp);
        std::cerr<<"stream number #"<<n+1<<std::endl;
        genWeightsKern<<< gridSize, blockSize, 0, _stream[n]>>>(device_genetics, seed, _deviceParams, device_offset);
        CUDA_SAFE_CALL(hipPeekAtLastError());
        CUDA_SAFE_CALL(hipMemcpyAsync(&host_genetics.array[global_offset], &device_genetics.array[device_offset], _streambytes, hipMemcpyDeviceToHost, _stream[n]));
        global_offset += _streamSize;
        device_offset += _streamSize;
    }
}

void NetworkGenetic::allocateHostAndGPUObjects( size_t deviceRam, size_t hostRam){
    CUDA_SAFE_CALL(hipFuncSetCacheConfig(reinterpret_cast<const void*>(NetKern), hipFuncCachePreferL1));
    CUDA_SAFE_CALL(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeFourByte));
    long int hostGenSize = hostRam;
    long int deviceGenSize = deviceRam;
    std::cerr<<"total free device ram : "<<deviceRam<<std::endl;
    std::cerr<<"total free host ram : "<<hostRam<<std::endl;

    //this block below makes sure that the number of objects in stream is exactly half of the total amount of allocatable space on the GPU
    _streamSize = deviceGenSize/(sizeof(double)*2);

    while(_streamSize%_hostParams.array[2] || (_streamSize/_hostParams.array[2])&(_streamSize/_hostParams.array[2]-1)) // get the largest number divisible by the individual size, the threads in a block, and the size of a double
        _streamSize= _streamSize -1;

    _streambytes = _streamSize * sizeof(double);
    deviceGenSize = _streambytes * 2;
    assert(deviceGenSize == _streambytes * 2);
    device_genetics.size = _streamSize * 2;

    //this block below makes sure that the allocated host ram for genetics is a number evently divisible by the stream size in bytes.
    host_genetics.size = hostGenSize/sizeof(double);
    _numOfStreams =  host_genetics.size/_streamSize; // number of streams = hostGenSize/streamBytes, device does not store extra weights for simplicity.
    host_genetics.size = _numOfStreams * _streamSize;
    hostGenSize = host_genetics.size * sizeof(double);
    assert(host_genetics.size == _numOfStreams * _streamSize);

    CUDA_SAFE_CALL(hipHostAlloc((void**)&host_genetics.array, hostGenSize, hipHostMallocWriteCombined));
    CUDA_SAFE_CALL(hipMalloc((void**) &device_genetics.array, deviceGenSize));
    std::fill(host_genetics.array, host_genetics.array+host_genetics.size, 0);
    CUDA_SAFE_CALL(hipMemset(device_genetics.array, 0, deviceGenSize));
    _stream.resize(_numOfStreams);

    for(int i=0; i<_numOfStreams; i++){
        CUDA_SAFE_CALL(hipStreamCreate(&_stream.at(i)));
        CUDA_SAFE_CALL( hipStreamQuery(_stream.at(i)));
    }
    this->confDeviceParams();
}

void NetworkGenetic:: confTestParams(const int &numOfSites, const int &sampleRate){
    this->setParams(23, numOfSites);
    this->setParams(24, sampleRate);
    this->setParams(2, _hostParams.array[0] + _hostParams.array[1] + 1 + 3*15); //size of an individual
}

void NetworkGenetic::confNetParams(const int &numInNeurons, const int &numHiddenNeurons,
                                   const int &numMemoryNeurons, const int &numMemoryIn, const int &numMemoryOut,
                                   const int &numMemoryForget, const int &numOutNeurons){

    this->setParams(0, numInNeurons + numHiddenNeurons + numMemoryNeurons + numMemoryIn + numMemoryOut + numMemoryForget + numOutNeurons);
    this->setParams(3, numInNeurons);
    this->setParams(4, numHiddenNeurons);
    this->setParams(5, numMemoryNeurons);
    this->setParams(6, numMemoryIn);
    this->setParams(7, numMemoryOut);
    this->setParams(8, numMemoryForget);
    this->setParams(9, numOutNeurons);

}

void NetworkGenetic::confOrder(const int &numOrders, const int &numWeights){

    this->setParams(1, numWeights);
    this->setParams(26, numOrders);
    std::cerr<<numOrders<<std::endl;
}


bool NetworkGenetic::loadFromFile(std::ifstream &stream){
    stream.clear();
    stream.seekg(0, stream.beg);
    assert(stream.good());
    std::string item;
    std::cerr<<"preparing to load from file..."<<std::endl;
    size_t entry=0;
    int itr =0;
    while(std::getline(stream, item)){ // each value in the array
        entry++;
    }
    std::cerr<<"number of data points: "<<entry<<std::endl;

    this->allocateHostAndGPUObjects(GetDeviceRamInBytes()*0.85, entry*sizeof(double));
    stream.clear();
    stream.seekg(0, stream.beg);
    assert(stream.good());
    std::cerr.precision(2);
    while(std::getline(stream, item)){ // each value in the array
        host_genetics.array[itr] = std::stod(item);
        if(itr%(host_genetics.size/100) == 0){
            std::cerr<<(float)itr/(float)host_genetics.size<<std::endl;
        }
        itr++;
    }
    std::cerr<<"finished loading from file"<<std::endl;
    return true;
}

void NetworkGenetic::saveToFile(std::ofstream &stream){
    std::cerr<<"saving to file."<<std::endl;
    std::cerr.precision(2);
    for(int itr=0; itr<host_genetics.size; itr++){
        stream<< host_genetics.array[itr]<<"\n";
        if(itr%(host_genetics.size/10) == 0){
            std::cerr<<(float)itr/(float)host_genetics.size<<std::endl;
        }
    }
    CUDA_SAFE_CALL(hipDeviceReset());
}

void NetworkGenetic::confDeviceParams(){
    this->setParams(10, _streamSize/_hostParams.array[2]); // number of individuals on device
    this->setParams(11, 0);
    this->setParams(12, _hostParams.array[11] + _hostParams.array[10] * _hostParams.array[1]);  // input neurons offset. (weights_offset + numweights*numindividuals)
    this->setParams(13, _hostParams.array[12] + _hostParams.array[10] * _hostParams.array[3]);  // hidden neurons offset. (input_offset +numInputs*numIndividuals)
    this->setParams(14, _hostParams.array[13] + _hostParams.array[10] * _hostParams.array[4]);  // short memory neurons offset. (hidden_offset + numHidden*numIndividuals)
    this->setParams(28, _hostParams.array[14] + _hostParams.array[10] * _hostParams.array[5]);  //long memory neurons offset (smem_offset + numsmem*numIndividuals)
    this->setParams(15, _hostParams.array[14] + _hostParams.array[10] * _hostParams.array[27]);  // memoryIn Gate nodes offset. (lmem_offset + numlMem*numIndividuals)
    this->setParams(16, _hostParams.array[15] + _hostParams.array[10] * _hostParams.array[6]);  // memoryOut Gate nodes offset. (memIn_offset + numMemIn*numIndividuals)
    this->setParams(17, _hostParams.array[16] + _hostParams.array[10] * _hostParams.array[7]);  // memoryForget Gate nodes offset. (memOut_offset + numMemOut*numIndividuals)
    this->setParams(18, _hostParams.array[17] + _hostParams.array[10] * _hostParams.array[8]);  // output neurons offset. (memForget_offset + numMemOut*numIndividuals)
    this->setParams(19, _hostParams.array[18] + _hostParams.array[10] * _hostParams.array[9]);  // fitness offset.
    this->setParams(20, _hostParams.array[20] + _hostParams.array[10] * 1);                     // community magnitude offset
    this->setParams(21, _hostParams.array[20] + _hostParams.array[10] * _hostParams.array[23]); // when offset.
    this->setParams(22, _hostParams.array[21] + _hostParams.array[10] * _hostParams.array[23]); // howCertain offset.

    CUDA_SAFE_CALL(hipMalloc((void**)&_deviceParams.array, _hostParams.size*sizeof(int)));
    std::cerr<<"number of individuals in stream is: "<<_hostParams.array[10]<<std::endl;
    std::cerr<<"size of connections array: "<<_hostParams.array[26]<<std::endl;
    CUDA_SAFE_CALL(hipMemcpy(_deviceParams.array, _hostParams.array, _hostParams.size*sizeof(int), hipMemcpyHostToDevice));
    _deviceParams.size = _hostParams.size;
}

void NetworkGenetic::setParams(int num, int val){
    _hostParams.array[num] = val;
}




void NetworkGenetic::reformatTraining(std::vector<int>&old_input, std::vector<double> &ans, std::vector<double> &sitedata, std::vector<double>&globalquakes, double &kp){ // increase the timestep and reduce resolution, takes too long.
    int trainingSize = 5;
    int * new_input = new int[trainingSize*3*_hostParams.array[23]];
    int *siteOffset = new int[15], *chanOffset = new int[3];
    long long stor[trainingSize*3*_hostParams.array[23]];
    memset(stor, 0, trainingSize*3*_hostParams.array[23]*sizeof(long long));
    siteOffset[0] = 0;
    for(int i=1; i<_hostParams.array[23]; i++){
        siteOffset[i] = trainingSize*3 + siteOffset[i-1];
    }
    chanOffset[0] = 0;
    for(int i=1; i<3; i++){
        chanOffset[i] = trainingSize + chanOffset[i-1];
    }

    for(int step=0; step<trainingSize; step++){
        for(int i=0; i<_hostParams.array[24]*3600/trainingSize; i++){
            for(int k=0; k<_hostParams.array[23]; k++){
                for(int j=0; j<3; j++){
                    stor[siteOffset[k]+chanOffset[j]+step] += old_input.at(k*_hostParams.array[24]*3600*3 + j*_hostParams.array[24]*3600 + step*_hostParams.array[24]*3600/trainingSize+i);
                }
            }
        }
    }
    for(int step=0; step<trainingSize; step++){
        for(int k=0; k<_hostParams.array[23]; k++){
            for(int j=0; j<3; j++){
                stor[siteOffset[k]+chanOffset[j]+step] = stor[siteOffset[k]+chanOffset[j]+step]/(_hostParams.array[24]*3600/trainingSize);
                new_input[siteOffset[k]+chanOffset[j]+step] = stor[siteOffset[k]+chanOffset[j]+step];
            }
        }
    }

    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(answers), ans.data(), ans.size()*sizeof(double), 0, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(siteData), sitedata.data(), sitedata.size()*sizeof(double), 0, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(globalQuakes),globalquakes.data(), globalquakes.size()*sizeof(double), 0, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(Kp), &kp, sizeof(double), 0, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(inputData), new_input, trainingSize*_hostParams.array[23]*3*sizeof(int), 0,  hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(trainingsize), &trainingSize, sizeof(int), 0, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(site_offset), siteOffset, _hostParams.array[23]*sizeof(int), 0, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(channel_offset), chanOffset, 3*sizeof(int), 0, hipMemcpyHostToDevice));
    delete[] siteOffset;
    delete[] chanOffset;
    CUDA_SAFE_CALL(hipDeviceSynchronize());
}

void NetworkGenetic::trainForecast(std::vector<double> *ret, int &hour, std::vector<int> &data,
                                   double &Kp, std::vector<double> &globalQuakes,
                                   Order *connections, std::vector<double> &ans, std::vector<double> &siteData){
    //were going to normalize the inputs using v` = v-mean/stdev, so we need mean and stdev for each channel.
    double meanCh[3]{0}, stdCh[3]{0};
    int num=0;
    std::cerr<<"starting training"<<std::endl;
    for(int i=0; i<3600*_hostParams.array[24]; i++){
        for(int j=0; j < _hostParams.array[23]; j++){
            for(int k=0; k<3; k++){
                meanCh[k] += data.at(3600*_hostParams.array[24]*j*3 + k*3600*_hostParams.array[24]+i);
            }

            num++;
        }
    }
    for(int k=0; k<3; k++){
        meanCh[k] = meanCh[k]/num;
        stdCh[k] = sqrt(meanCh[k]);
    }
    //input data from all sites and all channels normalized
    kernelArray<double> retVec, dmeanCh, dstdCh;

    int regBlockSize = 512;
    int regGridSize = (_hostParams.array[10])/regBlockSize;
    retVec.size = 2160*_hostParams.array[23];
    Order *dConnect;

    CUDA_SAFE_CALL(hipMalloc((void**)&retVec.array, ret->size()*sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc((void**)&dmeanCh.array, 3*sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc((void**)&dstdCh.array, 3*sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc((void**)&dConnect, _hostParams.array[26]*sizeof(Order)));

    CUDA_SAFE_CALL(hipMemcpy(dConnect, connections, _hostParams.array[26]*sizeof(Order), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(dmeanCh.array, meanCh, 3*sizeof(double), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(dstdCh.array, stdCh, 3*sizeof(double), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemset(retVec.array, 0, retVec.size*sizeof(double)));

    this->reformatTraining(data, ans, siteData,  globalQuakes, Kp);

    size_t host_offset = 0;
    size_t device_offset=0;

    hipEvent_t waitForLastStream;
    hipEventCreate(&waitForLastStream);
    std::cerr<<"forcast training loop for hour: "<<hour<<std::endl;
    for(int n=0; n<_numOfStreams; n++){
        if(n%2==0 && n!=0){
            device_offset=0;
            CUDA_SAFE_CALL(hipEventRecord(waitForLastStream, _stream[n-2]));
        }

        CUDA_SAFE_CALL(hipStreamWaitEvent(_stream[n], waitForLastStream, 0));
        CUDA_SAFE_CALL(hipPeekAtLastError());
        std::cerr<<"stream number #"<<n+1<<std::endl;
        CUDA_SAFE_CALL(hipMemcpyAsync(&device_genetics.array[device_offset], &host_genetics.array[host_offset], _streambytes, hipMemcpyHostToDevice, _stream[n]));

        NetKern<<<regGridSize, regBlockSize, _hostParams.array[26]*sizeof(Order), _stream[n]>>>(device_genetics,_deviceParams, dConnect, hour, dmeanCh, dstdCh, device_offset);

        CUDA_SAFE_CALL(hipMemcpyAsync(&host_genetics.array[host_offset], &device_genetics.array[device_offset], _streambytes, hipMemcpyDeviceToHost, _stream[n]));
        host_offset += _streamSize;
        device_offset += _streamSize;
    }
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    std::cerr.precision(40);
    for(int n=0; n<5; n++){
        std::cerr<<host_genetics.array[_hostParams.array[19]+n]<<std::endl;

        std::cerr<<"first weight is: "<<host_genetics.array[_hostParams.array[11]+n]<<std::endl;

            std::cerr<<"hidden 1 of "<<host_genetics.array[_hostParams.array[13]  + n ]<<std::endl;

            std::cerr<<"output 1 of "<<host_genetics.array[_hostParams.array[18] + n ]<<std::endl;

            for(int i=0; i<_hostParams.array[23]; i++){
                std::cerr<<"for site: "<<i<<std::endl;
                 std::cerr<<"certainty of: "<<host_genetics.array[_hostParams.array[22]+n+i*_hostParams.array[10]]<<std::endl;
                std::cerr<<"when of: "<<host_genetics.array[_hostParams.array[21]+n+i*_hostParams.array[10]]<<std::endl;

            }

        std::cerr<<std::endl;
    }
    CUDA_SAFE_CALL(hipFree(dConnect));
    CUDA_SAFE_CALL(hipFree(retVec.array));
    CUDA_SAFE_CALL(hipFree(dmeanCh.array));
    CUDA_SAFE_CALL(hipFree(dstdCh.array));

}

void NetworkGenetic::endOfTrial(){
    std::cerr<<"end of trial reached."<<std::endl;

    int regBlockSize = 512;
    int regGridSize = (_hostParams.array[10])/regBlockSize;

    kernelArray<double> partial_reduce_sums;
    double *hfitnessAvg, *dfitnessAvg;
    int *hparentChildCutoff, *dparentChildCutoff;
    int *evoGridSize;
    partial_reduce_sums.size = regBlockSize*_numOfStreams;
    CUDA_SAFE_CALL(hipHostAlloc((void**)&hparentChildCutoff, _numOfStreams*sizeof(int), hipHostMallocWriteCombined));
    CUDA_SAFE_CALL(hipHostAlloc((void**)&hfitnessAvg, _numOfStreams*sizeof(double), hipHostMallocWriteCombined));
    CUDA_SAFE_CALL(hipMalloc((void**)&partial_reduce_sums.array, regBlockSize*_numOfStreams*sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc((void**)&evoGridSize, _numOfStreams*sizeof(int)));
    CUDA_SAFE_CALL(hipMalloc((void**)&dparentChildCutoff, _numOfStreams*sizeof(int)));
    CUDA_SAFE_CALL(hipMalloc((void**)&dfitnessAvg, _numOfStreams*sizeof(double)));
    partial_reduce_sums.size = (regGridSize);

    size_t host_offset = 0;
    size_t device_offset=0;
    size_t *seed = new size_t[_numOfStreams];

    for(int i=0; i<_numOfStreams; i++){//set random numbers for evolution seed
        FILE *fp;
        fp = std::fopen("/dev/urandom", "r");
        size_t chk;
        chk =std::fread(&seed[i], 4, 1, fp);
        if(chk <1){std::cerr<<"couldn't read /dev/urandom"<<std::endl; exit(1);}
        std::fclose(fp);
    }

    hipEvent_t waitForLastStream;
    hipEventCreate(&waitForLastStream);

    for(int n=0; n<_numOfStreams; n++){
        if(n%2==0 && n!=0){
            device_offset=0;
            CUDA_SAFE_CALL(hipEventRecord(waitForLastStream, _stream[n-2]));
        }

        CUDA_SAFE_CALL(hipStreamWaitEvent(_stream[n], waitForLastStream, 0));
        CUDA_SAFE_CALL(hipPeekAtLastError());
        std::cerr<<"stream number #"<<n+1<<std::endl;

        CUDA_SAFE_CALL(hipMemcpyAsync(&device_genetics.array[device_offset], &host_genetics.array[host_offset], _streambytes, hipMemcpyHostToDevice, _stream[n]));
        CUDA_SAFE_CALL(hipPeekAtLastError());

        for(int k=2; k<= _hostParams.array[10]; k<<= 1){
            for(int j =k>>1; j>0; j=j>>1){
                bitonicSortKern<<<regGridSize, regBlockSize, 0, _stream[n]>>>(device_genetics, _deviceParams, j, k, device_offset);
            }
        }
        CUDA_SAFE_CALL(hipPeekAtLastError());

        reduceFirstKern<<<regGridSize, regBlockSize, regBlockSize*sizeof(double), _stream[n]>>>(device_genetics, partial_reduce_sums, _deviceParams, device_offset);
        CUDA_SAFE_CALL(hipPeekAtLastError());

        reduceSecondKern<<<1, 1, 0, _stream[n]>>>(partial_reduce_sums, _deviceParams, &dfitnessAvg[n]);
        CUDA_SAFE_CALL(hipPeekAtLastError());

        normalizeKern<<<regGridSize, regBlockSize, 0, _stream[n]>>>(device_genetics, _deviceParams, &dfitnessAvg[n], device_offset);
        CUDA_SAFE_CALL(hipPeekAtLastError());

        CUDA_SAFE_CALL(hipMemcpyAsync(&hfitnessAvg[n], &dfitnessAvg[n], sizeof(double), hipMemcpyDeviceToHost, _stream[n]));
        CUDA_SAFE_CALL(hipPeekAtLastError());

        cutoffKern<<<regGridSize, regBlockSize, 0, _stream[n]>>>(device_genetics, _deviceParams,  &dparentChildCutoff[n], &evoGridSize[n], &dfitnessAvg[n], device_offset);
        CUDA_SAFE_CALL(hipPeekAtLastError());

        CUDA_SAFE_CALL(hipMemcpyAsync(&hparentChildCutoff[n], &dparentChildCutoff[n], sizeof(int), hipMemcpyDeviceToHost, _stream[n]));
        CUDA_SAFE_CALL(hipPeekAtLastError());

        evolutionKern<<<regGridSize, regBlockSize, 0, _stream[n]>>>(device_genetics, _deviceParams, &dparentChildCutoff[n], &evoGridSize[n], seed[n], device_offset);
        CUDA_SAFE_CALL(hipPeekAtLastError());

        CUDA_SAFE_CALL(hipMemcpyAsync(&host_genetics.array[host_offset], &device_genetics.array[device_offset], _streambytes, hipMemcpyDeviceToHost, _stream[n]));
        CUDA_SAFE_CALL(hipPeekAtLastError());
        host_offset += _streamSize;
        device_offset += _streamSize;
    }
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    std::cerr<<"average fitness for first stream is: "<< hfitnessAvg[0]<<std::endl;

    CUDA_SAFE_CALL(hipFree(partial_reduce_sums.array));
    CUDA_SAFE_CALL(hipHostFree(hfitnessAvg));
    CUDA_SAFE_CALL(hipHostFree(hparentChildCutoff));
    CUDA_SAFE_CALL(hipFree(dfitnessAvg));
    CUDA_SAFE_CALL(hipFree(evoGridSize));
    CUDA_SAFE_CALL(hipFree(dparentChildCutoff));
    CUDA_SAFE_CALL(hipFree(device_genetics.array));
    delete[] seed;
}

void NetworkGenetic::challengeForecast(std::vector<double> *ret, int &hour, std::vector<int> &data, double &Kp,
                                       std::vector<double> &globalQuakes, Order *connections, std::vector<double> &siteData){
    //were going to normalize the inputs using v` = v-mean/stdev, so we need mean and stdev for each channel.
    //    double meanCh[3]{0}, stdCh[3]{0};
    //    int num=0;
    //    for(int i=0; i<3600*_hostParams.array[24]; i++){
    //        for(int j=0; j < _hostParams.array[23]; j++){
    //            for(int k=0; k<3; k++){
    //                meanCh[k] += data.at(3600*_hostParams.array[24]*j*3 + k*3600*_hostParams.array[24]+i);
    //            }
    //            num++;
    //        }
    //    }
    //    for(int k=0; k<3; k++){
    //        meanCh[k] = meanCh[k]/num;
    //        stdCh[k] = sqrt(meanCh[k]);
    //    }
    //    std::cerr<<"entered not training version.."<<std::endl;
    //    //replace this later
    //    //        _best.resize(_hostParams.array[1]);
    //    //        for(std::vector<double>::iterator it = _best.begin(); it != _best.end(); ++it){
    //    //            std::srand(std::time(NULL)+*it);
    //    //            *it = (double)(std::rand())/(RAND_MAX);
    //    //        }
    //    std::cerr<<"example best vector has been set."<<std::endl;
    //    double CommunityLat = 0;
    //    double CommunityLon = 0;
    //    std::vector<double> When(_hostParams.array[23], 0);
    //    std::vector<double> HowCertain(_hostParams.array[23],0);
    //    std::vector<double> CommunityMag(_hostParams.array[23], 1); //give all sites equal mag to start, this value is [0,1]
    //    std::cerr<<"all output vectors created and initialized."<<std::endl;
    //    for(int step=0; step<3600*_hostParams.array[24]; step++){
    //        for(int j=0; j<_hostParams.array[23]; j++){ //sitesWeighted Lat/Lon values are determined based on all previous sites mag output value.
    //            CommunityLat += siteData.at(j*2)*CommunityMag[j];
    //            CommunityLon += siteData.at(j*2+1)*CommunityMag[j];
    //        }
    //        CommunityLat = CommunityLat/_hostParams.array[23];
    //        CommunityLon = CommunityLon/_hostParams.array[23];

    //        for(int j=0; j<_hostParams.array[23]; j++){ // each site is run independently of others, but shares an output from the previous step
    //            double latSite = siteData.at(j*2);
    //            double lonSite = siteData.at(j*2+1);
    //            double avgLatGQuake = globalQuakes.at(0);
    //            double avgLonGQuake = globalQuakes.at(1);
    //            double GQuakeAvgMag = globalQuakes.at(3);
    //            double GQuakeAvgdist = distCalc(latSite, lonSite, avgLatGQuake, avgLonGQuake);
    //            double GQuakeAvgBearing = bearingCalc(latSite, lonSite, avgLatGQuake, avgLonGQuake);
    //            double CommunityDist = distCalc(latSite, lonSite, CommunityLat, CommunityLon);
    //            double CommunityBearing = bearingCalc(latSite, lonSite, CommunityLat, CommunityLon);
    //            std::vector<double> input;
    //            std::vector<double> hidden, output, mem, memGateOut, memGateIn, memGateForget;
    //            //replace these with real connections, num of inputs, and num of hidden & memory neurons (mem neurons probably accurate)
    //            input.resize(_hostParams.array[2], 0); // number of inputs is 9.
    //            hidden.resize(_hostParams.array[10], 0); // for practice sake, lets say each input has its own neuron (might be true!)
    //            mem.resize(_hostParams.array[11], 0); // stores the input if gate is high
    //            memGateOut.resize(_hostParams.array[11], 0); //connects to the input layer and the memN associated with input, if 1 it sends up stream and deletes, if low it keeps.
    //            memGateIn.resize(_hostParams.array[11], 0);
    //            memGateForget.resize(_hostParams.array[11], 0);
    //            output.resize(_hostParams.array[12], 0); /* 3 outputs, 1 with an hour in the future when the earthquake will hit,
    //                    1 with the porbability of that earthquake happening (between [0,1]) and 1 with the sites magnitude (for community feedback) */
    //            int n =0;
    //            for(int k=0; k<3; k++){
    //                input[k] = normalize((double)(data.at(3600*_hostParams.array[24]*j*3 + k*(3600*_hostParams.array[24])+step)), meanCh[k], stdCh[k]);

    //            }
    //            input[3] = shift(GQuakeAvgdist, 40075.1, 0);
    //            input[4] = shift(GQuakeAvgBearing, 360, 0);
    //            input[5] = shift(GQuakeAvgMag, 9.5, 0);
    //            input[6] = shift(Kp, 10, 0);
    //            input[7] = shift(CommunityDist,40075.1/2, 0);
    //            input[8] = shift(CommunityBearing, 360, 0);
    //            //lets reset all neuron values for this new timestep (except memory neurons)
    //            for(int itr=0; itr< _hostParams.array[26]; itr++){//every order is sequential and run after the previous order to massively simplify the workload in this kernel.

    //                //set stuff to zero
    //                if(connections[itr].first.def == typeInput && connections[itr].second.def == typeZero)
    //                    neuroZero(input[connections[itr].first.id]);

    //                else if(connections[itr].first.def == typeHidden && connections[itr].second.def == typeZero)
    //                    neuroZero(hidden[connections[itr].first.id]);

    //                else if(connections[itr].first.def == typeMemGateIn && connections[itr].second.def == typeZero)
    //                    neuroZero(memGateIn[+connections[itr].first.id]);

    //                else if(connections[itr].first.def == typeMemGateOut && connections[itr].second.def == typeZero)
    //                    neuroZero(memGateOut[connections[itr].first.id]);

    //                else if(connections[itr].first.def == typeMemGateForget && connections[itr].second.def == typeZero)
    //                    neuroZero(memGateForget[connections[itr].first.id]);

    //                else if(connections[itr].first.def == typeMemory && connections[itr].second.def == typeZero)
    //                    neuroZero(mem[connections[itr].first.id]);

    //                else if(connections[itr].first.def == typeOutput && connections[itr].second.def == typeZero)
    //                    neuroZero(output[connections[itr].first.id]);

    //                //first->second summations
    //                else if(connections[itr].first.def == typeInput && connections[itr].second.def == typeHidden)
    //                    neuroSum(hidden[connections[itr].second.id],
    //                            (input[connections[itr].first.id])*(_best[n++]));

    //                else if(connections[itr].first.def == typeInput && connections[itr].second.def == typeMemGateIn)
    //                    neuroSum(memGateIn[ + connections[itr].second.id],
    //                            (input[connections[itr].first.id])*(_best[n++]));

    //                else if(connections[itr].first.def == typeInput && connections[itr].second.def == typeMemGateOut)
    //                    neuroSum(memGateIn[connections[itr].second.id],
    //                            (input[connections[itr].first.id])*(_best[n++]));

    //                else if(connections[itr].first.def == typeInput && connections[itr].second.def == typeMemGateForget)
    //                    neuroSum(memGateForget[connections[itr].second.id],
    //                            (input[connections[itr].first.id])*(_best[n++]));

    //                else if(connections[itr].first.def == typeHidden && connections[itr].second.def == typeHidden)
    //                    neuroSum(hidden[connections[itr].second.id],
    //                            (hidden[connections[itr].first.id])*(_best[n++]));

    //                else if(connections[itr].first.def == typeHidden && connections[itr].second.def == typeMemGateIn)
    //                    neuroSum(memGateIn[connections[itr].second.id],
    //                            (hidden[connections[itr].first.id])*(_best[n++]));

    //                else if(connections[itr].first.def == typeHidden && connections[itr].second.def == typeMemGateOut)
    //                    neuroSum(memGateIn[connections[itr].second.id],
    //                            (hidden[connections[itr].first.id])*(_best[n++]));

    //                else if(connections[itr].first.def == typeHidden && connections[itr].second.def == typeMemGateForget)
    //                    neuroSum(memGateForget[connections[itr].second.id],
    //                            (hidden[connections[itr].first.id])*(_best[n++]));

    //                //memory gates
    //                else if(connections[itr].first.def == typeInput && connections[itr].second.def == typeMemory && connections[itr].third.def == typeMemGateIn)
    //                    neuroMemGate(memGateIn[connections[itr].third.id],
    //                            input[connections[itr].first.id],
    //                            mem[connections[itr].second.id], 0.5);

    //                else if(connections[itr].first.def == typeHidden && connections[itr].second.def == typeMemory && connections[itr].third.def == typeMemGateIn)
    //                    neuroMemGate(memGateIn[+connections[itr].third.id],
    //                            hidden[connections[itr].first.id],
    //                            mem[connections[itr].second.id], 0.5);

    //                else if(connections[itr].first.def == typeOutput && connections[itr].second.def == typeMemory && connections[itr].third.def == typeMemGateIn)
    //                    neuroMemGate(memGateIn[connections[itr].third.id],
    //                            output[connections[itr].first.id],
    //                            mem[connections[itr].second.id], 0.5);

    //                else if(connections[itr].first.def == typeMemory && connections[itr].second.def == typeHidden && connections[itr].third.def == typeMemGateOut)
    //                    neuroMemGate(memGateOut[connections[itr].third.id],
    //                            mem[connections[itr].first.id],
    //                            hidden[connections[itr].second.id], 0.5);

    //                else if(connections[itr].first.def == typeMemory && connections[itr].second.def == typeOutput && connections[itr].third.def == typeMemGateOut)
    //                    neuroMemGate(memGateOut[connections[itr].third.id],
    //                            mem[connections[itr].first.id],
    //                            output[connections[itr].second.id], 0.5);

    //                else if(connections[itr].first.def == typeMemory && connections[itr].second.def == typeMemGateForget)
    //                    neuroMemForget(memGateForget[connections[itr].second.id],
    //                            mem[connections[itr].first.id], 0.5);
    //                //bias
    //                else if(connections[itr].first.def == typeBias && connections[itr].second.def == typeHidden)
    //                    neuroSum(hidden[connections[itr].second.id], (1*(_best[n++])));

    //                else if(connections[itr].first.def == typeBias && connections[itr].second.def == typeMemGateIn)
    //                    neuroSum(memGateIn[connections[itr].second.id], (1*(_best[n++])));

    //                else if(connections[itr].first.def == typeBias && connections[itr].second.def == typeMemGateOut)
    //                    neuroSum(memGateIn[connections[itr].second.id], (1*(_best[n++])));

    //                else if(connections[itr].first.def == typeBias && connections[itr].second.def == typeMemGateForget)
    //                    neuroSum(memGateForget[connections[itr].second.id], (1*(_best[n++])));

    //                else if(connections[itr].first.def == typeBias && connections[itr].second.def == typeOutput)
    //                    neuroSum(output[connections[itr].second.id], (1*(_best[n++])));

    //                //squashing
    //                else if(connections[itr].first.def == typeHidden && connections[itr].second.def == typeSquash)
    //                    neuroSquash(hidden[connections[itr].second.id]);

    //                else if(connections[itr].first.def == typeMemGateIn && connections[itr].second.def == typeSquash)
    //                    neuroSquash(memGateIn[ + connections[itr].second.id]);

    //                else if(connections[itr].first.def == typeMemGateOut && connections[itr].second.def == typeSquash)
    //                    neuroSquash(memGateIn[connections[itr].second.id]);

    //                else if(connections[itr].first.def == typeMemGateForget && connections[itr].second.def == typeSquash)
    //                    neuroSquash(memGateForget[connections[itr].second.id]);

    //                else if(connections[itr].first.def == typeOutput && connections[itr].second.def == typeSquash)
    //                    neuroSquash(output[connections[itr].second.id]);

    //            }


    //            When[j] += output[0]*((2160-hour)-hour)+2160-hour; //return when back to an integer value (adjust to fit within boundaries)
    //            HowCertain[j] += output[1];
    //            CommunityMag[j] =  output[2]; // set the next sets communityMag = output #3.
    //        }
    //    }
    //        float maxCertainty=0;
    //        float whenGuess=0;
    //        float guessLat=0;
    //        float guessLon=0;
    //        for(int j=0; j<_hostParams.array[23]; j++){
    //            if(HowCertain[j] > maxCertainty){
    //                maxCertainty = HowCertain[j];
    //                whenGuess = When[j];
    //                guessLat = siteData.at(j*2);
    //                guessLon = siteData.at(j*2+1);
    //            }
    //        }

    //        int whenAns = (int)_answers[1]-hour;
    //        double oldFit = ret->at(0);
    //        ret->at(0) = scoreFunc(whenGuess, whenAns, guessLat, guessLon, ansLat, ansLon, oldFit);//larger is better, negative numbers are impossible.
}
