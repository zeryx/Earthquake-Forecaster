#include "hip/hip_runtime.h"
﻿#include <network.h>
#include <kernelDefs.h>
#include <getsys.h>
#include <fstream>
#include <sstream>
#include <ostream>
#include <vector>
#include <ctime>
#include <cstdio>
#include <assert.h>


NetworkGenetic::NetworkGenetic(const int &numInputNodes, const int &numHiddenNeurons, const int &numMemoryNeurons,
                               const int &numOutNeurons, const int &numWeights, std::vector< std::pair<int, int> >&connections){

    _hostParams.array = new int[26];
    _hostParams.size=23;
    _hostParams.array[0] = numInputNodes + numHiddenNeurons + numMemoryNeurons*4 + numOutNeurons;
    _hostParams.array[1] = numWeights;
    //    _hostParams.array[2] = _hostParams.array[0] + _hostParams.array[1] + 1 + 3*_numofSites; //1*numOfSites for community mag, 1*numOfSites for When, 1*numOfSites for HowCertain,  1 for fitness
    _hostParams.array[3] = numInputNodes;
    _hostParams.array[4] = numHiddenNeurons;
    _hostParams.array[5] =numMemoryNeurons;            //memory neurons per individual
    _hostParams.array[6] =numMemoryNeurons;            //memoryIn neurons per individual
    _hostParams.array[7] =numMemoryNeurons;            //memoryOut neurons per individual
    _hostParams.array[8] = numMemoryNeurons;       //memoryForget neurons per individual
    _hostParams.array[9] =numOutNeurons;           //output neurons per individual
    //    _hostParams.array[10] = number of individuals in stream
    //    _hostParams.array[11] = weights offset
    //    _hostParams.array[12] = input offset
    //    _hostParams.array[13] = hidden neurons offset
    //    _hostParams.array[14] = memory neurons offset
    //    _hostParams.array[15] = memoryIn neurons offset
    //    _hostParams.array[16] = memoryOut neurons offset
    //    _hostParams.array[17] = memoryForget neurons offset
    //    _hostParams.array[18] = output neurons offset
    //    _hostParams.array[19] = fitness offset
    //    _hostParams.array[20] = community magnitude offset
    //    _hostParams.array[21] = when offset
    //    _hostParams.array[22] = howCertain offset
    //    _deviceParams.array[23] = start of children offset
    _hostParams.array[25] = 1 ;// population ratio next_pop_size/current_pop_size, ie 1 = constant

    _connect = &connections;
}

void NetworkGenetic::generateWeights(){
    int blockSize = 512; // number of blocks in the grid
    int gridSize=(_hostParams.array[10]*_hostParams.array[1])/blockSize; //number of weights in stream/blocksize
    size_t global_offset=0;
    size_t device_offset=0;
    std::cerr<<"blocks in this grid: "<<gridSize<<std::endl;
    for(int n=0; n<_numOfStreams; n++){//fill the host first.
        if(n%2==0 && n !=0)
            device_offset =0;
        size_t seed;
        FILE *fp;
        fp = std::fopen("/dev/urandom", "r");
        size_t chk;
        chk =std::fread(&seed, 4, 1, fp);
        if(chk <1){std::cerr<<"couldn't read /dev/urandom"<<std::endl; exit(1);}
        std::fclose(fp);
        std::cerr<<"stream number #"<<n<<std::endl;
        std::cerr<<"global offset: "<<global_offset<<std::endl;
        std::cerr<<"device offset: "<<device_offset<<std::endl<<std::endl;
        genWeightsKern<<< gridSize, blockSize, 0, _stream[n]>>>(device_genetics, seed, _deviceParams, device_offset);
        CUDA_SAFE_CALL(hipPeekAtLastError());
        CUDA_SAFE_CALL(hipMemcpyAsync(&host_genetics.array[global_offset], &device_genetics.array[device_offset], _streambytes, hipMemcpyDeviceToHost, _stream[n]));
        global_offset += _streamSize;
        device_offset += _streamSize;
    }
}


void NetworkGenetic::allocateHostAndGPUObjects( float pMax, size_t deviceRam, size_t hostRam){
    CUDA_SAFE_CALL(hipDeviceReset());
    CUDA_SAFE_CALL(hipFuncSetCacheConfig(reinterpret_cast<const void*>(NetKern), hipFuncCachePreferL1));
    CUDA_SAFE_CALL(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeFourByte));
    size_t totalHost = hostRam*pMax;
    size_t totalDevice = deviceRam*pMax;
    std::cerr<<"total free device ram : "<<deviceRam<<std::endl;
    std::cerr<<"total free host ram : "<<hostRam<<std::endl;
    //each memory block is divisible by the size of an individual, the size of double, and the blocksize
    while(totalHost%_hostParams.array[2] || totalHost%sizeof(double) || totalHost%(sizeof(double)*2) || totalHost%512) // get the largest number divisible by the individual size, the threads in a block, and the size of a double
        totalHost= totalHost -1;
    while(totalDevice%_hostParams.array[2] || totalDevice%sizeof(double) || totalDevice%(sizeof(double)*2) || totalDevice%512)
        totalDevice = totalDevice -1;

    _streamSize = totalDevice/(sizeof(double)*2);
    _streambytes = _streamSize*sizeof(double);

    assert(_streambytes == _streamSize*sizeof(double));
    _numOfStreams = ceil((totalHost/sizeof(double))/_streamSize); // number of streams = totalHost/streamBytes, device does not store extra weights for simplicity.
    assert(_streambytes * _numOfStreams <= totalDevice+totalHost);
    std::cerr<<"number of streams: "<<_numOfStreams<<std::endl;
    device_genetics.size = (totalDevice)/sizeof(double);
    host_genetics.size = totalHost/sizeof(double);
    std::cerr<<"device ram to allocate: "<<totalDevice<<std::endl;
    std::cerr<<"host ram to allocate: "<<totalHost<<std::endl;
    CUDA_SAFE_CALL(hipHostAlloc((void**)&host_genetics.array, totalHost, hipHostMallocWriteCombined));
    CUDA_SAFE_CALL(hipMalloc((void**) &device_genetics.array, totalDevice));
    std::fill(host_genetics.array, host_genetics.array+host_genetics.size, 0);
    CUDA_SAFE_CALL(hipMemset(device_genetics.array, 0, totalDevice));
    std::cerr<<"all allocated, moving on."<<std::endl;
    _stream.resize(_numOfStreams);
    for(int i=0; i<_numOfStreams; i++){
        CUDA_SAFE_CALL(hipStreamCreate(&_stream.at(i)));
        CUDA_SAFE_CALL( hipStreamQuery(_stream.at(i)));
    }
    this->setParams();
}
bool NetworkGenetic::init(int sampleRate, int SiteNum, std::vector<double> *siteData){
    _sampleRate = sampleRate;
    _numofSites = SiteNum;
    _siteData = siteData;
    _istraining = false;
    _hostParams.array[2] = _hostParams.array[0] + _hostParams.array[1] + 1 + 3*_numofSites; //1*numOfSites for community mag, 1 for fitness
    return true;
}

void NetworkGenetic::setParams(){
    _hostParams.array[10] = _streamSize/_hostParams.array[2];
    _hostParams.array[11] = 0;
    _hostParams.array[12] = _hostParams.array[11] + _hostParams.array[10] * _hostParams.array[1];  // input neurons offset. (weights_offset + numweights*numindividuals)
    _hostParams.array[13] = _hostParams.array[12] + _hostParams.array[10] * _hostParams.array[3]; // hidden neurons offset. (input_offset +numInputs*numIndividuals)
    _hostParams.array[14] = _hostParams.array[13] + _hostParams.array[10] * _hostParams.array[4]; // memory neurons offset. (hidden_offset + numHidden*numIndividuals)
    _hostParams.array[15] = _hostParams.array[14] + _hostParams.array[10] * _hostParams.array[5]; // memoryIn Gate nodes offset. (mem_offset + numMem*numIndividuals)
    _hostParams.array[16] = _hostParams.array[15] + _hostParams.array[10] * _hostParams.array[6];// memoryOut Gate nodes offset. (memIn_offset + numMemIn*numIndividuals)
    _hostParams.array[17] = _hostParams.array[16] + _hostParams.array[10] * _hostParams.array[7];
    _hostParams.array[18] = _hostParams.array[17] + _hostParams.array[10] *_hostParams.array[8]; // output neurons offset. (memOut_offset + numMemOut*numIndividuals)
    _hostParams.array[19] = _hostParams.array[18] + _hostParams.array[10] *_hostParams.array[9]; // fitness offset.
    _hostParams.array[20] = _hostParams.array[19] + _hostParams.array[10] *1; //community Magnitude offset
    _hostParams.array[21] = _hostParams.array[20] + _hostParams.array[10] *_numofSites; // when offset.
    _hostParams.array[22] = _hostParams.array[21] + _hostParams.array[10] *_numofSites; // howCertain offset.
    std::cerr<<"allocating params memory"<<std::endl;
    CUDA_SAFE_CALL(hipMalloc((void**)&_deviceParams.array, _hostParams.size*sizeof(int)));
    std::cerr<<"setting params memory"<<std::endl;
    CUDA_SAFE_CALL(hipMemcpy(_deviceParams.array, _hostParams.array, _hostParams.size*sizeof(int), hipMemcpyHostToDevice));
    _deviceParams.size = _hostParams.size;
}

bool NetworkGenetic::checkForWeights(std::string filepath){
    std::ifstream weightFile(filepath.c_str(), std::ios_base::ate | std::ios_base::binary);
    std::cerr<<"checking for weights.."<<std::endl;
    if(weightFile){
        std::cerr<<"the weightfile exists"<<std::endl;
        std::string line;
        int filesize = weightFile.tellg();
        weightFile.seekg(0, weightFile.beg);
        int itr =0;
        this->allocateHostAndGPUObjects(0.85, GetDeviceRamInBytes(), filesize - GetDeviceRamInBytes());
        for( int n=0; n<_numOfStreams; n++){
            int offset = n*_streambytes/sizeof(double);
            while(std::getline(weightFile, line) && itr <= device_genetics.size){ // each line
                std::string item;
                std::stringstream ss(line);
                while(std::getline(ss, item, ',') && itr <= device_genetics.size){ // each weight
                    device_genetics.array[itr] = std::atoi(item.c_str());
                }
            }
            CUDA_SAFE_CALL(hipMemcpyAsync(&host_genetics.array[offset], &device_genetics.array[offset], _streambytes, hipMemcpyDeviceToHost, _stream[n]));
            itr = 0;
        }
        weightFile.close();
        return true;
    }
    else{
        std::cerr<<"no weightfile found"<<std::endl;
        weightFile.close();
        return false;
    }
}

void NetworkGenetic::doingTraining(int site, int hour, double lat,
                                   double lon, double mag, double dist){
    _answers.push_back(site);
    _answers.push_back(hour);
    _answers.push_back(lat);
    _answers.push_back(lon);
    _answers.push_back(mag);
    _answers.push_back(dist);
    _istraining = true;
}

void NetworkGenetic::storeWeights(std::string filepath){
    std::ofstream ret;
    ret.open(filepath.c_str(), std::ios_base::out | std::ios_base::trunc);
    for(int i=0; i<device_genetics.size; i++){
        ret << device_genetics.array[i]<<","<<std::endl;
    }
    ret.close();
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    for(int i=0; i<_numOfStreams; i++){
        CUDA_SAFE_CALL(hipStreamDestroy(_stream[i]));
    }
    CUDA_SAFE_CALL(hipFree(device_genetics.array));
    CUDA_SAFE_CALL(hipFree(host_genetics.array));
    CUDA_SAFE_CALL(hipDeviceReset());
}

void NetworkGenetic::reformatTraining(std::vector<int>* old_input, std::vector<double> ans, std::vector<double>* sitedata, std::vector<double>* globalquakes, double kp){ // increase the timestep and reduce resolution, takes too long.
    int trainingSize = 10;
    int * new_input = new int[trainingSize*3*_numofSites];
    int *siteOffset = new int[15], *chanOffset = new int[3];
    long long stor[trainingSize*3*_numofSites];
    memset(stor, 0, trainingSize*3*_numofSites*sizeof(long long));
    siteOffset[0] = 0;
    for(int i=1; i<_numofSites; i++){
        siteOffset[i] = trainingSize*3 + siteOffset[i-1];
    }
    chanOffset[0] = 0;
    for(int i=1; i<3; i++){
        chanOffset[i] = trainingSize + chanOffset[i-1];
    }

    for(int step=0; step<trainingSize; step++){
        for(int i=0; i<_sampleRate*3600/trainingSize; i++){
            for(int k=0; k<_numofSites; k++){
                for(int j=0; j<3; j++){
                    stor[siteOffset[k]+chanOffset[j]+step] += old_input->at(k*_sampleRate*3600*3 + j*_sampleRate*3600 + step*_sampleRate*3600/trainingSize+i);
                }
            }
        }
    }
    for(int step=0; step<trainingSize; step++){
        for(int k=0; k<_numofSites; k++){
            for(int j=0; j<3; j++){
                stor[siteOffset[k]+chanOffset[j]+step] = stor[siteOffset[k]+chanOffset[j]+step]/(_sampleRate*3600/trainingSize);
                new_input[siteOffset[k]+chanOffset[j]+step] = stor[siteOffset[k]+chanOffset[j]+step];
            }
        }
    }

    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(answers), ans.data(), ans.size()*sizeof(double), 0, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(siteData), sitedata->data(), sitedata->size()*sizeof(double), 0, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(globalQuakes),globalquakes->data(), globalquakes->size()*sizeof(double), 0, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(Kp), &kp, sizeof(double), 0, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(input), new_input, trainingSize*_numofSites*3*sizeof(int), 0,  hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(trainingsize), &trainingSize, sizeof(int), 0, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(site_offset), siteOffset, _numofSites*sizeof(int), 0, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(channel_offset), chanOffset, 3*sizeof(int), 0, hipMemcpyHostToDevice));
    delete[] siteOffset;
    delete[] chanOffset;
    CUDA_SAFE_CALL(hipDeviceSynchronize());
}

void NetworkGenetic::forecast(std::vector<double> *ret, int &hour, std::vector<int> *data, double &Kp, std::vector<double> *globalQuakes)
{
    //were going to normalize the inputs using v` = v-mean/stdev, so we need mean and stdev for each channel.
    double meanCh[3]{0}, stdCh[3]{0};
    int num=0;
    for(int i=0; i<3600*_sampleRate; i++){
        for(int j=0; j < _numofSites; j++){
            for(int k=0; k<3; k++){
                meanCh[k] += data->at(3600*_sampleRate*j*3 + k*3600*_sampleRate+i);
            }
            num++;
        }
    }
    for(int k=0; k<3; k++){
        meanCh[k] = meanCh[k]/num;
        stdCh[k] = sqrt(meanCh[k]);
    }
    //input data from all sites and all channels normalized
    if(_istraining == true){
        if(hour == 20){
            hipDeviceReset();
            exit(1);
        }
        kernelArray<double>retVec, partial_reduce_sums, dmeanCh, dstdCh;
        kernelArray<std::pair<const int, const int> > dConnect;
        int regBlockSize = 512;
        int regGridSize = (_hostParams.array[10])/regBlockSize;
        int evoGridsize[_numOfStreams];
        retVec.size = 2160*_numofSites;
        dConnect.size = _connect->size();
        partial_reduce_sums.size = (regGridSize);
        double *hfitnessAvg, *dfitnessAvg;
        int *hchildOffset, *dchildOffset;
        CUDA_SAFE_CALL(hipHostAlloc((void**)&hchildOffset, _numOfStreams*sizeof(int), hipHostMallocWriteCombined));
        CUDA_SAFE_CALL(hipHostAlloc((void**)&hfitnessAvg, _numOfStreams*sizeof(double), hipHostMallocWriteCombined));
        CUDA_SAFE_CALL(hipMalloc((void**)&dchildOffset, _numOfStreams*sizeof(int)));
        CUDA_SAFE_CALL(hipMalloc((void**)&dfitnessAvg, _numOfStreams*sizeof(double)));
        CUDA_SAFE_CALL(hipMalloc((void**)&retVec.array, ret->size()*sizeof(double)));
        CUDA_SAFE_CALL(hipMalloc((void**)&dConnect.array, _connect->size()*sizeof(std::pair<int, int>)));
        CUDA_SAFE_CALL(hipMalloc((void**)&partial_reduce_sums.array, partial_reduce_sums.size*sizeof(double)));
        CUDA_SAFE_CALL(hipMalloc((void**)&dmeanCh.array, 3*sizeof(double)));
        CUDA_SAFE_CALL(hipMalloc((void**)&dstdCh.array, 3*sizeof(double)));
        CUDA_SAFE_CALL(hipMemcpy(dConnect.array, _connect->data(), _connect->size()*sizeof(std::pair<int, int>), hipMemcpyHostToDevice));
        CUDA_SAFE_CALL(hipMemcpy(dmeanCh.array, meanCh, 3*sizeof(double), hipMemcpyHostToDevice));
        CUDA_SAFE_CALL(hipMemcpy(dstdCh.array, stdCh, 3*sizeof(double), hipMemcpyHostToDevice));
        CUDA_SAFE_CALL(hipMemset(retVec.array, 0, retVec.size*sizeof(double)));
        this->reformatTraining(data, _answers, _siteData,  globalQuakes, Kp);
        size_t host_offset = 0;
        size_t device_offset=0;
        size_t *seed = new size_t[_numOfStreams];
        for(int i=0; i<_numOfStreams; i++){//set random numbers for evolution seed
            FILE *fp;
            fp = std::fopen("/dev/urandom", "r");
            size_t chk;
            chk =std::fread(&seed[i], 4, 1, fp);
            if(chk <1){std::cerr<<"couldn't read /dev/urandom"<<std::endl; exit(1);}
            std::fclose(fp);
        }
        for(int n=0; n<_numOfStreams; n++){
            if(n%2==0 && n!=0){
                device_offset=0;
            }
            if(n>=2){
                CUDA_SAFE_CALL(hipStreamSynchronize(_stream[n-2]));
                CUDA_SAFE_CALL(hipPeekAtLastError());
            }
            std::cerr<<"stream number: "<<n<<std::endl;
            CUDA_SAFE_CALL(hipMemcpyAsync(&device_genetics.array[device_offset], &host_genetics.array[host_offset], _streambytes, hipMemcpyHostToDevice, _stream[n]));

            NetKern<<<regGridSize, regBlockSize, _connect->size()*sizeof(std::pair<const int, const int>), _stream[n]>>>(device_genetics,_deviceParams,  dConnect, _numofSites, hour, dmeanCh, dstdCh, device_offset);


            reduceFirstKern<<<regGridSize, regBlockSize, regBlockSize*sizeof(double), _stream[n]>>>(device_genetics, partial_reduce_sums, _deviceParams, device_offset);

            reduceSecondKern<<<1, 1, 0, _stream[n]>>>(partial_reduce_sums, _deviceParams, &dfitnessAvg[n]);

            for(int k=2; k<= _hostParams.array[10]; k<<= 1){
                for(int j =k>>1; j>0; j=j>>1){
                    sortKern<<<regGridSize, regBlockSize, 0, _stream[n]>>>(device_genetics, _deviceParams, j, k, device_offset);
                }
            }

//            normalizeKern<<<regGridSize, regBlockSize, 0, _stream[n]>>>(device_genetics, _deviceParams, &dfitnessAvg[n], device_offset);

            findChildrenKern<<<regGridSize, regBlockSize, 0, _stream[n]>>>(device_genetics, _deviceParams,  &dchildOffset[n], &dfitnessAvg[n], device_offset);

            CUDA_SAFE_CALL(hipMemcpyAsync(&hchildOffset[n], &dchildOffset[n], sizeof(int), hipMemcpyDeviceToHost, _stream[n]));

            evoGridsize[n] = (_hostParams.array[10]-hchildOffset[n])/regBlockSize;

            evolutionKern<<<evoGridsize[n], regBlockSize, 0, _stream[n]>>>(device_genetics, _deviceParams, &dchildOffset[n], seed[n], device_offset);

            CUDA_SAFE_CALL(hipMemcpyAsync(&hfitnessAvg[n], &dfitnessAvg[n], sizeof(double), hipMemcpyDeviceToHost, _stream[n]));

            CUDA_SAFE_CALL(hipMemcpyAsync(&host_genetics.array[host_offset], &device_genetics.array[device_offset], _streambytes, hipMemcpyDeviceToHost, _stream[n]));
            host_offset += _streamSize;
            device_offset += _streamSize;
        }
        CUDA_SAFE_CALL(hipDeviceSynchronize());
        for(int j=0; j<_numOfStreams; j++){
            std::cerr<<"for stream #: "<<j<<std::endl;
            std::cerr<<"average fitness is: "<<hfitnessAvg[j]<<std::endl;
        }
        int ctr=0;
        for(int i=0; i<_hostParams.array[10]; i++){
            if(host_genetics.array[_hostParams.array[19] + i] >0)
                ctr++;
        }
        std::cerr<<"for stream num#:0 the number of better than average individuals is: "<<ctr<<std::endl;
        std::cerr<<"percentage %: "<<((double)ctr/(double)_hostParams.array[10])*100<<std::endl;

        std::cerr.precision(15);
        for(int i=0; i<25; i++){
            std::cerr<<host_genetics.array[_hostParams.array[19]+i]<<std::endl;
        }
        CUDA_SAFE_CALL(hipFree(dConnect.array));
        CUDA_SAFE_CALL(hipFree(retVec.array));
        CUDA_SAFE_CALL(hipFree(partial_reduce_sums.array));
        CUDA_SAFE_CALL(hipHostFree(hfitnessAvg));
        CUDA_SAFE_CALL(hipHostFree(hchildOffset));
        CUDA_SAFE_CALL(hipFree(dfitnessAvg));
        CUDA_SAFE_CALL(hipFree(dchildOffset));
        CUDA_SAFE_CALL(hipFree(dmeanCh.array));
        CUDA_SAFE_CALL(hipFree(dstdCh.array));
        delete[] seed;
    }
    else{
        std::cerr<<"entered not training version.."<<std::endl;
        typedef std::vector<std::pair<int, int> > connectPairMatrix;
        //replace this later
        _best.resize(_hostParams.array[1]);
        for(std::vector<double>::iterator it = _best.begin(); it != _best.end(); ++it){
            std::srand(std::time(NULL)+*it);
            *it = (double)(std::rand())/(RAND_MAX);
        }
        std::cerr<<"example best vector has been set."<<std::endl;
        double CommunityLat = 0;
        double CommunityLon = 0;
        std::vector<double> When(_numofSites, 0);
        std::vector<double> HowCertain(_numofSites,0);
        std::vector<double> CommunityMag(_numofSites, 1); //give all sites equal mag to start, this value is [0,1]
        std::cerr<<"all output vectors created and initialized."<<std::endl;
        for(int step=0; step<3600*_sampleRate; step++){
            for(int j=0; j<_numofSites; j++){ //sitesWeighted Lat/Lon values are determined based on all previous sites mag output value.
                CommunityLat += _siteData->at(j*2)*CommunityMag[j];
                CommunityLon += _siteData->at(j*2+1)*CommunityMag[j];
            }
            CommunityLat = CommunityLat/_numofSites;
            CommunityLon = CommunityLon/_numofSites;

            for(int j=0; j<_numofSites; j++){ // each site is run independently of others, but shares an output from the previous step
                double latSite = _siteData->at(j*2);
                double lonSite = _siteData->at(j*2+1);
                double avgLatGQuake = globalQuakes->at(0);
                double avgLonGQuake = globalQuakes->at(1);
                double GQuakeAvgMag = globalQuakes->at(3);
                double GQuakeAvgdist = distCalc(latSite, lonSite, avgLatGQuake, avgLonGQuake);
                double GQuakeAvgBearing = bearingCalc(latSite, lonSite, avgLatGQuake, avgLonGQuake);
                double CommunityDist = distCalc(latSite, lonSite, CommunityLat, CommunityLon);
                double CommunityBearing = bearingCalc(latSite, lonSite, CommunityLat, CommunityLon);
                std::vector<double> input;
                std::vector<double> hidden, outputs, mem, memGateOut, memGateIn, memGateForget;
                //replace these with real connections, num of inputs, and num of hidden & memory neurons (mem neurons probably accurate)
                input.resize(_hostParams.array[2], 0); // number of inputs is 9.
                hidden.resize(_hostParams.array[10], 0); // for practice sake, lets say each input has its own neuron (might be true!)
                mem.resize(_hostParams.array[11], 0); // stores the input if gate is high
                memGateOut.resize(_hostParams.array[11], 0); //connects to the input layer and the memN associated with input, if 1 it sends up stream and deletes, if low it keeps.
                memGateIn.resize(_hostParams.array[11], 0);
                memGateForget.resize(_hostParams.array[11], 0);
                outputs.resize(_hostParams.array[12], 0); /* 3 outputs, 1 with an hour in the future when the earthquake will hit,
                    1 with the porbability of that earthquake happening (between [0,1]) and 1 with the sites magnitude (for community feedback) */
                int n =0;
                int startOfInput = 0;
                int startOfHidden = startOfInput +_hostParams.array[2];
                int startOfMem = startOfHidden + _hostParams.array[10];
                int startOfMemGateIn = startOfMem + _hostParams.array[11];
                int startOfMemGateOut = startOfMemGateIn + _hostParams.array[11];
                int startOfMemGateForget = startOfMemGateOut + _hostParams.array[11];
                int startOfOutput = startOfMemGateForget + _hostParams.array[11];
                for(int k=0; k<3; k++){
                    input[k] = normalize((double)(data->at(3600*_sampleRate*j*3 + k*(3600*_sampleRate)+step)), meanCh[k], stdCh[k]);

                }
                input[3] = shift(GQuakeAvgdist, 40075.1, 0);
                input[4] = shift(GQuakeAvgBearing, 360, 0);
                input[5] = shift(GQuakeAvgMag, 9.5, 0);
                input[6] = shift(Kp, 10, 0);
                input[7] = shift(CommunityDist,40075.1/2, 0);
                input[8] = shift(CommunityBearing, 360, 0);
                //lets reset all neuron values for this new timestep (except memory neurons)
                for(int gate=0; gate<_hostParams.array[11]; gate++){
                    memGateIn.at(gate) = 0;
                    memGateOut.at(gate) = 0;
                    memGateForget.at(gate) = 0;
                }
                for(int hid=0; hid<_hostParams.array[10]; hid++){
                    hidden[hid] = 0;
                }
                for(int out=0; out<_hostParams.array[12]; out++){
                    outputs[out] = 0;
                }
                //now that everything that should be zeroed is zeroed, lets start the network.
                //mem gates & LSTM nodes --
                for(int gate = 0; gate<_hostParams.array[11]; gate++){//calculate memory gate node values, you can connect inputs & hidden neurons to them.
                    for(connectPairMatrix::iterator it = _connect->begin(); it!= _connect->end(); ++it){//for memGateIn
                        if(it->second == gate+startOfMemGateIn && it->first < startOfHidden){ //for inputs
                            memGateIn.at(gate) += input[it->first-startOfInput]*_best[n++]; // memGateIn vect starts at 0
                        }
                        else if(it->second == gate+startOfMemGateIn && it->first >startOfHidden && it->first < startOfMem){//for hidden neurons
                            memGateIn.at(gate) += hidden[it->first-startOfHidden]*_best[n++];
                        }
                    }
                    for(connectPairMatrix::iterator it = _connect->begin(); it!= _connect->end(); ++it){//for memGateOut
                        if(it->second == gate+startOfMemGateOut && it->first < startOfHidden){//for inputs
                            memGateOut.at(gate) += input[it->first-startOfInput]*_best[n++];
                        }
                        else if(it->second == gate+startOfMemGateOut && it->first >startOfHidden && it->first <startOfMem){//for hidden neurons
                            memGateOut.at(gate) += hidden[it->first-startOfHidden]*_best[n++];
                        }
                    }
                    for(connectPairMatrix::iterator it = _connect->begin(); it!= _connect->end(); ++it){//for  memGateForget
                        if(it->second == gate+startOfMemGateForget && it->first < startOfHidden){//for inputs
                            memGateForget.at(gate) += input[it->first - startOfInput]*_best[n++];
                        }
                        else if(it->second == gate+startOfMemGateForget && it->first >startOfHidden && it->first <startOfMem){//for hidden neurons
                            memGateForget.at(gate) += hidden[it->first-startOfHidden]*_best[n++];
                        }
                    }
                    memGateIn.at(gate) = ActFunc(memGateIn.at(gate));
                    memGateOut.at(gate) = ActFunc(memGateOut.at(gate));
                    memGateForget.at(gate) = ActFunc(memGateForget.at(gate));

                }
                //since we calculated the values for memGateIn and memGateOut, and MemGateForget..
                for (int gate = 0; gate<_hostParams.array[11]; gate++){ // if memGateIn is greater than 0.3, then let mem = the sum inputs attached to memGateIn
                    if(memGateIn.at(gate) > 0.5){ //gate -startOfMemGateIn = [0, num of mem neurons]
                        for(connectPairMatrix::iterator it = _connect->begin(); it!= _connect->end(); ++it){
                            if(it->second == gate+startOfMemGateIn && it->first < gate+startOfHidden){//only pass inputs
                                mem.at(gate) += input[it->first-startOfInput]; // no weights attached, but the old value stored here is not removed.
                            }
                        }
                    }
                    if(memGateForget.at(gate) > 0.5){// if memGateForget is greater than 0.5, then tell mem to forget
                        mem.at(gate) = 0;
                    }
                    //if memGateForget fires, then memGateOut will output nothing.
                    if(memGateOut.at(gate) > 0.5){//if memGateOut is greater than 0.3, let the nodes mem is connected to recieve mem
                        for(connectPairMatrix::iterator it = _connect->begin(); it!= _connect->end(); ++it){
                            if(it->first == gate+startOfMem){// since mem node: memIn node : memOut node = 1:1:1, we can do this.
                                hidden[it->second-startOfHidden] += mem.at(gate);
                            }
                        }
                    }
                }

                // hidden neuron nodes --
                for(int hid=0; hid<_hostParams.array[10]; hid++){ // for all hidden neurons at layer 1, lets sum the inputs, the memory values were already added.
                    for(connectPairMatrix::iterator it = _connect->begin(); it!= _connect->end(); ++it){ // Add the inputs to the hidden neurons
                        if(it->second == hid+startOfHidden && it->first < startOfHidden){ // if an input connects with this hidden neuron
                            hidden[hid] += input[it->first]*_best[n++];
                        }
                    }
                    for(connectPairMatrix::iterator it = _connect->begin(); it!= _connect->end(); ++it){//add other hidden neuron inputs to each hidden neuron (if applicable)
                        if(it->second == hid+startOfHidden && it->first < startOfMem && it->first > startOfHidden){
                            hidden[hid] += hidden[it->first-startOfHidden]*_best[n++];
                        }
                    }
                    hidden[hid] += 1*_best[n++]; // add bias
                    hidden[hid] = ActFunc(hidden[hid]); // then squash it.
                }
                //output nodes --

                for(int out =0; out<_hostParams.array[12]; out++){// add hidden neurons to the output nodes
                    for(connectPairMatrix::iterator it = _connect->begin(); it!= _connect->end(); ++it){
                        if(it->second == out+startOfOutput){
                            outputs[out] += hidden[it->first-startOfHidden]*_best[n++];
                        }
                    }
                    outputs[out] += 1*_best[n++]; // add bias
                    outputs[out] = ActFunc(outputs[out]);// then squash it.
                }


                When[j] += outputs[0]*((2160-hour)-hour)+2160-hour; //return when back to an integer value (adjust to fit within boundaries)
                HowCertain[j] += outputs[1];
                CommunityMag[j] =  outputs[2]; // set the next sets communityMag = output #3.
            }
        }
        for(int j=0; j<_numofSites; j++){ // each site has its own when and howcertain vector
            When[j] = When[j]/3600*_sampleRate;
            HowCertain[j] = HowCertain[j]/3600*_sampleRate;
            std::cerr<<"When for site:"<<j<<" is: "<<When[j]<<std::endl;
        }
        //all done, lets output the return matrix.
        //since right now were using a point value for when & how certain (only one output per site),
        //we're going to approximate using a normal distribution around when with a sigma of howCertain, over the whole array from T=currentHour [T, 2160]
        for(int h=hour; h<2160; h++){
            for(int j=0; j<_numofSites; j++){
                ret->at(h*_numofSites+j)= 1/(1/HowCertain[j]*sqrt(2*M_PI))*exp(-pow(h-When[j], 2)/(2*pow(1/HowCertain[j], 2))); // normal distribution with a mu of When and a sigma of 1/HowCertain
            }
        }
    }
}
