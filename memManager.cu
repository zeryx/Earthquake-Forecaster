#include "hip/hip_runtime.h"
#include "memManager.h"
#include "getsys.h"
#include "datediff.h"
#include <iostream>
#include <string>
#include <sstream>
#include <vector>
#include <fstream>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/swap.h>
#include <thrust/system_error.h>
#include "tinyxml2.h"
#include <thrust/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>

//xml error message handling
#ifndef XMLCheckResult
#define XMLCheckResult(a_eResult) if (a_eResult != tinyxml2::XML_SUCCESS) { printf("Error: %i\n", a_eResult);  exit(a_eResult); }
#endif

MemManager::MemManager(){}


dataArray<double> MemManager::genetics(){
    return convertToKernel(_DGenetics);
}

dataArray<int> MemManager::input(){
    return convertToKernel(_DInput);
}

dataArray<double> MemManager::training(){
    return convertToKernel(_DTraining);
}

dataArray<double> MemManager::sites(){
    return convertToKernel(_DSites);
}

dataArray<double> MemManager::kpIndex(){
    return convertToKernel(_DKpIndex);
}



bool MemManager::memoryAlloc(std::map<const std::string, float> pHostRam,
                             std::map<const std::string, float> pDeviceRam,
                             int individualLength, float pMaxHost, float pMaxDevice){//allocates memory for genetis & input vectors
    long long hostMem = GetHostRamInBytes()*pMaxHost; //make a host memory container, this is the max
    long long deviceMem = GetDeviceRamInBytes()*pMaxDevice; //dito for gpu
    int dub = 8, integer = 4;
    _hostGeneticsAlloc = hostMem*pHostRam.at("genetics")/dub; //since these are doubles, divide bytes by 8
    _hostInputAlloc = hostMem*pHostRam.at("input & training")/integer; // their either floats or ints, same amount of bytes.
    _deviceGeneticsAlloc = deviceMem*pDeviceRam.at("genetics")/dub;
    _deviceInputAlloc = deviceMem*pDeviceRam.at("input & training")/(integer*2); // a half of the alloced input is goign to XML data
    //round the genetics allocators to whole individuals.
    _hostGeneticsAlloc = (_hostGeneticsAlloc/individualLength)*individualLength;
    _deviceGeneticsAlloc = (_deviceGeneticsAlloc/individualLength)*individualLength;
    std::cout<<"allocating..."<<std::endl;
    //initialize all large vectors (everything not from an xml file)
    try{
        this->_HGenetics.setMax(_hostGeneticsAlloc);
        this->_HInput.setMax(_hostInputAlloc);
    }
    catch(thrust::system_error &e){
        std::cerr<<"Error resizing vector Element: "<<e.what()<<std::endl;
        exit(1);
    }
    catch(std::bad_alloc &e){
        std::cerr<<"Ran out of space due to : "<<"host"<<std::endl;
        std::cerr<<e.what()<<std::endl;
        std::cout<<GetHostRamInBytes()<<std::endl;
        exit(1);
    }
    try{
        this->_DGenetics.resize(_deviceGeneticsAlloc);
        this->_DInput.resize(_deviceInputAlloc);
    }
    catch(thrust::system_error &e){
        std::cerr<<"Error resizing vector Element: "<<e.what()<<std::endl;
        exit(1);
    }
    catch(std::bad_alloc &e){
        std::cerr<<"Ran out of space due to : "<<"device"<<std::endl;
        std::cerr<<e.what()<<std::endl;
        std::cout<<GetDeviceRamInBytes()<<std::endl;
        exit(1);
    }
    std::cout<<"allocated."<<std::endl;
    return true;
}
bool MemManager::geneticsBufferSwap(dataArray<double> *dGen){
    return false;
}

bool MemManager::GeneticsPushToHost(dataArray<double> *dGen){
    long long dGenLength = dGen->_size;
    long long currpos = dGenLength + _HGenetics._itr;
    std::cout<<"length of device vector: "<<dGenLength<<std::endl;
    std::cout<<"host vector max length: "<<_HGenetics._maxLen<<std::endl;
    if(currpos*2 <= _HGenetics._maxLen && currpos < _HGenetics._maxLen){ //if _HGenetics can take 2 more at the current size, keep going
        thrust::copy(dGen->_array, dGen->_array + dGenLength, _HGenetics._hVect.begin()+_HGenetics._itr);
        _HGenetics._itr = currpos; //set the iterator  to the new position.
        std::cout<<"#1"<<std::endl;
        return true;
    }
    else if(currpos*2 > _HGenetics._maxLen && currpos < _HGenetics._maxLen){//if _HGenetics can only take 1 or exactly 2 at current size, resize dgen to fit
        thrust::copy(dGen->_array, dGen->_array + dGenLength, _HGenetics._hVect.begin()+_HGenetics._itr);
        _DGenetics.resize(_HGenetics._maxLen - currpos);// the device_vector for genetics was resized to fit the remaining host mem container.
        std::cout<<"resized genetics to: "<<_HGenetics._maxLen - _HGenetics._itr<<std::endl;
        dGen->_size = _DGenetics.size();
        _HGenetics._itr = currpos;
        std::cout<<"#2"<<std::endl;
        return true;
    }
    else if(currpos == _HGenetics._maxLen && dGenLength !=0){//if the _HGenetics vector is full, tell the GPU
        thrust::copy(dGen->_array, dGen->_array + dGenLength, _HGenetics._hVect.begin()+_HGenetics._itr);
        _HGenetics._itr = 0;
        _DGenetics.resize(_deviceGeneticsAlloc);
        dGen->_size =_DGenetics.size();
        std::cout<<"#3"<<std::endl;
        return false;
    }
    else if(currpos > _HGenetics._maxLen){
        std::cout<<"#4"<<std::endl;
        return false;
    }

    return false;
}


bool MemManager::InputRefresh(dataArray<int> *input){
    return false;
}
void MemManager::setPath(std::string pathToData){
    this->_dataDirectory = pathToData;
}

void MemManager::setTest(int testNum){
    _testDirectory = _dataDirectory;
    std::ostringstream oss;
    oss << "/" << testNum;
    _testDirectory.append(oss.str());

}

void MemManager::importSitesData(){
    tinyxml2::XMLDocument doc;
    tinyxml2::XMLError eResult;
    _DSites.clear();
    _DSites.shrink_to_fit();
    std::string siteInfoStr = this->_testDirectory;
    siteInfoStr.append("/SiteInfo.xml");
    eResult = doc.LoadFile(siteInfoStr.c_str());
    XMLCheckResult(eResult);
    tinyxml2::XMLNode * pRoot = doc.FirstChild();
    if(pRoot == NULL){
        std::cout<<"file read error"<<std::endl;
        exit(tinyxml2::XML_ERROR_FILE_READ_ERROR);
    }
    tinyxml2::XMLElement * pElement = pRoot->NextSiblingElement("Sites");
    if(pElement == NULL){
        std::cout<<"error parsing element"<<std::endl;
        exit(tinyxml2::XML_ERROR_PARSING_ELEMENT);
    }
    tinyxml2::XMLElement *SitesList = pRoot->NextSiblingElement("Site");

    while(SitesList != NULL){
        int sampleRate, siteNumber;
        float longitude, latitude;
        eResult = SitesList->QueryIntAttribute("sample_rate", &sampleRate);
        XMLCheckResult(eResult);
        eResult = SitesList->QueryFloatAttribute("latitude", &latitude);
        XMLCheckResult(eResult);
        eResult = SitesList->QueryFloatAttribute("longitude", &longitude);
        XMLCheckResult(eResult);
        eResult = SitesList->QueryIntText(&siteNumber);
        XMLCheckResult(eResult);
        _DSites.push_back(siteNumber);
        _DSites.push_back(sampleRate);
        _DSites.push_back(latitude);
        _DSites.push_back(longitude);
        SitesList = SitesList->NextSiblingElement("Site");
    }
}

void MemManager::importKpData(){
    tinyxml2::XMLDocument doc;
    tinyxml2::XMLError eResult;
    _DKpIndex.clear();
    _DKpIndex.shrink_to_fit();
    std::string KpStr = this->_testDirectory;
    KpStr.append("/Kp.xml");
    eResult = doc.LoadFile(KpStr.c_str());
    XMLCheckResult(eResult);

    tinyxml2::XMLNode *pRoot = doc.FirstChild();
    if(pRoot == NULL){
        std::cout<<"file read error"<<std::endl;
        exit(tinyxml2::XML_ERROR_FILE_READ_ERROR);
    }
    tinyxml2::XMLElement * pElement = pRoot->NextSiblingElement("Kp");
    if(pElement == NULL){
        std::cout<<"error parsing element"<<std::endl;
        exit(tinyxml2::XML_ERROR_PARSING_ELEMENT);
    }
    tinyxml2::XMLElement * KpList = pElement->FirstChildElement("Kp_hr");
    while(KpList != NULL){
        int time;
        float magnitude;
        eResult = KpList->QueryIntAttribute("secs", &time);
        XMLCheckResult(eResult);
        time = time/3600;
        eResult = KpList->QueryFloatText(&magnitude);
        XMLCheckResult(eResult);
        _DKpIndex.push_back(time);//in hours
        _DKpIndex.push_back(magnitude);
        KpList = KpList->NextSiblingElement("Kp_hr");
    }
}

void MemManager::importGQuakes(){
    tinyxml2::XMLDocument doc;
    tinyxml2::XMLError eResult;
    _DGQuakes.clear();
    _DGQuakes.shrink_to_fit();
    std::string QuakesStr = this->_testDirectory;
    QuakesStr.append("/Quakes.xml");
    eResult = doc.LoadFile(QuakesStr.c_str());
    XMLCheckResult(eResult);
    tinyxml2::XMLNode *pRoot = doc.FirstChild();
    if(pRoot == NULL){
        std::cout<<"file read error"<<std::endl;
        exit(tinyxml2::XML_ERROR_FILE_READ_ERROR);
    }
    tinyxml2::XMLElement * pElement = pRoot->NextSiblingElement("Quakes");
    if(pElement == NULL){
        std::cout<<"error parsing element"<<std::endl;
        exit(tinyxml2::XML_ERROR_PARSING_ELEMENT);
    }
    tinyxml2::XMLElement * quakeList = pElement->FirstChildElement("Quake");
    std::vector<float> tmp;//don't store this yet, we need to reduce resolution for the network.
    int numQuakes=0;
    while(quakeList != NULL){
        int seconds;
        float latitude, longitude, magnitude, depth, hours;
        eResult = quakeList->QueryIntAttribute("secs", &seconds);
        XMLCheckResult(eResult);
        hours = seconds/3600;
        eResult = quakeList->QueryFloatAttribute("latitude", &latitude);
        XMLCheckResult(eResult);
        eResult = quakeList->QueryFloatAttribute("longitude", &longitude);
        XMLCheckResult(eResult);
        eResult = quakeList->QueryFloatAttribute("magnitude", &magnitude);
        XMLCheckResult(eResult);
        eResult = quakeList->QueryFloatAttribute("depth", &depth);
        tmp.push_back(hours);
        tmp.push_back(latitude);
        tmp.push_back(longitude);
        tmp.push_back(magnitude);
        tmp.push_back(depth);
        numQuakes++;
        quakeList = quakeList->NextSiblingElement("Quake");
    }
    _DGQuakes.resize(2160*5, 0.0);
    for(int hour=0; hour<2610; hour++){
        int accVal=0;
        _DGQuakes[hour*5] = hour+1;
        for (int i=0; i<numQuakes; i++){
            if(tmp[i*5]>= hour && tmp[i*5]< hour+1){
                for(int k=1; k<5; k++){//don't start at 0 because 0 is time.
                    _DGQuakes[hour*5+k] += tmp[i*5 +k];
                    accVal++;
                }
            }
        }
        for(int k=1; k<5; k++){
            if(_DGQuakes[hour*5+k] !=0)
                _DGQuakes[hour*5+k] = _DGQuakes[hour*5+k]/accVal; // push the hourly average into _DGQuakes for all parameters.
        }
        for(int k=0; k<5; k++){
            std::cout<<_DGQuakes[hour*5+k]<<std::endl;
        }
    }
}

void MemManager::importTrainingData(){ // this is only called once for the entire life of the program, also uses CSV so it's done with fopen
    std::string answerStr = this->_dataDirectory;
    answerStr.append("/gtf.csv");
    std::ifstream answerfile(answerStr.c_str());
    std::string line;
    std::getline(answerfile, line);
    float numOfTests;
    std::istringstream(line) >> numOfTests;
    std::cout<<"pushing back.."<<std::endl;
    _DTraining.push_back(numOfTests);
    while(std::getline(answerfile, line)){
        std::vector<std::string> token;
        std::string item;
        std::stringstream ss(line);
        while(std::getline(ss,  item, ',')){
            token.push_back(item);
        }
        _DTraining.push_back(std::atoi(token[0].c_str())); // setID
        std::string startTime = token[1];
        std::string EqTime = token[2];
        _DTraining.push_back(timeDifferenceCalculation(startTime, EqTime));
        _DTraining.push_back(std::atoi(token[3].c_str())); // magnitude
        _DTraining.push_back(std::atoi(token[4].c_str())); // latitude
        _DTraining.push_back(std::atoi(token[5].c_str())); // longitude
        _DTraining.push_back(std::atoi(token[6].c_str())); // siteNumber
        _DTraining.push_back(std::atoi(token[7].c_str())); // distance to quake
    }
}
