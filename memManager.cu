#include "hip/hip_runtime.h"
#include "memManager.h"
#include "getsys.h"
#include "datediff.h"
#include <iostream>
#include <string>
#include <sstream>
#include <vector>
#include <fstream>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/swap.h>
#include <thrust/system_error.h>
#include "tinyxml2.h"
#include <thrust/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>

//xml error message handling
#ifndef XMLCheckResult
#define XMLCheckResult(a_eResult) if (a_eResult != tinyxml2::XML_SUCCESS) { printf("Error: %i\n", a_eResult);  exit(a_eResult); }
#endif

MemManager::MemManager(){}


dataArray<double> MemManager::genetics(){
    return convertToKernel(_DGenetics);
}

dataArray<int> MemManager::input(){
    return convertToKernel(_DInput);
}

dataArray<Answers> MemManager::training(){
    return convertToKernel(_DTraining);
}

dataArray<SiteInfo> MemManager::sites(){
    return convertToKernel(_DSites);
}

dataArray<Kp> MemManager::kpIndex(){
    return convertToKernel(_DKpIndex);
}



bool MemManager::memoryAlloc(std::map<const std::string, float> pHostRam,
                             std::map<const std::string, float> pDeviceRam,
                             int individualLength, float pMaxHost, float pMaxDevice){//allocates memory for genetis & input vectors
    long long hostMem = GetHostRamInBytes()*pMaxHost; //make a host memory container, this is the max
    long long deviceMem = GetDeviceRamInBytes()*pMaxDevice; //dito for gpu
    int dub = 8, integer = 4;
    _hostGeneticsAlloc = hostMem*pHostRam.at("genetics")/dub; //since these are doubles, divide bytes by 8
    _hostInputAlloc = hostMem*pHostRam.at("input & training")/integer; // their either floats or ints, same amount of bytes.
    _deviceGeneticsAlloc = deviceMem*pDeviceRam.at("genetics")/dub;
    _deviceInputAlloc = deviceMem*pDeviceRam.at("input & training")/(integer*2); // a half of the alloced input is goign to XML data
    //round the genetics allocators to whole individuals.
    _hostGeneticsAlloc = (_hostGeneticsAlloc/individualLength)*individualLength;
    _deviceGeneticsAlloc = (_deviceGeneticsAlloc/individualLength)*individualLength;
    std::cout<<"allocating..."<<std::endl;
    //initialize all large vectors (everything not from an xml file)
    try{
        this->_HGenetics.setMax(_hostGeneticsAlloc);
        this->_HInput.setMax(_hostInputAlloc);
    }
    catch(thrust::system_error &e){
        std::cerr<<"Error resizing vector Element: "<<e.what()<<std::endl;
        exit(1);
    }
    catch(std::bad_alloc &e){
        std::cerr<<"Ran out of space due to : "<<"host"<<std::endl;
        std::cerr<<e.what()<<std::endl;
        std::cout<<GetHostRamInBytes()<<std::endl;
        exit(1);
    }
    try{
        this->_DGenetics.resize(_deviceGeneticsAlloc);
        this->_DInput.resize(_deviceInputAlloc);
    }
    catch(thrust::system_error &e){
        std::cerr<<"Error resizing vector Element: "<<e.what()<<std::endl;
        exit(1);
    }
    catch(std::bad_alloc &e){
        std::cerr<<"Ran out of space due to : "<<"device"<<std::endl;
        std::cerr<<e.what()<<std::endl;
        std::cout<<GetDeviceRamInBytes()<<std::endl;
        exit(1);
    }
    std::cout<<"allocated."<<std::endl;
    return true;
}
bool MemManager::geneticsBufferSwap(dataArray<double> *dGen){
    return false;
}

bool MemManager::GeneticsPushToHost(dataArray<double> *dGen){
    long long dGenLength = dGen->_size;
    long long currpos = dGenLength + _HGenetics._itr;
    std::cout<<"length of device vector: "<<dGenLength<<std::endl;
    std::cout<<"host vector max length: "<<_HGenetics._maxLen<<std::endl;
    if(currpos*2 <= _HGenetics._maxLen && currpos < _HGenetics._maxLen){ //if _HGenetics can take 2 more at the current size, keep going
        thrust::copy(dGen->_array, dGen->_array + dGenLength, _HGenetics._hVect.begin()+_HGenetics._itr);
        _HGenetics._itr = currpos; //set the iterator  to the new position.
        std::cout<<"#1"<<std::endl;
        return true;
    }
    else if(currpos*2 > _HGenetics._maxLen && currpos < _HGenetics._maxLen){//if _HGenetics can only take 1 or exactly 2 at current size, resize dgen to fit
        thrust::copy(dGen->_array, dGen->_array + dGenLength, _HGenetics._hVect.begin()+_HGenetics._itr);
        _DGenetics.resize(_HGenetics._maxLen - currpos);// the device_vector for genetics was resized to fit the remaining host mem container.
        std::cout<<"resized genetics to: "<<_HGenetics._maxLen - _HGenetics._itr<<std::endl;
        dGen->_size = _DGenetics.size();
        _HGenetics._itr = currpos;
        std::cout<<"#2"<<std::endl;
        return true;
    }
    else if(currpos == _HGenetics._maxLen && dGenLength !=0){//if the _HGenetics vector is full, tell the GPU
        thrust::copy(dGen->_array, dGen->_array + dGenLength, _HGenetics._hVect.begin()+_HGenetics._itr);
        _HGenetics._itr = 0;
        _DGenetics.resize(_deviceGeneticsAlloc);
        dGen->_size =_DGenetics.size();
        std::cout<<"#3"<<std::endl;
        return false;
    }
    else if(currpos > _HGenetics._maxLen){
        std::cout<<"#4"<<std::endl;
        return false;
    }

    return false;
}


bool MemManager::InputRefresh(dataArray<int> *input){
    return false;
}
void MemManager::setPath(std::string pathToData){
    this->_dataDirectory = pathToData;
}

void MemManager::setTest(int testNum){
    this->_testDirectory = _dataDirectory.append("/"+testNum);
}

void MemManager::importSitesData(){
    tinyxml2::XMLDocument doc;
    tinyxml2::XMLError eResult;
        _DSites.clear();
        _DSites.shrink_to_fit();
    std::string siteInfoStr = this->_testDirectory.append("/SiteInfo.xml");
    doc.LoadFile(siteInfoStr.c_str());
    tinyxml2::XMLNode * pRoot = doc.FirstChild();
    if(pRoot == NULL) exit(tinyxml2::XML_ERROR_FILE_READ_ERROR);
    tinyxml2::XMLElement * pElement = pRoot->NextSiblingElement("Sites");
    if(pElement == NULL) exit(tinyxml2::XML_ERROR_PARSING_ELEMENT);

    tinyxml2::XMLElement *SitesList = pRoot->NextSiblingElement("Site");

    while(SitesList != NULL){
        int sampleRate, siteNumber;
        float longitude, latitude;
        eResult = SitesList->QueryIntAttribute("sample_rate", &sampleRate);
        XMLCheckResult(eResult);
        eResult = SitesList->QueryFloatAttribute("latitude", &latitude);
        XMLCheckResult(eResult);
        eResult = SitesList->QueryFloatAttribute("longitude", &longitude);
        XMLCheckResult(eResult);
        eResult = SitesList->QueryIntText(&siteNumber);
        XMLCheckResult(eResult);
        SiteInfo tmp;
        tmp.siteNumber = siteNumber;
        tmp.sampleRate = sampleRate;
        tmp.latitude = latitude;
        tmp.longitude = longitude;
        _DSites.push_back(tmp);
        SitesList = SitesList->NextSiblingElement("Site");
    }
}

void MemManager::importKpData(){
    tinyxml2::XMLDocument doc;
    tinyxml2::XMLError eResult;
    _DKpIndex.clear();
    _DKpIndex.shrink_to_fit();
    std::string KpStr = this->_testDirectory.append("/Kp.xml");
    doc.LoadFile(KpStr.c_str());
    tinyxml2::XMLNode *pRoot = doc.FirstChild();
    if(pRoot == NULL) exit(tinyxml2::XML_ERROR_FILE_READ_ERROR);
    tinyxml2::XMLElement * pElement = pRoot->NextSiblingElement("Kp");
    if(pElement == NULL) exit(tinyxml2::XML_ERROR_PARSING_ELEMENT);
    tinyxml2::XMLElement * KpList = pElement->FirstChildElement("Kp_hr");
    while(KpList != NULL){
        int seconds;
        float magnitude;
        eResult = KpList->QueryIntAttribute("secs", &seconds);
        XMLCheckResult(eResult);
        eResult = KpList->QueryFloatText(&magnitude);
        XMLCheckResult(eResult);
        Kp tmp;
        tmp.seconds = seconds;
        tmp.magnitude = magnitude;
        _DKpIndex.push_back(tmp);
        KpList = KpList->NextSiblingElement("Kp_hr");
    }
}

void MemManager::importGQuakes(){
    tinyxml2::XMLDocument doc;
    tinyxml2::XMLError eResult;
    _DGQuakes.clear();
    _DGQuakes.shrink_to_fit();
    std::string QuakesStr = this->_testDirectory.append("/Quakes.xml");
    doc.LoadFile(QuakesStr.c_str());
    tinyxml2::XMLNode *pRoot = doc.FirstChild();
    if(pRoot == NULL) exit(tinyxml2::XML_ERROR_FILE_READ_ERROR);
    tinyxml2::XMLElement * pElement = pRoot->NextSiblingElement("Quakes");
    if(pElement == NULL) exit(tinyxml2::XML_ERROR_PARSING_ELEMENT);
    tinyxml2::XMLElement * quakeList = pElement->FirstChildElement("Quake");
    while(quakeList != NULL){
        int seconds;
        float latitude, longitude, magnitude, depth;
        eResult = quakeList->QueryIntAttribute("secs", &seconds);
        XMLCheckResult(eResult);
        eResult = quakeList->QueryFloatAttribute("latitude", &latitude);
        XMLCheckResult(eResult);
        eResult = quakeList->QueryFloatAttribute("longitude", &longitude);
        XMLCheckResult(eResult);
        eResult = quakeList->QueryFloatAttribute("magnitude", &magnitude);
        XMLCheckResult(eResult);
        eResult = quakeList->QueryFloatAttribute("depth", &depth);
        GQuakes tmp;
        tmp.seconds = seconds;
        tmp.latitude = latitude;
        tmp.longitude = longitude;
        tmp.magnitude = magnitude;
        tmp.depth = depth;
        _DGQuakes.push_back(tmp);
        quakeList = quakeList->NextSiblingElement("Quake");
    }
}

void MemManager::importTrainingData(){ // this is only called once for the entire life of the program, also uses CSV so it's done with fopen
    std::string answerStr = this->_dataDirectory.append("/gtf.csv");
    std::ifstream answerfile(answerStr.c_str());
    std::string line;
    std::getline(answerfile, line);
    int numOfTests;
    std::istringstream(line) >> numOfTests;
    while(std::getline(answerfile, line)){
        std::vector<std::string> token;
        std::string item;
        std::stringstream ss(line);
        while(std::getline(ss,  item, ',')){
            token.push_back(item);
        }
        Answers tmp;
        std::istringstream(token[0]) >> tmp.setID;
        std::string startTime = token[1];
        std::string EqTime = token[2];
        tmp.hrOfQuake = timeDifferenceCalculation(startTime, EqTime);
        std::istringstream(token[3]) >> tmp.magnitude;
        std::istringstream(token[4]) >> tmp.latitude;
        std::istringstream(token[5]) >> tmp.longitude;
        std::istringstream(token[6]) >> tmp.siteNum;
        std::istringstream(token[7]) >> tmp.distToQuake;
    }

}
