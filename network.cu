#include "hip/hip_runtime.h"
﻿#include "network.h"
#include <fstream>
#include <thrust/device_vector.h>
#include <thrust/random.h>
#include <thrust/system_error.h>
#include <utility>
#include <vector>
#include <ctime>
#include <thrust/host_vector.h>

//macros
//cuda error message handling
#define CUDA_SAFE_CALL(call)                                          \
    do {                                                                  \
    hipError_t err = call;                                           \
    if (hipSuccess != err) {                                         \
    fprintf (stderr, "Cuda error in file '%s' in line %i : %s.\n",\
    __FILE__, __LINE__, hipGetErrorString(err) );                  \
    exit(EXIT_FAILURE);                                             \
    }                                                                 \
    } while (0)

//neural functions
__host__ __device__ inline double sind(double x)
{
    double ret= sin(x * M_PI / 180);;
    return ret;
}

__host__ __device__ inline double cosd(double x)
{
    return cos(x * M_PI / 180);
}
__host__ __device__ inline double distCalc(double lat1, double lon1, double lat2, double lon2)
{
    double earthRad = 6371.01;
    double deltalon = abs(lon1 - lon2);
    if(deltalon > 180)
        deltalon = 360 - deltalon;
    double ret = earthRad * atan2( sqrt( pow( cosd(lat1) * sind(deltalon), 2) +
                                         pow( cosd(lat2) * sind(lat1) - sind(lat2) * cosd(lat1) * cosd(deltalon), 2) ),
                                   sind(lat2) * sind(lat1) + cosd(lat2) * cosd(lat1) * cosd(deltalon));
    return ret;
}

__host__ __device__ inline double bearingCalc(double lat1, double lon1, double lat2, double lon2)
{
    double dLon = (lon2 - lon1);

    double y = sin(dLon) * cos(lat2);
    double x = cos(lat1) * sin(lat2) - sin(lat1) * cos(lat2) * cos(dLon);

    double brng = atan2(y, x);

    brng = brng*M_PI/180;
    brng += 360;
    while(brng>= 360)
        brng -= 360;
    brng = 360 - brng;

    return brng;
}

__host__ __device__ inline double ActFunc(double x)
{
    double ret = 1+1/exp(-x);
    return ret;
}
__host__ __device__ inline double normalize(double x, double mean, double stdev)
{
    double ret = (abs(x-mean))/stdev*2;
    return ret;
}

__host__ __device__ inline double shift(double x, double max, double min)
{
    double ret = (x-min)/(max-min);
    return ret;
}

__global__ void genWeights( dataArray<double> ref, long in, dataArray<int> params)
{
    long idx = blockIdx.x * blockDim.x + threadIdx.x;
    long ind = idx*params.array[7];
    thrust::minstd_rand0 randEng;
    randEng.seed(idx);
    long seed = idx+ref.size*in;
    for(int i=0; i<params.array[2]; i++){
        thrust::uniform_real_distribution<double> uniDist(0,1);
        randEng.discard(seed+1);
        ref.array[ind + i] = uniDist(randEng);
    }
}

__global__ void Net(dataArray<double> weights, dataArray<int> params, dataArray<double> globalQuakes,
                    dataArray<int> inputVal, dataArray<double> siteData,
                    dataArray<double> answers, dataArray<thrust::pair<int, int> > connections,
                    double Kp, int sampleRate,int numOfSites, int hour,
                    double meanCh1, double meanCh2, double meanCh3, double stdCh1, double stdCh2, double stdCh3)
{



    int idx = blockIdx.x * blockDim.x + threadIdx.x; // for each thread, calculate a individuals weight.
    int ind = idx*params.array[7];
    typedef thrust::device_ptr<thrust::pair<int, int> > connectPairMatrix;
    double CommunityLat = 0;
    double CommunityLon = 0;
    double *When = (double*)malloc(numOfSites*sizeof(double));
    double *HowCertain = (double*)malloc(numOfSites*sizeof(double));
    double *CommunityMag = (double*)malloc(numOfSites*sizeof(double)); //give all sites equal mag to start, this value is [0,1]

    for(int step=0; step<3600*sampleRate; step++){

        for(int j=0; j<sampleRate; j++){//sitesWeighted Lat/Lon values are determined based on all previous sites mag output value.
            CommunityLat += siteData.array[j*2]*CommunityMag[j];
            CommunityLon += siteData.array[j*2+1]*CommunityMag[j];
        }
        CommunityLat = CommunityLat/numOfSites;
        CommunityLon = CommunityLon/numOfSites;

        for(int j=0; j<numOfSites; j++){ //each site is run independently of others, but shares an output from the previous step
            double latSite = siteData.array[j*2];
            double lonSite = siteData.array[j*2+1];
            double avgLatGQuake = globalQuakes.array[0];
            double avgLonGQuake = globalQuakes.array[1];
            //double avgDepthGQuake = globalQuakes.array[2); don't think I care about depth that much.
            double GQuakeAvgMag = globalQuakes.array[3];
            double GQuakeAvgdist = distCalc(latSite, lonSite, avgLatGQuake, avgLonGQuake);
            double GQuakeAvgBearing = bearingCalc(latSite, lonSite, avgLatGQuake, avgLonGQuake);
            double CommunityDist = distCalc(latSite, lonSite, CommunityLat, CommunityLon);
            double CommunityBearing = bearingCalc(latSite, lonSite, CommunityLat, CommunityLon);
            //replace these with real connections, num of inputs, and num of hidden & memory neurons (mem neurons probably accurate)
            int *input = (int*)malloc(params.array[3]*sizeof(int)); // number of inputs is 9.
            double *hidden = (double*)malloc(params.array[4]*sizeof(double)); // for practice sake, lets say each input has its own neuron (might be true!)
            double *mem = (double*)malloc(params.array[5]*sizeof(double)); // stores the input if gate is high
            double *memGateIn = (double*)malloc(params.array[5]*sizeof(double)); //connects to the input layer and the memN associated with input, if 1 it sends up stream and deletes, if low it keeps.
            double *memGateOut = (double*)malloc(params.array[5]*sizeof(double));
            double *memGateForget = (double*)malloc(params.array[5]*sizeof(double));
            double *outputs = (double*)malloc(params.array[6]*sizeof(double)); /* 3 outputs, 1 with an hour in the future when the earthquake will hit,
                1 with the porbability of that earthquake happening (between [0,1]) and 1 with the sites magnitude (for community feedback) */
            int n =0;
            int startOfInput = 0;
            int startOfHidden = startOfInput +params.array[3];
            int startOfMem = startOfHidden + params.array[4];
            int startOfMemGateIn = startOfMem + params.array[5];
            int startOfMemGateOut = startOfMemGateIn + params.array[5];
            int startOfMemGateForget = startOfMemGateOut + params.array[5];
            int startOfOutput = startOfMemGateForget + params.array[5];
            input[0] = normalize(inputVal.array[(3600*sampleRate*j*3 + 1*(3600*sampleRate)+step)], meanCh1, stdCh1);//channel 1
            input[1] = normalize(inputVal.array[(3600*sampleRate*j*3 + 2*(3600*sampleRate)+step)], meanCh2, stdCh2);//channel 2
            input[2] = normalize(inputVal.array[(3600*sampleRate*j*3 + 3*(3600*sampleRate)+step)], meanCh3, stdCh3);//channel 3
            input[3] = shift(GQuakeAvgdist, 40075.1, 0);
            input[4] = shift(GQuakeAvgBearing, 360, 0);
            input[5] = shift(GQuakeAvgMag, 9.5, 0);
            input[6] = shift(Kp, 10, 0);
            input[7] = shift(CommunityDist,40075.1/2, 0);
            input[8] = shift(CommunityBearing, 360, 0);
            //lets reset all neuron values for this new timestep (except memory neurons)
            for(int gate=0; gate<params.array[5]; gate++){
                memGateIn[gate] = 0;
                memGateOut[gate] = 0;
                memGateForget[gate] = 0;
            }
            for(int hid=0; hid<params.array[4]; hid++){
                hidden[hid] = 0;
            }
            for(int out=0; out<params.array[6]; out++){
                outputs[out] = 0;
            }

            //now that everything that should be zeroed is zeroed, lets start the network.
            //mem gates & LSTM nodes --
            for(int gate = 0; gate<params.array[5]; gate++){//calculate memory gate node values, you can connect inputs & hidden neurons to them.
                for(connectPairMatrix it = connections.array; it!= connections.array+connections.size; ++it){//for memGateIn
                    thrust::pair<int, int>itr = static_cast<thrust::pair<int, int> >(*it); // this needs to be created to use the iterator it correctly.
                    if(itr.second == gate+startOfMemGateIn && itr.second < startOfHidden){ //for inputs
                        memGateIn[gate] += input[itr.first-startOfInput]*weights.array[ind + n++]; // memGateIn vect starts at 0
                    }
                    else if(itr.second == gate+startOfMemGateIn && itr.second >startOfHidden && itr.second <startOfMem){//for hidden neurons
                        memGateIn[gate] += hidden[itr.first-startOfHidden]*weights.array[ind + n++];
                    }
                }
                for(connectPairMatrix it = connections.array; it!= connections.array+connections.size; ++it){//for memGateOut
                    thrust::pair<int, int>itr = static_cast<thrust::pair<int, int> >(*it);
                    if(itr.second == gate+startOfMemGateOut && itr.second < startOfHidden){//for inputs
                        memGateOut[gate] += input[itr.first-startOfInput]*weights.array[ind + n++];
                    }
                    else if(itr.second == gate+startOfMemGateOut && itr.second >startOfHidden && itr.second <startOfMem){//for hidden neurons
                        memGateOut[gate] += hidden[itr.first-startOfHidden]*weights.array[ind + n++];
                    }
                }
                for(connectPairMatrix it = connections.array; it!= connections.array+connections.size; ++it){//for  memGateForget
                    thrust::pair<int, int>itr = static_cast<thrust::pair<int, int> >(*it);
                    if(itr.second == gate+startOfMemGateForget && itr.second < startOfHidden){//for inputs
                        memGateForget[gate] += input[itr.first - startOfInput]*weights.array[ind + n++];
                    }
                    else if(itr.second == gate+startOfMemGateForget && itr.second >startOfHidden && itr.second <startOfMem){//for hidden neurons
                        memGateForget[gate] += hidden[itr.first-startOfHidden]*weights.array[ind + n++];
                    }
                }
                memGateIn[gate] = ActFunc(memGateIn[gate]);
                memGateOut[gate] = ActFunc(memGateOut[gate]);
                memGateForget[gate] = ActFunc(memGateForget[gate]);
            }
            //since we calculated the values for memGateIn and memGateOut, and MemGateForget..
            for (int gate = 0; gate<params.array[5]; gate++){ // if memGateIn is greater than 0.3, then let mem = the sum inputs attached to memGateIn
                if(memGateIn[gate] > 0.5){ //gate -startOfMemGateIn = [0, num of mem neurons]
                    for(connectPairMatrix it = connections.array; it!= connections.array+connections.size; ++it){
                        thrust::pair<int, int>itr = static_cast<thrust::pair<int, int> >(*it);
                        if(itr.second == gate+startOfMemGateIn && itr.first < gate+startOfHidden){//only pass inputs
                            mem[gate] += input[itr.first-startOfInput]; // no weights attached, but the old value stored here is not removed.
                        }
                    }
                }
                if(memGateForget[gate] > 0.5){// if memGateForget is greater than 0.5, then tell mem to forget
                    mem[gate] = 0;
                }
                //if memGateForget fires, then memGateOut will output nothing.
                if(memGateOut[gate] > 0.5){//if memGateOut is greater than 0.3, let the nodes mem is connected to recieve mem
                    for(connectPairMatrix it = connections.array; it!= connections.array+connections.size; ++it){
                        thrust::pair<int, int>itr = static_cast<thrust::pair<int, int> >(*it);
                        if(itr.first == gate+startOfMem){// since mem node: memIn node : memOut node = 1:1:1, we can do this.
                            hidden[itr.second-startOfHidden] += mem[gate];
                        }
                    }
                }
            }

            // hidden neuron nodes --
            for(int hid=0; hid<params.array[4]; hid++){ // for all hidden neurons at layer 1, lets sum the inputs, the memory values were already added.
                for(connectPairMatrix it = connections.array; it!= connections.array+connections.size; ++it){ // Add the inputs to the hidden neurons
                    thrust::pair<int, int>itr = static_cast<thrust::pair<int, int> >(*it);
                    if(itr.second == hid+startOfHidden && itr.first < startOfHidden && itr.first >= startOfInput){ // if an input connects with this hidden neuron
                        hidden[hid] += input[itr.first]*weights.array[ind + n++];
                    }
                }
                for(connectPairMatrix it = connections.array; it!= connections.array+connections.size; ++it){//add other hidden neuron inputs to each hidden neuron (if applicable)
                    thrust::pair<int, int>itr = static_cast<thrust::pair<int, int> >(*it);
                    if(itr.second == hid+startOfHidden && itr.first < startOfMem && itr.first >= startOfHidden){
                        hidden[hid] += hidden[itr.first-startOfHidden]*weights.array[ind + n++];
                    }
                }
                hidden[hid] += 1*weights.array[ind + n++]; // add bias
                hidden[hid] = ActFunc(hidden[hid]); // then squash itr.
            }
            //output nodes --

            for(int out =0; out<params.array[6]; out++){// add hidden neurons to the output nodes
                for(connectPairMatrix it = connections.array; it!= connections.array+connections.size; ++it){
                    thrust::pair<int, int>itr = static_cast<thrust::pair<int, int> >(*it);
                    if(itr.second == out+startOfOutput){
                        outputs[out] += hidden[itr.first-startOfHidden]*weights.array[ind + n++];
                    }
                }
                outputs[out] += 1*weights.array[ind + n++]; // add bias
                outputs[out] = ActFunc(outputs[out]);// then squash itr.
            }


            When[j] += outputs[0]*((2160-hour)-hour)+2160-hour; // nv = ((ov - omin)*(nmax-nmin) / (omax - omin))+nmin
            HowCertain[j] += outputs[1];
            CommunityMag[j] =  outputs[2]; // set the next sets communityMag = output #3.
        }
    }
    for(int j=0; j<numOfSites; j++){ // now lets get the average when and howcertain values.
        When[j] = When[j]/3600*sampleRate;
        HowCertain[j] = HowCertain[j]/3600*sampleRate;
    }
    // calculate performance for this individual - score = 1/(abs(whenGuess-whenReal)*distToQuake), for whenGuess = when[j] where HowCertain is max for set.
    //distToQuake is from the current sites parameters, it emphasizes higher scores for the closest site, a smaller distance is a higher score.
    int maxCertainty=0;
    double whenGuess=0;
    double latSite;
    double lonSite;
    for(int j=0; j<numOfSites; j++){
        if(HowCertain[j] > maxCertainty){
            whenGuess = When[j];
            latSite = siteData.array[j*2];
            lonSite = siteData.array[j*2+1];
        }
    }
    double SiteToQuakeDist = distCalc(latSite, lonSite, answers.array[2], answers.array[3]); // [2] is latitude, [3] is longitude.
    double fitness = 1/(abs(whenGuess - answers.array[1]-hour)*SiteToQuakeDist);//larger is better, negative numbers are impossible.
    weights.array[ind + params.array[2]+2] = fitness; // set the fitness number for the individual.
}

__global__ void reduce_by_block(dataArray<double> weights,
                          dataArray<double> per_block_results,
                          dataArray<int> params)
{
  extern __shared__ float sdata[];

  unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int fit = idx*params.array[7]+params.array[2]+2;

  // load input into __shared__ memory
  float x = 0;
  if(idx < params.array[8])
  {
    x = weights.array[fit];
  }
  sdata[threadIdx.x] = x;
  __syncthreads();

  // contiguous range pattern
  for(int offset = blockDim.x / 2;
      offset > 0;
      offset >>= 1)
  {
    if(threadIdx.x < offset)
    {
      // add a partial sum upstream to our own
      sdata[threadIdx.x] += sdata[threadIdx.x + offset];
    }

    // wait until all threads in the block have
    // updated their partial sums
    __syncthreads();
  }

  // thread 0 writes the final result
  if(threadIdx.x == 0)
  {
    per_block_results.array[blockIdx.x] = sdata[0];
  }
}

NetworkGenetic::NetworkGenetic(const int &numInputNodes, const int &numHiddenNeurons, const int &numMemoryNeurons,
                               const int &numOutNeurons, const int &numWeights, std::vector< thrust::pair<int, int> >&connections){
    this->_NNParams.resize(15, 0); // room to grow
    _NNParams[1] = numInputNodes + numHiddenNeurons + numMemoryNeurons + numOutNeurons;
    _NNParams[2] = numWeights;
    _NNParams[3] = numInputNodes;
    _NNParams[4] = numHiddenNeurons;
    _NNParams[5] = numMemoryNeurons;
    _NNParams[6] = numOutNeurons;
    _NNParams[7] = numWeights + 1 + 1; //1 for fitness, 1 for community output composite vector
    _connect = &connections;
}

void NetworkGenetic::initializeWeights(){
    int blocksPerGrid; //the blocksize defined by the configurator
    int threadsblock = 512; // the actual grid size needed
    int seedItr = 0;

    _NNParams[8] = _memVirtualizer._DGenetics.size()/(_NNParams[7]); // number of individuals on device.
    std::cerr<<"num of individuals about to have weights genned is: "<<_NNParams[8]<<std::endl;
    long seed = std::clock() + std::clock()*seedItr++;
    blocksPerGrid=(_NNParams[8]+threadsblock-1)/threadsblock;
    genWeights<<<blocksPerGrid, threadsblock>>>(_memVirtualizer.genetics(), seed, convertToKernel(_NNParams));
    CUDA_SAFE_CALL( hipPeekAtLastError() );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    CUDA_SAFE_CALL( hipPeekAtLastError() );
    //        }while(_memVirtualizer.GeneticsPushToHost(&_genetics));
    //        _NNParams[9] = _genetics.size/(_NNParams[8]); // number of individuals on device.
    //        long seed = std::clock() + std::clock()*seedItr++;
    //        blocksPerGrid=(_NNParams[9]+threadsblock-1)/threadsblock;
    //        genWeights<double><<< blocksPerGrid, threadsblock>>>(_genetics, seed, _NNParams[2], _NNParams[8]);
    //        hipDeviceSynchronize();
}


void NetworkGenetic::allocateHostAndGPUObjects(float pMaxHost, float pMaxDevice){
    _memVirtualizer.memoryAlloc(_NNParams[7], pMaxHost, pMaxDevice);

}
bool NetworkGenetic::init(int sampleRate, int SiteNum, std::vector<double> siteData){
    _sampleRate = sampleRate;
    _numofSites = SiteNum;
    _siteData.resize(siteData.size());
    try{thrust::copy(siteData.begin(), siteData.end(), _siteData.begin());}
    catch(thrust::system_error &e){
        std::cerr<<"Error resizing vector Element: "<<e.what()<<std::endl;
        exit(-1);
    }
    catch(std::bad_alloc &e){
        std::cerr<<"Ran out of space due to : "<<"host"<<std::endl;
        std::cerr<<e.what()<<std::endl;
        exit(-1);
    }
    _istraining = false;
    return true;
}

bool NetworkGenetic::checkForWeights(std::string filepath){
    std::ifstream weightFile;
    std::cerr<<"checking for weights.."<<std::endl;
    weightFile.open(filepath.c_str(), std::ios_base::in);
    if(weightFile){
        std::cerr<<"the weightfile exists"<<std::endl;
        _memVirtualizer.initFromStream(weightFile);
        weightFile.close();
        return true;
    }
    else{
        std::cerr<<"no weightfile found"<<std::endl;
        weightFile.close();
        return false;
    }
}

void NetworkGenetic::doingTraining(int site, int hour, double lat,
                                   double lon, double mag, double dist){
    _answers.push_back(site);
    _answers.push_back(hour);
    _answers.push_back(lat);
    _answers.push_back(lon);
    _answers.push_back(mag);
    _answers.push_back(dist);
    _istraining = true;
}

void NetworkGenetic::storeWeights(std::string filepath){
    _memVirtualizer.pushToStream(filepath);
}

void NetworkGenetic::forecast(double *ret, int &hour, std::vector<int> *data, double &Kp, std::vector<double> *globalQuakes)
{
    //were going to normalize the inputs using v` = v-mean/stdev, so we need mean and stdev for each channel.
    double meanCh1=0, meanCh2=0, meanCh3=0, stdCh1=0, stdCh2=0, stdCh3=0;
    int num=0;
    std::cerr<<"right before mean & std calc"<<std::endl;
    for(int i=0; i<3600*_sampleRate; i++){
        for(int j=0; j < _numofSites; j++){
            meanCh1 += data->at(3600*_sampleRate*j*3 + 0*3600*_sampleRate+i);
            meanCh2 += data->at(3600*_sampleRate*j*3 + 1*3600*_sampleRate+i);
            meanCh3 += data->at(3600*_sampleRate*j*3 + 2*3600*_sampleRate+i);
            num++;
        }
    }
    meanCh1 = meanCh1/num;
    meanCh2 = meanCh2/num;
    meanCh3 = meanCh3/num;
    stdCh1 = sqrt(meanCh1);
    stdCh2 = sqrt(meanCh2);
    stdCh3 = sqrt(meanCh3);
    std::cerr<<"means are: "<<meanCh1<<" "<<meanCh2<<" "<<meanCh3<<std::endl;
    std::cerr<<"stdevs are: "<<stdCh1<<" "<<stdCh2<<" "<<stdCh3<<std::endl;
    std::cerr<<"channels std and mean calculated"<<std::endl;
    //input data from all sites and all channels normalized
    if(_istraining == true){
        std::cerr<<"about to create device vectors"<<std::endl;
        thrust::device_vector<int> input;
        thrust::device_vector<double> retVec;
        thrust::device_vector<double> gQuakeAvg;
        thrust::device_vector<thrust::pair<int, int> > dConnect;
        try{input.resize(data->size());}
        catch(thrust::system_error &err){fprintf (stderr, "thrust error in file '%s' in line %i : %s.\n",__FILE__, __LINE__, err.what() );}
        try{retVec.resize(2160*_numofSites);}
        catch(thrust::system_error &err){fprintf (stderr, "thrust error in file '%s' in line %i : %s.\n",__FILE__, __LINE__, err.what() );}
        try{gQuakeAvg.resize(globalQuakes->size());}
        catch(thrust::system_error &err){fprintf (stderr, "thrust error in file '%s' in line %i : %s.\n",__FILE__, __LINE__, err.what() );}
        try{dConnect.resize(_connect->size());}
        catch(thrust::system_error &err){fprintf (stderr, "thrust error in file '%s' in line %i : %s.\n",__FILE__, __LINE__, err.what() );}
        std::cerr<<"all vectors resized"<<std::endl;
        try{thrust::copy(_connect->begin(), _connect->end(), dConnect.begin());}
        catch(thrust::system_error &err){fprintf (stderr, "thrust error in file '%s' in line %i : %s.\n",__FILE__, __LINE__, err.what() );}
        try{thrust::copy(data->begin(), data->end(), input.begin());}
        catch(thrust::system_error &err){fprintf (stderr, "thrust error in file '%s' in line %i : %s.\n",__FILE__, __LINE__, err.what() );}
        try{thrust::copy(globalQuakes->begin(), globalQuakes->end(), gQuakeAvg.begin());}
        catch(thrust::system_error &err){fprintf (stderr, "thrust error in file '%s' in line %i : %s.\n",__FILE__, __LINE__, err.what() );}
        int blocksPerGrid; //the blocksize defined by the configurator
        int blockSize = 512; // the actual grid size needed
        std::cerr<<"about to run cuda kernel.."<<std::endl;
        _NNParams[8] = _memVirtualizer._DGenetics.size()/(_NNParams[7]);
        std::cerr<<"number of threads is :"<<_NNParams[8]<<std::endl;
        blocksPerGrid=(_NNParams[8]+blockSize-1)/blockSize;
        Net<<<blockSize, blocksPerGrid>>>(_memVirtualizer.genetics(), convertToKernel(_NNParams),convertToKernel(gQuakeAvg),
                                          convertToKernel(input),convertToKernel(_siteData),convertToKernel(_answers),
                                          convertToKernel(dConnect),Kp,_sampleRate,_numofSites,hour,
                                          meanCh1, meanCh2, meanCh3, stdCh1, stdCh2, stdCh3);
        CUDA_SAFE_CALL(hipPeekAtLastError());
        CUDA_SAFE_CALL(hipDeviceSynchronize());
        int num_blocks = (_NNParams[8]/blockSize)+((_NNParams[8]%blockSize) ? 1 : 0);
        thrust::device_vector<double> partial_reduce_sums(num_blocks+1);
        reduce_by_block<<<num_blocks, blockSize, blockSize*sizeof(double)>>>(_memVirtualizer.genetics(), // calculate the partial sums on the GPU.
                                                                             convertToKernel(partial_reduce_sums),convertToKernel(_NNParams));
        double fitnessAvg=0;
        for(thrust::device_vector<double>::iterator it = partial_reduce_sums.begin(); // then since there shouldn't be THAT many blocks (~23437) lets calculate it on the CPU.
            it != partial_reduce_sums.end(); ++it){
            fitnessAvg += *it;
        }
        fitnessAvg = fitnessAvg /(num_blocks+1);
        std::cerr<<"the average fitness for this round is: "<<fitnessAvg<<std::endl;
    }
    else{
        std::cerr<<"entered not training version.."<<std::endl;
        typedef std::vector<thrust::pair<int, int> > connectPairMatrix;
        //replace this later
        _best.resize(_NNParams[2]);
        for(std::vector<double>::iterator it = _best.begin(); it != _best.end(); ++it){
            std::srand(std::time(NULL)+*it);
            *it = (double)(std::rand())/(RAND_MAX);
        }
        std::cerr<<"example best vector has been set."<<std::endl;
        double CommunityLat = 0;
        double CommunityLon = 0;
        std::vector<double> When(_numofSites, 0);
        std::vector<double> HowCertain(_numofSites,0);
        std::vector<double> CommunityMag(_numofSites, 1); //give all sites equal mag to start, this value is [0,1]
        std::cerr<<"all output vectors created and initialized."<<std::endl;
        for(int step=0; step<3600*_sampleRate; step++){
            std::cerr<<"entering step #"<<step<<std::endl;
            for(int j=0; j<_numofSites; j++){ //sitesWeighted Lat/Lon values are determined based on all previous sites mag output value.
                CommunityLat += _siteData[j*2]*CommunityMag[j];
                CommunityLon += _siteData[j*2+1]*CommunityMag[j];
            }
            CommunityLat = CommunityLat/_numofSites;
            CommunityLon = CommunityLon/_numofSites;

            for(int j=0; j<_numofSites; j++){ // each site is run independently of others, but shares an output from the previous step
                std::cerr<<"entering site #"<<j<<std::endl;
                double latSite = _siteData[j*2];
                double lonSite = _siteData[j*2+1];
                double avgLatGQuake = globalQuakes->at(0);
                double avgLonGQuake = globalQuakes->at(1);
                double GQuakeAvgMag = globalQuakes->at(3);
                double GQuakeAvgdist = distCalc(latSite, lonSite, avgLatGQuake, avgLonGQuake);
                double GQuakeAvgBearing = bearingCalc(latSite, lonSite, avgLatGQuake, avgLonGQuake);
                double CommunityDist = distCalc(latSite, lonSite, CommunityLat, CommunityLon);
                double CommunityBearing = bearingCalc(latSite, lonSite, CommunityLat, CommunityLon);
                std::vector<double> input;
                std::vector<double> hidden, outputs, mem, memGateOut, memGateIn, memGateForget;
                //replace these with real connections, num of inputs, and num of hidden & memory neurons (mem neurons probably accurate)
                input.resize(_NNParams[3], 0); // number of inputs is 9.
                hidden.resize(_NNParams[4], 0); // for practice sake, lets say each input has its own neuron (might be true!)
                mem.resize(_NNParams[5], 0); // stores the input if gate is high
                memGateOut.resize(_NNParams[5], 0); //connects to the input layer and the memN associated with input, if 1 it sends up stream and deletes, if low it keeps.
                memGateIn.resize(_NNParams[5], 0);
                memGateForget.resize(_NNParams[5], 0);
                outputs.resize(_NNParams[6], 0); /* 3 outputs, 1 with an hour in the future when the earthquake will hit,
                    1 with the porbability of that earthquake happening (between [0,1]) and 1 with the sites magnitude (for community feedback) */
                std::cerr<<"all neuron vectors are sized, all pre-net calculations done."<<std::endl;
                int n =0;
                int startOfInput = 0;
                int startOfHidden = startOfInput +_NNParams[3];
                int startOfMem = startOfHidden + _NNParams[4];
                int startOfMemGateIn = startOfMem + _NNParams[5];
                int startOfMemGateOut = startOfMemGateIn + _NNParams[5];
                int startOfMemGateForget = startOfMemGateOut + _NNParams[5];
                int startOfOutput = startOfMemGateForget + _NNParams[5];
                input[0] = shift((double)(data->at(3600*_sampleRate*j*3 + 0*(3600*_sampleRate)+step)), meanCh1, stdCh1);
                input[1] = normalize((double)(data->at(3600*_sampleRate*j*3 + 1*(3600*_sampleRate)+step)), meanCh2, stdCh2);
                input[2] = normalize((double)(data->at(3600*_sampleRate*j*3 + 2*(3600*_sampleRate)+step)), meanCh3, stdCh3);
                input[3] = shift(GQuakeAvgdist, 40075.1, 0);
                input[4] = shift(GQuakeAvgBearing, 360, 0);
                input[5] = shift(GQuakeAvgMag, 9.5, 0);
                input[6] = shift(Kp, 10, 0);
                input[7] = shift(CommunityDist,40075.1/2, 0);
                input[8] = shift(CommunityBearing, 360, 0);
                //lets reset all neuron values for this new timestep (except memory neurons)
                for(int gate=0; gate<_NNParams[5]; gate++){
                    memGateIn.at(gate) = 0;
                    memGateOut.at(gate) = 0;
                    memGateForget.at(gate) = 0;
                }
                for(int hid=0; hid<_NNParams[4]; hid++){
                    hidden[hid] = 0;
                }
                for(int out=0; out<_NNParams[6]; out++){
                    outputs[out] = 0;
                }
                std::cerr<<"memGate, hidden, and output neurons are zeroed."<<std::endl;
                //now that everything that should be zeroed is zeroed, lets start the network.
                //mem gates & LSTM nodes --
                std::cerr<<"preparing to set the values for memoryGates."<<std::endl;
                for(int gate = 0; gate<_NNParams[5]; gate++){//calculate memory gate node values, you can connect inputs & hidden neurons to them.
                    for(connectPairMatrix::iterator it = _connect->begin(); it!= _connect->end(); ++it){//for memGateIn
                        if(it->second == gate+startOfMemGateIn && it->first < startOfHidden){ //for inputs
                            std::cerr<<"weights for memGateIn #"<<gate<<" is: "<<_best[n];
                            memGateIn.at(gate) += input[it->first-startOfInput]*_best[n++]; // memGateIn vect starts at 0
                        }
                        else if(it->second == gate+startOfMemGateIn && it->first >startOfHidden && it->first < startOfMem){//for hidden neurons
                            memGateIn.at(gate) += hidden[it->first-startOfHidden]*_best[n++];
                        }
                    }
                    for(connectPairMatrix::iterator it = _connect->begin(); it!= _connect->end(); ++it){//for memGateOut
                        if(it->second == gate+startOfMemGateOut && it->first < startOfHidden){//for inputs
                            std::cerr<<"weights for memGateOut #"<<gate<<" is: "<<_best[n];
                            memGateOut.at(gate) += input[it->first-startOfInput]*_best[n++];
                        }
                        else if(it->second == gate+startOfMemGateOut && it->first >startOfHidden && it->first <startOfMem){//for hidden neurons
                            memGateOut.at(gate) += hidden[it->first-startOfHidden]*_best[n++];
                        }
                    }
                    for(connectPairMatrix::iterator it = _connect->begin(); it!= _connect->end(); ++it){//for  memGateForget
                        if(it->second == gate+startOfMemGateForget && it->first < startOfHidden){//for inputs
                            std::cerr<<"weights for memGateForget #"<<gate<<" is: "<<_best[n];
                            memGateForget.at(gate) += input[it->first - startOfInput]*_best[n++];
                        }
                        else if(it->second == gate+startOfMemGateForget && it->first >startOfHidden && it->first <startOfMem){//for hidden neurons
                            memGateForget.at(gate) += hidden[it->first-startOfHidden]*_best[n++];
                        }
                    }
                    memGateIn.at(gate) = ActFunc(memGateIn.at(gate));
                    memGateOut.at(gate) = ActFunc(memGateOut.at(gate));
                    memGateForget.at(gate) = ActFunc(memGateForget.at(gate));
                    std::cerr<<"memGateIn val: "<<memGateIn.at(gate)<<std::endl;
                    std::cerr<<"memGateOut val: "<<memGateOut.at(gate)<<std::endl;
                    std::cerr<<"memGateForget val: "<<memGateForget.at(gate)<<std::endl;
                }
                //since we calculated the values for memGateIn and memGateOut, and MemGateForget..
                for (int gate = 0; gate<_NNParams[5]; gate++){ // if memGateIn is greater than 0.3, then let mem = the sum inputs attached to memGateIn
                    if(memGateIn.at(gate) > 0.5){ //gate -startOfMemGateIn = [0, num of mem neurons]
                        for(connectPairMatrix::iterator it = _connect->begin(); it!= _connect->end(); ++it){
                            if(it->second == gate+startOfMemGateIn && it->first < gate+startOfHidden){//only pass inputs
                                mem.at(gate) += input[it->first-startOfInput]; // no weights attached, but the old value stored here is not removed.
                            }
                        }
                    }
                    if(memGateForget.at(gate) > 0.5){// if memGateForget is greater than 0.5, then tell mem to forget
                        mem.at(gate) = 0;
                    }
                    //if memGateForget fires, then memGateOut will output nothing.
                    if(memGateOut.at(gate) > 0.5){//if memGateOut is greater than 0.3, let the nodes mem is connected to recieve mem
                        for(connectPairMatrix::iterator it = _connect->begin(); it!= _connect->end(); ++it){
                            if(it->first == gate+startOfMem){// since mem node: memIn node : memOut node = 1:1:1, we can do this.
                                hidden[it->second-startOfHidden] += mem.at(gate);
                            }
                        }
                    }
                    std::cerr<<"mem val stored is: "<<mem.at(gate)<<std::endl;
                }

                // hidden neuron nodes --
                for(int hid=0; hid<_NNParams[4]; hid++){ // for all hidden neurons at layer 1, lets sum the inputs, the memory values were already added.
                    for(connectPairMatrix::iterator it = _connect->begin(); it!= _connect->end(); ++it){ // Add the inputs to the hidden neurons
                        if(it->second == hid+startOfHidden && it->first < startOfHidden){ // if an input connects with this hidden neuron
                            hidden[hid] += input[it->first]*_best[n++];
                        }
                    }
                    for(connectPairMatrix::iterator it = _connect->begin(); it!= _connect->end(); ++it){//add other hidden neuron inputs to each hidden neuron (if applicable)
                        if(it->second == hid+startOfHidden && it->first < startOfMem && it->first > startOfHidden){
                            hidden[hid] += hidden[it->first-startOfHidden]*_best[n++];
                        }
                    }
                    hidden[hid] += 1*_best[n++]; // add bias
                    hidden[hid] = ActFunc(hidden[hid]); // then squash it.
                    std::cerr<<"hidden nueron values: "<<hidden[hid]<<std::endl;
                }
                //output nodes --

                for(int out =0; out<_NNParams[6]; out++){// add hidden neurons to the output nodes
                    for(connectPairMatrix::iterator it = _connect->begin(); it!= _connect->end(); ++it){
                        if(it->second == out+startOfOutput){
                            outputs[out] += hidden[it->first-startOfHidden]*_best[n++];
                        }
                    }
                    outputs[out] += 1*_best[n++]; // add bias
                    outputs[out] = ActFunc(outputs[out]);// then squash it.
                }


                When[j] += outputs[0]*((2160-hour)-hour)+2160-hour;; //return when back to an integer value (adjust to fit within boundaries)
                std::cerr<<"When for site: "<<j<<" and for step: "<<step<< " is: "<<When[j]<<std::endl;
                HowCertain[j] += outputs[1];
                std::cerr<<"howCertain for site: "<<j<<" and for step: "<<step<< " is: "<<HowCertain[j]<<std::endl;
                CommunityMag[j] =  outputs[2]; // set the next sets communityMag = output #3.
                std::cerr<<"ComunityMagnitude for site: "<<j<<" and for step: "<<step<< " is: "<<CommunityMag[j]<<std::endl;
            }
        }
        for(int j=0; j<_numofSites; j++){ // each site has its own when and howcertain vector
            When[j] = When[j]/3600*_sampleRate;
            HowCertain[j] = HowCertain[j]/3600*_sampleRate;
        }
        //all done, lets output the return matrix.
        //since right now were using a point value for when & how certain (only one output per site),
        //we're going to approximate using a normal distribution around when with a sigma of howCertain, over the whole array from T=currentHour [T, 2160]
        for(int h=hour; h<2160; h++){
            for(int j=0; j<_numofSites; j++){
                ret[h*_numofSites+j] = 1/(1/HowCertain[j]*sqrt(2*M_PI))*exp(-pow(h-When[j], 2)/(2*pow(1/HowCertain[j], 2))); // normal distribution with a mu of When and a sigma of 1/HowCertain
            }
        }
    }
}
