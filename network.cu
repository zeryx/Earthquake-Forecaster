#include "network.h"
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/random.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/system_error.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <hip/hip_runtime_api.h>

struct genRand: thrust::unary_function<Individual, int>{

    int numWeights;

    genRand(int _numWeights) : numWeights(_numWeights){}

    __host__ __device__
    Individual operator()(Individual n) const{
        unsigned int idx= threadIdx.x*blockDim.x;

        n._weights = new float[numWeights];

        for(int i=0; i<numWeights; i++){
            idx = idx +i;
            thrust::default_random_engine randEng;
            thrust::uniform_real_distribution<float> uniDist(0,1);
            randEng.discard(idx);
            n._weights[i] =  uniDist(randEng);
        }
        return n;
    }};


NetworkGenetic::NetworkGenetic(){}

NetworkGenetic::NetworkGenetic(const int &numInNeurons, const int &numHiddenNeurons,
                               const int &numOutNeurons, std::map<const int, int> &connections){
    _constantNNParams.push_back(numInNeurons);
    _constantNNParams.push_back(numHiddenNeurons);
    _constantNNParams.push_back(numOutNeurons);
    _neuronsTotalNum = numInNeurons + numHiddenNeurons + numOutNeurons;
    _connections = connections;
}

bool NetworkGenetic::generatePop(int popsize){
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    thrust::device_vector<Individual> testing;
    testing.resize(popsize);

    try{
        hipEventRecord(start);
        thrust::transform(testing.begin(),
                          testing.end(), testing.begin(), genRand(_neuronsTotalNum));
    }
    catch(thrust::system_error &err){
        std::cerr<<"error transforming: "<<err.what()<<std::endl;
        return false;
    }
    hipEventRecord(stop);
    float miliseconds = 0;
    std::vector<Individual> printer(testing.size());
    for(int i=0; i<popsize; i++){
        thrust::copy(testing.begin(), testing.end(),printer.begin());
        }
    for(int i=0; i<popsize; i++){
        for(int k=0; k<_neuronsTotalNum; k++){
            std::cout<< printer[k]._weights[k]<<std::endl;
        }
    }
    hipEventElapsedTime(&miliseconds, start, stop);

    return true;
}
