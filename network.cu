#include "hip/hip_runtime.h"
#include "network.h"
#include <iostream>
#include <fstream>
#include <thrust/device_vector.h>
#include <thrust/random.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>
#include <ctime>
#include <hip/hip_runtime.h>
#include <vector>
#include <hip/hip_runtime.h>
#include <tinyxml2.h>

template <typename T>
__global__ void genWeights( DataArray<T> ref, long in, int nRegWeights, int indLength){
    long idx = blockDim.x*blockIdx.x + threadIdx.x;
    long seed= idx+in;
    thrust::default_random_engine randEng;
    for(int i=0; i<nRegWeights; i++){
        thrust::uniform_real_distribution<double> uniDist(0,1);
        randEng.discard(seed);
        ref._array[idx*indLength+i] = uniDist(randEng);
    }
}

NetworkGenetic::NetworkGenetic(const int &numInNeurons, const int &numHiddenNeurons, const int &numMemoryNeurons,
                               const int &numOutNeurons, const int &numHiddenLayers,  const std::map<const int, int> &connections){
    this->_NNParams.resize(15, 0); // room to grow
    _NNParams[1] = numInNeurons + numHiddenNeurons + numMemoryNeurons + numOutNeurons;
    _NNParams[2] = numInNeurons + numHiddenNeurons + numOutNeurons;
    _NNParams[3] = numInNeurons;
    _NNParams[4] = numHiddenNeurons;
    _NNParams[5] = numMemoryNeurons;
    _NNParams[6] = numOutNeurons;
    _NNParams[7] = numHiddenLayers;
    _connections = connections;
}

void NetworkGenetic::initializeWeights(){
    hipEvent_t start, stop;
    CUDA_SAFE_CALL (hipEventCreate(&start));
    CUDA_SAFE_CALL (hipEventCreate(&stop));
    int blocksize; //the blocksize defined by the configurator
    int minGridSize; //the minimum grid size needed to achive max occupancy
    int gridSize; // the actual grid size needed
    int individualSize = _NNParams[1]+1;//contains all neruons, plus 1 for fitness vals
    _initialPopsize = _DGeneticsData.size()/(individualSize);
    std::cout<<"initial population: "<<_initialPopsize<<std::endl;
    long time = std::clock();
    hipEventRecord(start, 0);
    hipDeviceSynchronize();
    CUDA_SAFE_CALL (hipOccupancyMaxPotentialBlockSize( &minGridSize, &blocksize, (void*)genWeights<double>, 0, _initialPopsize));
    gridSize = (_initialPopsize + blocksize -1)/blocksize;
    genWeights<double><<<gridSize, blocksize>>>(convertToKernel<double>(_DGeneticsData), time, _NNParams[2], individualSize);
    hipDeviceSynchronize();
    float miliseconds = 0;
    CUDA_SAFE_CALL (hipEventRecord(stop, 0));
    hipDeviceSynchronize();
    CUDA_SAFE_CALL (hipEventElapsedTime(&miliseconds, start, stop));
    std::cout<<"weight generation: total compute time: "<<miliseconds<<" ms"<<std::endl;
    std::cout<<"effective bandwidth (GB/s) : "<<(_DGeneticsData.size()*8)/((miliseconds/1000)*1e9)<<std::endl;
}



void NetworkGenetic::init(std::string siteInfo){
    int SLEN, sampleRate;
    std::vector<double> sitesData;
   tinyxml2::XMLDocument doc;
   doc.LoadFile(siteInfo.c_str());
   std::string title = doc.FirstChildElement("Sites")->FirstChildElement("site")->GetText();
   std::cout<<"title"<<std::endl;
//    this->_HInitData[0] = doc.first_node()->
//    this->_HInitData[1] = sampleRate;
//    for(int i=0; i<SLEN; i++){
//        this->_HInitData[2+i] = sitesData[i];
//    }
}


void NetworkGenetic::allocateHostAndGPUObjects(unsigned int hostMemory, unsigned int deviceMemory,
                                               std::map<const std::string, float> pHostRam,  std::map<const std::string, float> pDeviceRam){
    unsigned int hostInitDataAlloc = (9*2+2)/sizeof(double); //sampleRate, number of sites, site lon/lat data.
    unsigned int hostGeneticsAlloc = hostMemory*pHostRam.at("genetics")/sizeof(double); //since these are doubles, divide bytes by 8
    unsigned int hostTrainingAlloc = hostMemory*pHostRam.at("input & training")/(sizeof(double)+2);//half for training, half for input I think?
    unsigned int hostInputsAlloc = hostMemory*pHostRam.at("input & training")/(sizeof(float)+2); // their either floats or ints, same amount of bytes.
    unsigned int deviceGeneticsAlloc = deviceMemory*pDeviceRam.at("genetics")/sizeof(double);
    unsigned int deviceTrainingAlloc = deviceMemory*pDeviceRam.at("input & training")/(sizeof(double)+2);
    unsigned int deviceInputsAlloc = deviceMemory*pDeviceRam.at("input & training")/(sizeof(double)+2);
    unsigned int devicePMAIAlloc = 2160*80/sizeof(double); //1.4 mb worth of planetary magnetic activity index for all tests, can store outside of container with other constants.
    //initialize all vectors
    this->_HInitData.resize(hostInitDataAlloc);
    this->_HGeneticsData.resize(hostGeneticsAlloc);
    this->_HTrainingData.resize(hostTrainingAlloc);
    this->_HInputData.resize(hostInputsAlloc);
    this->_DGeneticsData.resize(deviceGeneticsAlloc);
    this->_DTrainingData.resize(deviceTrainingAlloc);
    this->_DInputData.resize(deviceInputsAlloc);
    this->_DPMAIndex.resize(devicePMAIAlloc);
}

void NetworkGenetic::errorFunc(){
    //    hipEvent_t start, stop;
    //    hipEventCreate(&start);
    //    hipEventCreate(&stop);
    //    int blocksize; //the blocksize defined by the configurator
    //    int minGridSize; //the minimum grid size needed to achive max occupancy
    //    int gridSize; // the actual grid size needed
}
