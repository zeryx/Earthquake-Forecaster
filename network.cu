#include "hip/hip_runtime.h"
﻿#include "network.h"
#include <fstream>
#include <thrust/device_vector.h>
#include <thrust/random.h>
#include <utility>
#include <ctime>
#include <thrust/host_vector.h>

//macros
//cuda error message handling
#define CUDA_SAFE_CALL(call)                                          \
    do {                                                                  \
    hipError_t err = call;                                           \
    if (hipSuccess != err) {                                         \
    fprintf (stderr, "Cuda error in file '%s' in line %i : %s.\n",\
    __FILE__, __LINE__, hipGetErrorString(err) );       \
    exit(EXIT_FAILURE);                                           \
    }                                                                 \
    } while (0)

//neural functions
__host__ __device__ inline double sind(double x)
{
    return sin(x * M_PI / 180);
}

__host__ __device__ inline double cosd(double x)
{
    return cos(x * M_PI / 180);
}
__host__ __device__ inline double distCalc(double lat1, double lon1, double lat2, double lon2){
    double earthRad = 6371.01;
    double deltalon = abs(lon1 - lon2);
    if(deltalon > 180)
        deltalon = 360 - deltalon;
    return earthRad * atan2( sqrt( pow( cosd(lat1) * sind(deltalon), 2) +
                                   pow( cosd(lat2) * sind(lat1) - sind(lat2) * cosd(lat1) * cosd(deltalon), 2) ),
                             sind(lat2) * sind(lat1) + cosd(lat2) * cosd(lat1) * cosd(deltalon));
}

__host__ __device__ inline double bearingCalc(double lat1, double lon1, double lat2, double lon2){
    double dLon = (lon2 - lon1);

    double y = sin(dLon) * cos(lat2);
    double x = cos(lat1) * sin(lat2) - sin(lat1) * cos(lat2) * cos(dLon);

    double brng = atan2(y, x);

    brng = brng*M_PI/180;
    brng += 360;
    while(brng>= 360)
        brng -= 360;
    brng = 360 - brng;

    return brng;
}

__host__ __device__ inline double ActFunc(double x){
    return tanh(x);
}

template <typename T>
__global__ void genWeights( dataArray<T> ref, long in, int nRegWeights, int indLength){
    long idx = blockIdx.x * blockDim.x + threadIdx.x;
    thrust::minstd_rand0 randEng;
    randEng.seed(idx);
    long seed = idx+ref.size*in;
    for(int i=0; i<nRegWeights; i++){
        thrust::uniform_real_distribution<double> uniDist(0,1);
        randEng.discard(seed);
        ref.array[idx*indLength + i] = uniDist(randEng);
    }
}

__global__ void Net(dataArray<double> weights, dataArray<int> params, dataArray<double> globalQuakes,
                    dataArray<int> inputVal, dataArray<double> siteData,
                    dataArray<double> answers, dataArray<double> returnVal,
                    dataArray<thrust::pair<int, int> > connections, double Kp, int sampleRate,int numOfSites, int hour){



    int idx = blockIdx.x * blockDim.x + threadIdx.x; // for each thread, calculate a individuals weight.
    int ind = idx*params.array[7];
    typedef thrust::device_ptr<thrust::pair<int, int> > connectPairMatrix;
    double CommunityLat = 0;
    double CommunityLon = 0;
    double *When = (double*)malloc(numOfSites*sizeof(double));
    double *HowCertain = (double*)malloc(numOfSites*sizeof(double));
    double *CommunityMag = (double*)malloc(numOfSites*sizeof(double)); //give all sites equal mag to start, this value is [0,1]

    for(int step=0; step<3600*sampleRate; step++){

        for(int j=0; j<sampleRate; j++){//sitesWeighted Lat/Lon values are determined based on all previous sites mag output value.
            CommunityLat += siteData.array[j*2]*CommunityMag[j];
            CommunityLon += siteData.array[j*2+1]*CommunityMag[j];
        }
        CommunityLat = CommunityLat/numOfSites;
        CommunityLon = CommunityLon/numOfSites;

        for(int j=0; j<numOfSites; j++){ //each site is run independently of others, but shares an output from the previous step
            double latSite = siteData.array[j*2];
            double lonSite = siteData.array[j*2+1];
            double avgLatGQuake = globalQuakes.array[0];
            double avgLonGQuake = globalQuakes.array[1];
            //double avgDepthGQuake = globalQuakes.array[2); don't think I care about depth that much.
            double GQuakeAvgMag = globalQuakes.array[3];
            double GQuakeAvgdist = distCalc(latSite, lonSite, avgLatGQuake, avgLonGQuake);
            double GQuakeAvgBearing = bearingCalc(latSite, lonSite, avgLatGQuake, avgLonGQuake);
            double CommunityDist = distCalc(latSite, lonSite, CommunityLat, CommunityLon);
            double CommunityBearing = bearingCalc(latSite, lonSite, CommunityLat, CommunityLon);
            //replace these with real connections, num of inputs, and num of hidden & memory neurons (mem neurons probably accurate)
            int *input = (int*)malloc(params.array[3]*sizeof(int)); // number of inputs is 9.
            double *hidden = (double*)malloc(params.array[4]*sizeof(double)); // for practice sake, lets say each input has its own neuron (might be true!)
            double *mem = (double*)malloc(params.array[5]*sizeof(double)); // stores the input if gate is high
            double *memGateIn = (double*)malloc(params.array[5]*sizeof(double)); //connects to the input layer and the memN associated with input, if 1 it sends up stream and deletes, if low it keeps.
            double *memGateOut = (double*)malloc(params.array[5]*sizeof(double));
            double *memGateForget = (double*)malloc(params.array[5]*sizeof(double));
            double *outputs = (double*)malloc(params.array[6]*sizeof(double)); /* 3 outputs, 1 with an hour in the future when the earthquake will hit,
                1 with the porbability of that earthquake happening (between [0,1]) and 1 with the sites magnitude (for community feedback) */
            int n =0;
            int startOfInput = 0;
            int startOfHidden = startOfInput +params.array[3];
            int startOfMem = startOfHidden + params.array[4];
            int startOfMemGateIn = startOfMem + params.array[5];
            int startOfMemGateOut = startOfMemGateIn + params.array[5];
            int startOfMemGateForget = startOfMemGateOut + params.array[5];
            int startOfOutput = startOfMemGateForget + params.array[5];
            input[0] = 1/inputVal.array[(3600*sampleRate*j*3 + 1*(3600*sampleRate)+step)];//channel 1
            input[1] = 1/inputVal.array[(3600*sampleRate*j*3 + 2*(3600*sampleRate)+step)];//channel 2
            input[2] = 1/inputVal.array[(3600*sampleRate*j*3 + 3*(3600*sampleRate)+step)];//channel 3
            input[3] = 1/GQuakeAvgdist;
            input[4] = 1/GQuakeAvgBearing;
            input[5] = 1/GQuakeAvgMag;
            input[6] = 1/Kp;
            input[7] = 1/CommunityDist;
            input[8] = 1/CommunityBearing;
            //lets reset all neuron values for this new timestep (except memory neurons)
            for(int gate=0; gate<params.array[5]; gate++){
                memGateIn[gate] = 0;
                memGateOut[gate] = 0;
                memGateForget[gate] = 0;
            }
            for(int hid=0; hid<params.array[4]; hid++){
                hidden[hid] = 0;
            }
            for(int out=0; out<params.array[6]; out++){
                outputs[out] = 0;
            }

            //now that everything that should be zeroed is zeroed, lets start the network.
            //mem gates & LSTM nodes --
            for(int gate = 0; gate<params.array[5]; gate++){//calculate memory gate node values, you can connect inputs & hidden neurons to them.
                for(connectPairMatrix it = connections.array; it!= connections.array+connections.size; ++it){//for memGateIn
                    thrust::pair<int, int>itr = static_cast<thrust::pair<int, int> >(*it); // this needs to be created to use the iterator it correctly.
                    if(itr.second == gate+startOfMemGateIn && itr.second < startOfHidden){ //for inputs
                        memGateIn[gate] += input[itr.first-startOfInput]*weights.array[ind + n++]; // memGateIn vect starts at 0
                    }
                    else if(itr.second == gate+startOfMemGateIn && itr.second >startOfHidden && itr.second <startOfMem){//for hidden neurons
                        memGateIn[gate] += hidden[itr.first-startOfHidden]*weights.array[ind + n++];
                    }
                }
                for(connectPairMatrix it = connections.array; it!= connections.array+connections.size; ++it){//for memGateOut
                    thrust::pair<int, int>itr = static_cast<thrust::pair<int, int> >(*it);
                    if(itr.second == gate+startOfMemGateOut && itr.second < startOfHidden){//for inputs
                        memGateOut[gate] += input[itr.first-startOfInput]*weights.array[ind + n++];
                    }
                    else if(itr.second == gate+startOfMemGateOut && itr.second >startOfHidden && itr.second <startOfMem){//for hidden neurons
                        memGateOut[gate] += hidden[itr.first-startOfHidden]*weights.array[ind + n++];
                    }
                }
                for(connectPairMatrix it = connections.array; it!= connections.array+connections.size; ++it){//for  memGateForget
                    thrust::pair<int, int>itr = static_cast<thrust::pair<int, int> >(*it);
                    if(itr.second == gate+startOfMemGateForget && itr.second < startOfHidden){//for inputs
                        memGateForget[gate] += input[itr.first - startOfInput]*weights.array[ind + n++];
                    }
                    else if(itr.second == gate+startOfMemGateForget && itr.second >startOfHidden && itr.second <startOfMem){//for hidden neurons
                        memGateForget[gate] += hidden[itr.first-startOfHidden]*weights.array[ind + n++];
                    }
                }
                memGateIn[gate] = ActFunc(memGateIn[gate]);
                memGateOut[gate] = ActFunc(memGateOut[gate]);
                memGateForget[gate] = ActFunc(memGateForget[gate]);
            }
            //since we calculated the values for memGateIn and memGateOut, and MemGateForget..
            for (int gate = 0; gate<params.array[5]; gate++){ // if memGateIn is greater than 0.3, then let mem = the sum inputs attached to memGateIn
                if(memGateIn[gate] > 0.5){ //gate -startOfMemGateIn = [0, num of mem neurons]
                    for(connectPairMatrix it = connections.array; it!= connections.array+connections.size; ++it){
                        thrust::pair<int, int>itr = static_cast<thrust::pair<int, int> >(*it);
                        if(itr.second == gate+startOfMemGateIn && itr.first < gate+startOfHidden){//only pass inputs
                            mem[gate] += input[itr.first-startOfInput]; // no weights attached, but the old value stored here is not removed.
                        }
                    }
                }
                if(memGateForget[gate] > 0.5){// if memGateForget is greater than 0.5, then tell mem to forget
                    mem[gate] = 0;
                }
                //if memGateForget fires, then memGateOut will output nothing.
                if(memGateOut[gate] > 0.5){//if memGateOut is greater than 0.3, let the nodes mem is connected to recieve mem
                    for(connectPairMatrix it = connections.array; it!= connections.array+connections.size; ++it){
                        thrust::pair<int, int>itr = static_cast<thrust::pair<int, int> >(*it);
                        if(itr.first == gate+startOfMem){// since mem node: memIn node : memOut node = 1:1:1, we can do this.
                            hidden[itr.second-startOfHidden] += mem[gate];
                        }
                    }
                }
            }

            // hidden neuron nodes --
            for(int hid=0; hid<params.array[4]; hid++){ // for all hidden neurons at layer 1, lets sum the inputs, the memory values were already added.
                for(connectPairMatrix it = connections.array; it!= connections.array+connections.size; ++it){ // Add the inputs to the hidden neurons
                    thrust::pair<int, int>itr = static_cast<thrust::pair<int, int> >(*it);
                    if(itr.second == hid+startOfHidden && itr.first < startOfHidden && itr.first >= startOfInput){ // if an input connects with this hidden neuron
                        hidden[hid] += input[itr.first]*weights.array[ind + n++];
                    }
                }
                for(connectPairMatrix it = connections.array; it!= connections.array+connections.size; ++it){//add other hidden neuron inputs to each hidden neuron (if applicable)
                    thrust::pair<int, int>itr = static_cast<thrust::pair<int, int> >(*it);
                    if(itr.second == hid+startOfHidden && itr.first < startOfMem && itr.first >= startOfHidden){
                        hidden[hid] += hidden[itr.first-startOfHidden]*weights.array[ind + n++];
                    }
                }
                hidden[hid] += 1*weights.array[ind + n++]; // add bias
                hidden[hid] = ActFunc(hidden[hid]); // then squash itr.
            }
            //output nodes --

            for(int out =0; out<params.array[6]; out++){// add hidden neurons to the output nodes
                for(connectPairMatrix it = connections.array; it!= connections.array+connections.size; ++it){
                    thrust::pair<int, int>itr = static_cast<thrust::pair<int, int> >(*it);
                    if(itr.second == out+startOfOutput){
                        outputs[out] += hidden[itr.first-startOfHidden]*weights.array[ind + n++];
                    }
                }
                outputs[out] += 1*weights.array[ind + n++]; // add bias
                outputs[out] = ActFunc(outputs[out]);// then squash itr.
            }


            When[j] += 1/outputs[0]; //return when back to an integer value (adjust to fit within boundaries)
            HowCertain[j] += outputs[1];
            CommunityMag[j] =  outputs[2]; // set the next sets communityMag = output #3.
        }
    }
    for(int j=0; j<numOfSites; j++){ // each site has its own when and howcertain vector
        When[j] = When[j]/3600*sampleRate;
        HowCertain[j] = HowCertain[j]/3600*sampleRate;
    }
    //all done, lets output the return matrix.
    //since right now were using a point value for when & how certain (only one output per site),
    //we're going to approximate using a normal distribution around when with a sigma of howCertain, over the whole array from T=currentHour [T, 2160]
    for(int h=hour; h<2160; h++){
        for(int j=0; j<numOfSites; j++){
            returnVal.array[h*numOfSites+j] = 1/(1/HowCertain[j]*sqrt(2*M_PI))*exp(-pow(h-When[j], 2)/(2*pow(1/HowCertain[j], 2))); // normal distribution with a mu of When and a sigma of 1/HowCertain
        }
    }
}



NetworkGenetic::NetworkGenetic(const int &numInputNodes, const int &numHiddenNeurons, const int &numMemoryNeurons,
                               const int &numOutNeurons, std::vector< thrust::pair<int, int> >&connections){
    this->_NNParams.resize(15, 0); // room to grow
    _NNParams[1] = numInputNodes + numHiddenNeurons + numMemoryNeurons + numOutNeurons;
    _NNParams[2] = numInputNodes + numHiddenNeurons + numOutNeurons;
    _NNParams[3] = numInputNodes;
    _NNParams[4] = numHiddenNeurons;
    _NNParams[5] = numMemoryNeurons;
    _NNParams[6] = numOutNeurons;
    _NNParams[7] = numInputNodes + numHiddenNeurons + numMemoryNeurons + numOutNeurons + 1 + 1; //1 for fitness, 1 for community output composite vector
    _connect = &connections;
}

void NetworkGenetic::initializeWeights(){
    int blocksPerGrid; //the blocksize defined by the configurator
    int threadsblock = 512; // the actual grid size needed
    int seedItr = 0;
    //    do{
    _NNParams[8] = _memVirtualizer._DGenetics.size()/(_NNParams[7]); // number of individuals on device.
    long seed = std::clock() + std::clock()*seedItr++;
    blocksPerGrid=(_NNParams[9]+threadsblock-1)/threadsblock;
    genWeights<double><<<blocksPerGrid, threadsblock>>>(_memVirtualizer.genetics(), seed, _NNParams[2], _NNParams[8]);
    hipDeviceSynchronize();
    //    }while(_memVirtualizer.GeneticsPushToHost(&_genetics));
    //    _NNParams[9] = _genetics.size/(_NNParams[8]); // number of individuals on device.
    //    long seed = std::clock() + std::clock()*seedItr++;
    //    blocksPerGrid=(_NNParams[9]+threadsblock-1)/threadsblock;
    //    genWeights<double><<< blocksPerGrid, threadsblock>>>(_genetics, seed, _NNParams[2], _NNParams[8]);
    //    hipDeviceSynchronize();
}


void NetworkGenetic::allocateHostAndGPUObjects(float pMaxHost, float pMaxDevice){
    _memVirtualizer.memoryAlloc(_NNParams[8], pMaxHost, pMaxDevice);

}
bool NetworkGenetic::init(int sampleRate, int SiteNum, std::vector<double> siteData){
    _sampleRate = sampleRate;
    _numofSites = SiteNum;
    _siteData.resize(siteData.size());
    try{thrust::copy(siteData.begin(), siteData.end(), _siteData.begin());}
    catch(thrust::system_error &e){
        std::cerr<<"Error resizing vector Element: "<<e.what()<<std::endl;
        return false;
    }
    catch(std::bad_alloc &e){
        std::cerr<<"Ran out of space due to : "<<"host"<<std::endl;
        std::cerr<<e.what()<<std::endl;
        return false;
    }
    _istraining = false;
    return true;
}

bool NetworkGenetic::checkForWeights(std::string filepath){
    std::ifstream weightFile;
    weightFile.open(filepath.c_str(), std::ios_base::in);
    if(weightFile){
        _memVirtualizer.initFromStream(weightFile);
        return true;
    }
    else
        return false;
}

void NetworkGenetic::doingTraining(int site, int hour, double lat, double lon, double mag, double dist){
    _answers.push_back(site);
    _answers.push_back(hour);
    _answers.push_back(lat);
    _answers.push_back(lon);
    _answers.push_back(mag);
    _answers.push_back(dist);
    _istraining = true;
}

void NetworkGenetic::storeWeights(std::string filepath){
    _memVirtualizer.pushToStream(filepath);
}

void NetworkGenetic::forecast(double *ret, int &hour, std::vector<int> *data, double &Kp, std::vector<double> *globalQuakes)
{
    std::cerr<<"entered forecast"<<std::endl;
    if(_istraining){
        thrust::device_vector<double> retVec(2160*_numofSites, 0);
        thrust::device_vector<int> input(data->size());
        thrust::device_vector<double>gQuakeAvg(globalQuakes->size());
        thrust::device_vector<thrust::pair<int, int> > dConnect(_connect->size());
        thrust::copy(_connect->begin(), _connect->end(), dConnect.begin());
        thrust::copy(data->begin(), data->end(), input.begin());
        thrust::copy(globalQuakes->begin(), globalQuakes->end(), gQuakeAvg.begin());

        int blocksPerGrid; //the blocksize defined by the configurator
        int threadsblock = 512; // the actual grid size needed

        _NNParams[8] = _memVirtualizer._DGenetics.size()/(_NNParams[7]);
        blocksPerGrid=(_NNParams[9]+threadsblock-1)/threadsblock;
        Net<<<blocksPerGrid, threadsblock>>>(_memVirtualizer.genetics(),
                                             convertToKernel(_NNParams),
                                             convertToKernel(gQuakeAvg),
                                             convertToKernel(input),
                                             convertToKernel(_siteData),
                                             convertToKernel(_answers),
                                             convertToKernel(retVec),
                                             convertToKernel(dConnect),
                                             Kp,
                                             _sampleRate,
                                             _numofSites,
                                             hour);
        hipDeviceSynchronize();
        thrust::copy(retVec.begin(), retVec.end(), ret);
    }
    else{
        std::cerr<<"entered not training version.."<<std::endl;
        typedef std::vector<thrust::pair<int, int> > connectPairMatrix;
        //replace this later
        _best.resize(45);
        for(std::vector<double>::iterator it = _best.begin(); it != _best.end(); ++it){
            *it = 1/rand();
        }
        std::cerr<<"example best vector has been set."<<std::endl;
        double CommunityLat = 0;
        double CommunityLon = 0;
        std::vector<double> When(_numofSites);
        std::vector<double> HowCertain(_numofSites,0);
        std::vector<double> CommunityMag(_numofSites, 1); //give all sites equal mag to start, this value is [0,1]
        std::cerr<<"all output vectors created and initialized."<<std::endl;
        for(int step=0; step<3600*_sampleRate; step++){
            std::cerr<<"entering step #"<<step<<std::endl;
            for(int j=0; j<_numofSites; j++){ //sitesWeighted Lat/Lon values are determined based on all previous sites mag output value.
                CommunityLat += _siteData[j*2]*CommunityMag[j];
                CommunityLon += _siteData[j*2+1]*CommunityMag[j];
            }
            CommunityLat = CommunityLat/_numofSites;
            CommunityLon = CommunityLon/_numofSites;

            for(int j=0; j<_numofSites; j++){ // each site is run independently of others, but shares an output from the previous step
                std::cerr<<"entering site #"<<j<<std::endl;
                double latSite = _siteData[j*2];
                double lonSite = _siteData[j*2+1];
                double avgLatGQuake = globalQuakes->at(0);
                double avgLonGQuake = globalQuakes->at(1);
                //                double avgDepthGQuake = globalQuakes->at(2); don't think I care about depth that much.
                double GQuakeAvgMag = globalQuakes->at(3);
                double GQuakeAvgdist = distCalc(latSite, lonSite, avgLatGQuake, avgLonGQuake);
                double GQuakeAvgBearing = bearingCalc(latSite, lonSite, avgLatGQuake, avgLonGQuake);
                double CommunityDist = distCalc(latSite, lonSite, CommunityLat, CommunityLon);
                double CommunityBearing = bearingCalc(latSite, lonSite, CommunityLat, CommunityLon);
                std::vector<int> input;
                std::vector<double> hidden, outputs, mem, memGateOut, memGateIn, memGateForget;
                //replace these with real connections, num of inputs, and num of hidden & memory neurons (mem neurons probably accurate)
                input.resize(_NNParams[3], 0); // number of inputs is 9.
                hidden.resize(_NNParams[4], 0); // for practice sake, lets say each input has its own neuron (might be true!)
                mem.resize(_NNParams[5], 0); // stores the input if gate is high
                memGateOut.resize(_NNParams[5], 0); //connects to the input layer and the memN associated with input, if 1 it sends up stream and deletes, if low it keeps.
                memGateIn.resize(_NNParams[5], 0);
                memGateForget.resize(_NNParams[5], 0);
                outputs.resize(_NNParams[6], 0); /* 3 outputs, 1 with an hour in the future when the earthquake will hit,
                    1 with the porbability of that earthquake happening (between [0,1]) and 1 with the sites magnitude (for community feedback) */
                int n =0;
                int startOfInput = 0;
                int startOfHidden = startOfInput +_NNParams[3];
                int startOfMem = startOfHidden + _NNParams[4];
                int startOfMemGateIn = startOfMem + _NNParams[5];
                int startOfMemGateOut = startOfMemGateIn + _NNParams[5];
                int startOfMemGateForget = startOfMemGateOut + _NNParams[5];
                int startOfOutput = startOfMemGateForget + _NNParams[5];
                input[0] = 1/data->at(3600*_sampleRate*j*3 + 1*(3600*_sampleRate)+step);
                input[1] = 1/data->at(3600*_sampleRate*j*3 + 2*(3600*_sampleRate)+step);
                input[2] = 1/data->at(3600*_sampleRate*j*3 + 3*(3600*_sampleRate)+step);
                input[3] = 1/GQuakeAvgdist;
                input[4] = 1/GQuakeAvgBearing;
                input[5] = 1/GQuakeAvgMag;
                input[6] = 1/Kp;
                input[7] = 1/CommunityDist;
                input[8] = 1/CommunityBearing;
                //lets reset all neuron values for this new timestep (except memory neurons)
                for(int gate=0; gate<_NNParams[5]; gate++){
                    memGateIn[gate] = 0;
                    memGateOut[gate] = 0;
                    memGateForget[gate] = 0;
                }
                for(int hid=0; hid<_NNParams[4]; hid++){
                    hidden[hid] = 0;
                }
                for(int out=0; out<_NNParams[6]; out++){
                    outputs[out] = 0;
                }

                //now that everything that should be zeroed is zeroed, lets start the network.
                //mem gates & LSTM nodes --
                for(int gate = 0; gate<_NNParams[5]; gate++){//calculate memory gate node values, you can connect inputs & hidden neurons to them.
                    for(connectPairMatrix::iterator it = _connect->begin(); it!= _connect->end(); ++it){//for memGateIn
                        if(it->second == gate+startOfMemGateIn && it->second < startOfHidden){ //for inputs
                            memGateIn[gate] += input[it->first-startOfInput]*_best[n++]; // memGateIn vect starts at 0
                        }
                        else if(it->second == gate+startOfMemGateIn && it->second >startOfHidden && it->second <startOfMem){//for hidden neurons
                            memGateIn[gate] += hidden[it->first-startOfHidden]*_best[n++];
                        }
                    }
                    for(connectPairMatrix::iterator it = _connect->begin(); it!= _connect->end(); ++it){//for memGateOut
                        if(it->second == gate+startOfMemGateOut && it->second < startOfHidden){//for inputs
                            memGateOut[gate] += input[it->first-startOfInput]*_best[n++];
                        }
                        else if(it->second == gate+startOfMemGateOut && it->second >startOfHidden && it->second <startOfMem){//for hidden neurons
                            memGateOut[gate] += hidden[it->first-startOfHidden]*_best[n++];
                        }
                    }
                    for(connectPairMatrix::iterator it = _connect->begin(); it!= _connect->end(); ++it){//for  memGateForget
                        if(it->second == gate+startOfMemGateForget && it->second < startOfHidden){//for inputs
                            memGateForget[gate] += input[it->first - startOfInput]*_best[n++];
                        }
                        else if(it->second == gate+startOfMemGateForget && it->second >startOfHidden && it->second <startOfMem){//for hidden neurons
                            memGateForget[gate] += hidden[it->first-startOfHidden]*_best[n++];
                        }
                    }
                    memGateIn[gate] = ActFunc(memGateIn[gate]);
                    memGateOut[gate] = ActFunc(memGateOut[gate]);
                    memGateForget[gate] = ActFunc(memGateForget[gate]);
                }
                //since we calculated the values for memGateIn and memGateOut, and MemGateForget..
                for (int gate = 0; gate<_NNParams[5]; gate++){ // if memGateIn is greater than 0.3, then let mem = the sum inputs attached to memGateIn
                    if(memGateIn[gate] > 0.5){ //gate -startOfMemGateIn = [0, num of mem neurons]
                        for(connectPairMatrix::iterator it = _connect->begin(); it!= _connect->end(); ++it){
                            if(it->second == gate+startOfMemGateIn && it->first < gate+startOfHidden){//only pass inputs
                                mem[gate] += input[it->first-startOfInput]; // no weights attached, but the old value stored here is not removed.
                            }
                        }
                    }
                    if(memGateForget[gate] > 0.5){// if memGateForget is greater than 0.5, then tell mem to forget
                        mem[gate] = 0;
                    }
                    //if memGateForget fires, then memGateOut will output nothing.
                    if(memGateOut[gate] > 0.5){//if memGateOut is greater than 0.3, let the nodes mem is connected to recieve mem
                        for(connectPairMatrix::iterator it = _connect->begin(); it!= _connect->end(); ++it){
                            if(it->first == gate+startOfMem){// since mem node: memIn node : memOut node = 1:1:1, we can do this.
                                hidden[it->second-startOfHidden] += mem[gate];
                            }
                        }
                    }
                }

                // hidden neuron nodes --
                for(int hid=0; hid<_NNParams[4]; hid++){ // for all hidden neurons at layer 1, lets sum the inputs, the memory values were already added.
                    for(connectPairMatrix::iterator it = _connect->begin(); it!= _connect->end(); ++it){ // Add the inputs to the hidden neurons
                        if(it->second == hid+startOfHidden && it->first < startOfHidden && it->first >= startOfInput){ // if an input connects with this hidden neuron
                            hidden[hid] += input[it->first]*_best[n++];
                        }
                    }
                    for(connectPairMatrix::iterator it = _connect->begin(); it!= _connect->end(); ++it){//add other hidden neuron inputs to each hidden neuron (if applicable)
                        if(it->second == hid+startOfHidden && it->first < startOfMem && it->first >= startOfHidden){
                            hidden[hid] += hidden[it->first-startOfHidden]*_best[n++];
                        }
                    }
                    hidden[hid] += 1*_best[n++]; // add bias
                    hidden[hid] = ActFunc(hidden[hid]); // then squash it.
                }
                //output nodes --

                for(int out =0; out<_NNParams[6]; out++){// add hidden neurons to the output nodes
                    for(connectPairMatrix::iterator it = _connect->begin(); it!= _connect->end(); ++it){
                        if(it->second == out+startOfOutput){
                            outputs[out] += hidden[it->first-startOfHidden]*_best[n++];
                        }
                    }
                    outputs[out] += 1*_best[n++]; // add bias
                    outputs[out] = ActFunc(outputs[out]);// then squash it.
                }


                When[j] += 1/outputs[0]; //return when back to an integer value (adjust to fit within boundaries)
                HowCertain[j] += outputs[1];
                CommunityMag[j] =  outputs[2]; // set the next sets communityMag = output #3.
            }
        }
        for(int j=0; j<_numofSites; j++){ // each site has its own when and howcertain vector
            When[j] = When[j]/3600*_sampleRate;
            HowCertain[j] = HowCertain[j]/3600*_sampleRate;
        }
        //all done, lets output the return matrix.
        //since right now were using a point value for when & how certain (only one output per site),
        //we're going to approximate using a normal distribution around when with a sigma of howCertain, over the whole array from T=currentHour [T, 2160]
        for(int h=hour; h<2160; h++){
            for(int j=0; j<_numofSites; j++){
                ret[h*_numofSites+j] = 1/(1/HowCertain[j]*sqrt(2*M_PI))*exp(-pow(h-When[j], 2)/(2*pow(1/HowCertain[j], 2))); // normal distribution with a mu of When and a sigma of 1/HowCertain
            }
        }
    }
}
