#include "hip/hip_runtime.h"
#include "network.h"
#include <iostream>
#include <fstream>
#include <thrust/device_vector.h>
#include <thrust/random.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>
#include <ctime>
#include <hip/hip_runtime.h>
#include <vector>
#include <hip/hip_runtime.h>
#include <tinyxml2.h>

//macros
//xml error message handling
#ifndef XMLCheckResult
#define XMLCheckResult(a_eResult) if (a_eResult != tinyxml2::XML_SUCCESS) { printf("Error: %i\n", a_eResult);  exit(a_eResult); }
#endif
//cuda error message handling
#define CUDA_SAFE_CALL(call)                                          \
    do {                                                                  \
    hipError_t err = call;                                           \
    if (hipSuccess != err) {                                         \
    fprintf (stderr, "Cuda error in file '%s' in line %i : %s.\n",\
    __FILE__, __LINE__, hipGetErrorString(err) );       \
    exit(EXIT_FAILURE);                                           \
    }                                                                 \
    } while (0)


template <typename T>
__global__ void genWeights( DataArray<T> ref, long in, int nRegWeights, int indLength){
    long idx = blockDim.x*blockIdx.x + threadIdx.x;
    long seed= idx+in;
    thrust::default_random_engine randEng;
    for(int i=0; i<nRegWeights; i++){
        thrust::uniform_real_distribution<double> uniDist(0,1);
        randEng.discard(seed);
        ref._array[idx*indLength+i] = uniDist(randEng);
    }
}




NetworkGenetic::NetworkGenetic(const int &numInNeurons, const int &numHiddenNeurons, const int &numMemoryNeurons,
                               const int &numOutNeurons, const int &numHiddenLayers,  const thrust::pair<int, int> &connections){
    this->_NNParams.resize(15, 0); // room to grow
    _NNParams[1] = numInNeurons + numHiddenNeurons + numMemoryNeurons + numOutNeurons;
    _NNParams[2] = numInNeurons + numHiddenNeurons + numOutNeurons;
    _NNParams[3] = numInNeurons;
    _NNParams[4] = numHiddenNeurons;
    _NNParams[5] = numMemoryNeurons;
    _NNParams[6] = numOutNeurons;
    _NNParams[7] = numHiddenLayers;
    _connections = connections;
}

void NetworkGenetic::initializeWeights(){
    hipEvent_t start, stop;
    CUDA_SAFE_CALL (hipEventCreate(&start));
    CUDA_SAFE_CALL (hipEventCreate(&stop));
    int blocksize; //the blocksize defined by the configurator
    int minGridSize; //the minimum grid size needed to achive max occupancy
    int gridSize; // the actual grid size needed
    int individualSize = _NNParams[1]+1;//contains all neruons, plus 1 for fitness vals
    _NNParams[8] = _DGeneticsData.size()/(individualSize);
    std::cout<<"initial population: "<<_NNParams[8]<<std::endl;
    long time = std::clock();
    hipEventRecord(start, 0);
    hipDeviceSynchronize();
    CUDA_SAFE_CALL (hipOccupancyMaxPotentialBlockSize( &minGridSize, &blocksize, (void*)genWeights<double>, 0, _NNParams[8]));
    gridSize = (_NNParams[8] + blocksize -1)/blocksize;
    genWeights<double><<<gridSize, blocksize>>>(convertToKernel<double>(_DGeneticsData), time, _NNParams[2], individualSize);
    hipDeviceSynchronize();
    float miliseconds = 0;
    CUDA_SAFE_CALL (hipEventRecord(stop, 0));
    hipDeviceSynchronize();
    CUDA_SAFE_CALL (hipEventElapsedTime(&miliseconds, start, stop));
    std::cout<<"weight generation: total compute time: "<<miliseconds<<" ms"<<std::endl;
    std::cout<<"effective bandwidth (GB/s) : "<<(_DGeneticsData.size()*8)/((miliseconds/1000)*1e9)<<std::endl;
}



void NetworkGenetic::importSitesData(std::string siteInfo){
    int dataSet, SLEN;
    tinyxml2::XMLDocument doc;
    _DInitData.clear(); //empty any previous data located in array, both are small enough to be of no consquence
    _DInitData.shrink_to_fit();
    _DSitesData.clear();
    _DSitesData.shrink_to_fit();
    doc.LoadFile(siteInfo.c_str());
    tinyxml2::XMLNode * pRoot = doc.FirstChild();
    if(pRoot == NULL) exit(tinyxml2::XML_ERROR_FILE_READ_ERROR);
    tinyxml2::XMLElement * pElement = pRoot->NextSiblingElement("Sites");
    if(pElement == NULL) exit(tinyxml2::XML_ERROR_PARSING_ELEMENT);
    tinyxml2::XMLError eResult = pElement->QueryIntAttribute("data_set", &dataSet);
    XMLCheckResult(eResult);

    eResult = pElement->QueryIntAttribute("num_sites", &SLEN);
    XMLCheckResult(eResult);
    _DInitData.push_back(SLEN);
    _DInitData.push_back(dataSet);
    tinyxml2::XMLElement *SitesList = pRoot->NextSiblingElement("Site");

    while(SitesList != NULL){
        int sampleData;
        double longitude, latitude;
        eResult = SitesList->QueryIntAttribute("sample_rate", &sampleData);
        XMLCheckResult(eResult);
        _DInitData.push_back(sampleData);
        eResult = SitesList->QueryDoubleAttribute("latitude", &latitude);
        XMLCheckResult(eResult);
        _DSitesData.push_back(latitude);
        eResult = SitesList->QueryDoubleAttribute("longitude", &longitude);
        XMLCheckResult(eResult);
        _DSitesData.push_back(longitude);
        SitesList = SitesList->NextSiblingElement("Site");
    }
}

void NetworkGenetic::importKpData(std::string Kp){
    tinyxml2::XMLDocument doc;
    tinyxml2::XMLError eResult;
    _DKpIndex.clear();
    _DKpIndex.shrink_to_fit();
    doc.LoadFile(Kp.c_str());
    tinyxml2::XMLNode *pRoot = doc.FirstChild();
    if(pRoot == NULL) exit(tinyxml2::XML_ERROR_FILE_READ_ERROR);
    tinyxml2::XMLElement * pElement = pRoot->NextSiblingElement("Kp");
    if(pElement == NULL) exit(tinyxml2::XML_ERROR_PARSING_ELEMENT);
    tinyxml2::XMLElement * KpList = pElement->FirstChildElement("Kp_hr");
    while(KpList != NULL){
        int seconds;
        float magnitude;
        eResult = KpList->QueryIntAttribute("secs", &seconds);
        XMLCheckResult(eResult);
        _DKpIndex.push_back(seconds);
        eResult = KpList->QueryFloatText(&magnitude);
        XMLCheckResult(eResult);
        _DKpIndex.push_back(magnitude);

        KpList = KpList->NextSiblingElement("Kp_hr");
    }

}


void NetworkGenetic::allocateHostAndGPUObjects(unsigned int hostMemory, unsigned int deviceMemory,
                                               std::map<const std::string, float> pHostRam,  std::map<const std::string, float> pDeviceRam){
    unsigned int hostGeneticsAlloc = hostMemory*pHostRam.at("genetics")/sizeof(double); //since these are doubles, divide bytes by 8
    unsigned int hostTrainingAlloc = hostMemory*pHostRam.at("input & training")/(sizeof(double)+2);//half for training, half for input I think?
    unsigned int hostInputsAlloc = hostMemory*pHostRam.at("input & training")/(sizeof(float)+2); // their either floats or ints, same amount of bytes.
    unsigned int deviceGeneticsAlloc = deviceMemory*pDeviceRam.at("genetics")/sizeof(double);
    unsigned int deviceTrainingAlloc = deviceMemory*pDeviceRam.at("input & training")/(sizeof(double)+2);
    unsigned int deviceInputsAlloc = deviceMemory*pDeviceRam.at("input & training")/(sizeof(double)+2);
    //initialize all vectors except ones initialized by xml docs (small enough to fit outside of the memory container and on the device)

    this->_HGeneticsData.resize(hostGeneticsAlloc);
    this->_HTrainingData.resize(hostTrainingAlloc);
    this->_HInputData.resize(hostInputsAlloc);
    this->_DGeneticsData.resize(deviceGeneticsAlloc);
    this->_DTrainingData.resize(deviceTrainingAlloc);
    this->_DInputData.resize(deviceInputsAlloc);
}

void NetworkGenetic::errorFunc(){
    //    hipEvent_t start, stop;
    //    hipEventCreate(&start);
    //    hipEventCreate(&stop);
    //    int blocksize; //the blocksize defined by the configurator
    //    int minGridSize; //the minimum grid size needed to achive max occupancy
    //    int gridSize; // the actual grid size needed
}
