#include "network.h"
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/random.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/system_error.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <hip/hip_runtime_api.h>

struct genRand: thrust::unary_function<Individual, int>{

    int numWeights;

    genRand(int _numWeights) : numWeights(_numWeights){}

    __host__ __device__
    Individual operator()(Individual n) const{
        unsigned int idx= threadIdx.x*blockDim.x;

        for(int i=0; i<numWeights; i++){
            idx = idx +i;
            thrust::default_random_engine randEng;
            thrust::uniform_real_distribution<float> uniDist(0,1);
            randEng.discard(idx);
            n._weights[i] =  uniDist(randEng);
        }
        return n;
    }};


NetworkGenetic::NetworkGenetic(){}

NetworkGenetic::NetworkGenetic(const int &numInNeurons, const int &numHiddenNeurons,
                               const int &numOutNeurons, std::map<const int, int> &connections){
    _constantNNParams.push_back(numInNeurons);
    _constantNNParams.push_back(numHiddenNeurons);
    _constantNNParams.push_back(numOutNeurons);
    _neuronsTotalNum = numInNeurons + numHiddenNeurons + numOutNeurons;
    _connections = connections;
}

bool NetworkGenetic::generatePop(int popsize){
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    thrust::device_vector<Individual> testing;
    for(int i=0; i<popsize; i++){
        Individual obj(_neuronsTotalNum);
        testing.push_back(obj);
    }

    try{
        hipEventRecord(start);
        thrust::transform(testing.begin(),
                          testing.end(), testing.begin(), genRand(_neuronsTotalNum));
        hipDeviceSynchronize();
    }
    catch(thrust::system_error &err){
        std::cerr<<"error transforming: "<<err.what()<<std::endl;
        return false;
    }
    hipEventRecord(stop);
    float miliseconds = 0;
    hipEventElapsedTime(&miliseconds, start, stop);
    std::cout<<miliseconds<<" ms"<<std::endl;

    return true;
}
