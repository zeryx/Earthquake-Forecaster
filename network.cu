#include "hip/hip_runtime.h"
﻿#include "network.h"
#include "getsys.h"
#include <thrust/device_vector.h>
#include <thrust/random.h>
#include <thrust/system_error.h>
#include <thrust/host_vector.h>
#include <fstream>
#include <sstream>
#include <ostream>
#include <utility>
#include <vector>
#include <ctime>
#include <assert.h>

//macros
//cuda error message handling
#ifndef CUDA_SAFE_CALL
#define CUDA_SAFE_CALL(call) do{hipError_t err = call; if (hipSuccess != err) {fprintf (stderr, "Cuda error in file '%s' in line %i : %s.\n",__FILE__, __LINE__, hipGetErrorString(err) ); exit(EXIT_FAILURE);}} while (0)
#endif
//neural functions
__host__ __device__ inline double sind(double x)
{
    double ret= sin(x * M_PI / 180);
    return ret;
}

__host__ __device__ inline double cosd(double x)
{
    return cos(x * M_PI / 180);
}
__host__ __device__ inline double distCalc(double lat1, double lon1, double lat2, double lon2)
{
    double earthRad = 6371.01;
    double deltalon = abs(lon1 - lon2);
    if(deltalon > 180)
        deltalon = 360 - deltalon;
    double ret = earthRad * atan2( sqrt( pow( cosd(lat1) * sind(deltalon), 2) +
                                         pow( cosd(lat2) * sind(lat1) - sind(lat2) * cosd(lat1) * cosd(deltalon), 2) ),
                                   sind(lat2) * sind(lat1) + cosd(lat2) * cosd(lat1) * cosd(deltalon));
    return ret;
}

__host__ __device__ inline double bearingCalc(double lat1, double lon1, double lat2, double lon2)
{
    double dLon = (lon2 - lon1);

    double y = sin(dLon) * cos(lat2);
    double x = cos(lat1) * sin(lat2) - sin(lat1) * cos(lat2) * cos(dLon);

    double brng = atan2(y, x);

    brng = brng*M_PI/180;
    brng += 360;
    while(brng>= 360)
        brng -= 360;
    brng = 360 - brng;

    return brng;
}

__host__ __device__ inline double ActFunc(double x)
{
    double ret = 1+1/exp(-x);
    return ret;
}
__host__ __device__ inline double normalize(double x, double mean, double stdev)
{
    double ret = (abs(x-mean))/stdev*2;
    return ret;
}

__host__ __device__ inline double shift(double x, double max, double min)
{
    double ret = (x-min)/(max-min);
    return ret;
}

__global__ void genWeights( unifiedArray<double> ref, long in, dataArray<int> params)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int ind = idx*params.array[7];
    thrust::minstd_rand0 randEng;
    randEng.seed(idx);
    int seed = idx+ref.size*in;
    thrust::uniform_real_distribution<double> uniDist(0,1);
    for(int i=0; i<params.array[2]; i++){
        randEng.discard(seed+1);
        ref.array[ind+i] = uniDist(randEng);
    }
}

__global__ void Net(unifiedArray<double> weights, dataArray<int> params,
                    dataArray<double> globalQuakes, dataArray<int> inputVal, dataArray<double> siteData,
                    dataArray<double> answers, dataArray<thrust::pair<int, int> > connections,
                    double Kp, int sampleRate,int numOfSites, int hour,
                    double meanCh1, double meanCh2, double meanCh3, double stdCh1, double stdCh2, double stdCh3)
{

    int idx = blockIdx.x * blockDim.x + threadIdx.x; // for each thread is one individual
    int ind = idx*params.array[7];
    typedef thrust::device_ptr<thrust::pair<int, int> >  connectPairMatrix;
    double CommunityLat = 0;
    double CommunityLon = 0;
    double *When = new double[numOfSites];
    double *HowCertain = new double[numOfSites];
    double *CommunityMag = new double[numOfSites]; //give all sites equal mag to start, this value is [0,1]

    for(int step=0; step<3600*sampleRate; step++){

        for(int j=0; j<sampleRate; j++){//sitesWeighted Lat/Lon values are determined based on all previous sites mag output value.
            CommunityLat += siteData.array[j*2]*CommunityMag[j];
            CommunityLon += siteData.array[j*2+1]*CommunityMag[j];
        }
        CommunityLat = CommunityLat/numOfSites;
        CommunityLon = CommunityLon/numOfSites;

        for(int j=0; j<numOfSites; j++){ //each site is run independently of others, but shares an output from the previous step
            double latSite = siteData.array[j*2];
            double lonSite = siteData.array[j*2+1];
            double avgLatGQuake = globalQuakes.array[0];
            double avgLonGQuake = globalQuakes.array[1];
            //double avgDepthGQuake = globalQuakes.array[2); don't think I care about depth that much.
            double GQuakeAvgMag = globalQuakes.array[3];
            double GQuakeAvgdist = distCalc(latSite, lonSite, avgLatGQuake, avgLonGQuake);
            double GQuakeAvgBearing = bearingCalc(latSite, lonSite, avgLatGQuake, avgLonGQuake);
            double CommunityDist = distCalc(latSite, lonSite, CommunityLat, CommunityLon);
            double CommunityBearing = bearingCalc(latSite, lonSite, CommunityLat, CommunityLon);
            /* 3 outputs, 1 with an hour in the future when the earthquake will hit,
                1 with the porbability of that earthquake happening (between [0,1]) and 1 with the sites magnitude (for community feedback) */
            int n =0;
            int startOfInput = 0;
            int startOfHidden = startOfInput +params.array[3];
            int startOfMem = startOfHidden + params.array[4];
            int startOfMemGateIn = startOfMem + params.array[5];
            int startOfMemGateOut = startOfMemGateIn + params.array[5];
            int startOfMemGateForget = startOfMemGateOut + params.array[5];
            int startOfOutput = startOfMemGateForget + params.array[5];
            // the weights array carries the neuron scratch space used for the net kernel, I'd like to replace this and reduce the memory allocation asap.
            double *input = &weights.array[startOfInput]; // number of inputs is 9.
            double *hidden = &weights.array[startOfHidden]; // for practice sake, lets say each input has its own neuron (might be true!)
            double *mem = &weights.array[startOfMem]; // stores the input if gate is high
            double *memGateIn = &weights.array[startOfMemGateIn]; //connects to the input layer and the memN associated with input, if 1 it sends up stream and deletes, if low it keeps.
            double *memGateOut = &weights.array[startOfMemGateOut];
            double *memGateForget = &weights.array[startOfMemGateForget];
            double *outputs = &weights.array[startOfOutput];

            input[0] = normalize(inputVal.array[(3600*sampleRate*j*3 + 1*(3600*sampleRate)+step)], meanCh1, stdCh1);//channel 1
            input[1] = normalize(inputVal.array[(3600*sampleRate*j*3 + 2*(3600*sampleRate)+step)], meanCh2, stdCh2);//channel 2
            input[2] = normalize(inputVal.array[(3600*sampleRate*j*3 + 3*(3600*sampleRate)+step)], meanCh3, stdCh3);//channel 3
            input[3] = shift(GQuakeAvgdist, 40075.1, 0);
            input[4] = shift(GQuakeAvgBearing, 360, 0);
            input[5] = shift(GQuakeAvgMag, 9.5, 0);
            input[6] = shift(Kp, 10, 0);
            input[7] = shift(CommunityDist,40075.1/2, 0);
            input[8] = shift(CommunityBearing, 360, 0);
            //lets reset all neuron values for this new timestep (except memory neurons)
            for(int gate=0; gate<params.array[5]; gate++){
                memGateIn[gate] = 0;
                memGateOut[gate] = 0;
                memGateForget[gate] = 0;
            }
            for(int hid=0; hid<params.array[4]; hid++){
                hidden[hid] = 0;
            }
            for(int out=0; out<params.array[6]; out++){
                outputs[out] = 0;
            }

            //now that everything that should be zeroed is zeroed, lets start the network.
            //mem gates & LSTM nodes --
            for(int gate = 0; gate<params.array[5]; gate++){//calculate memory gate node values, you can connect inputs & hidden neurons to them.
                for(connectPairMatrix it = connections.array; it!= connections.array+connections.size; ++it){//for memGateIn
                    thrust::pair<int, int>itr = static_cast<thrust::pair<int, int> >(*it); // this needs to be created to use the iterator it correctly.
                    if(itr.second == gate+startOfMemGateIn && itr.second < startOfHidden){ //for inputs
                        memGateIn[gate] += input[itr.first-startOfInput]*weights.array[ind + n++]; // memGateIn vect starts at 0
                    }
                    else if(itr.second == gate+startOfMemGateIn && itr.second >startOfHidden && itr.second <startOfMem){//for hidden neurons
                        memGateIn[gate] += hidden[itr.first-startOfHidden]*weights.array[ind + n++];
                    }
                }
                for(connectPairMatrix it = connections.array; it!= connections.array+connections.size; ++it){//for memGateOut
                    thrust::pair<int, int>itr = static_cast<thrust::pair<int, int> >(*it);
                    if(itr.second == gate+startOfMemGateOut && itr.second < startOfHidden){//for inputs
                        memGateOut[gate] += input[itr.first-startOfInput]*weights.array[ind + n++];
                    }
                    else if(itr.second == gate+startOfMemGateOut && itr.second >startOfHidden && itr.second <startOfMem){//for hidden neurons
                        memGateOut[gate] += hidden[itr.first-startOfHidden]*weights.array[ind + n++];
                    }
                }
                for(connectPairMatrix it = connections.array; it!= connections.array+connections.size; ++it){//for  memGateForget
                    thrust::pair<int, int>itr = static_cast<thrust::pair<int, int> >(*it);
                    if(itr.second == gate+startOfMemGateForget && itr.second < startOfHidden){//for inputs
                        memGateForget[gate] += input[itr.first - startOfInput]*weights.array[ind + n++];
                    }
                    else if(itr.second == gate+startOfMemGateForget && itr.second >startOfHidden && itr.second <startOfMem){//for hidden neurons
                        memGateForget[gate] += hidden[itr.first-startOfHidden]*weights.array[ind + n++];
                    }
                }
                memGateIn[gate] = ActFunc(memGateIn[gate]);
                memGateOut[gate] = ActFunc(memGateOut[gate]);
                memGateForget[gate] = ActFunc(memGateForget[gate]);
            }
            //since we calculated the values for memGateIn and memGateOut, and MemGateForget..
            for (int gate = 0; gate<params.array[5]; gate++){ // if memGateIn is greater than 0.3, then let mem = the sum inputs attached to memGateIn
                if(memGateIn[gate] > 0.5){ //gate -startOfMemGateIn = [0, num of mem neurons]
                    for(connectPairMatrix it = connections.array; it!= connections.array+connections.size; ++it){
                        thrust::pair<int, int>itr = static_cast<thrust::pair<int, int> >(*it);
                        if(itr.second == gate+startOfMemGateIn && itr.first < gate+startOfHidden){//only pass inputs
                            mem[gate] += input[itr.first-startOfInput]; // no weights attached, but the old value stored here is not removed.
                        }
                    }
                }
                if(memGateForget[gate] > 0.5){// if memGateForget is greater than 0.5, then tell mem to forget
                    mem[gate] = 0;
                }
                //if memGateForget fires, then memGateOut will output nothing.
                if(memGateOut[gate] > 0.5){//if memGateOut is greater than 0.3, let the nodes mem is connected to recieve mem
                    for(connectPairMatrix it = connections.array; it!= connections.array+connections.size; ++it){
                        thrust::pair<int, int>itr = static_cast<thrust::pair<int, int> >(*it);
                        if(itr.first == gate+startOfMem){// since mem node: memIn node : memOut node = 1:1:1, we can do this.
                            hidden[itr.second-startOfHidden] += mem[gate];
                        }
                    }
                }
            }

            // hidden neuron nodes --
            for(int hid=0; hid<params.array[4]; hid++){ // for all hidden neurons at layer 1, lets sum the inputs, the memory values were already added.
                for(connectPairMatrix it = connections.array; it!= connections.array+connections.size; ++it){ // Add the inputs to the hidden neurons
                    thrust::pair<int, int>itr = static_cast<thrust::pair<int, int> >(*it);
                    if(itr.second == hid+startOfHidden && itr.first < startOfHidden && itr.first >= startOfInput){ // if an input connects with this hidden neuron
                        hidden[hid] += input[itr.first]*weights.array[ind + n++];
                    }
                }
                for(connectPairMatrix it = connections.array; it!= connections.array+connections.size; ++it){//add other hidden neuron inputs to each hidden neuron (if applicable)
                    thrust::pair<int, int>itr = static_cast<thrust::pair<int, int> >(*it);
                    if(itr.second == hid+startOfHidden && itr.first < startOfMem && itr.first >= startOfHidden){
                        hidden[hid] += hidden[itr.first-startOfHidden]*weights.array[ind + n++];
                    }
                }
                hidden[hid] += 1*weights.array[ind + n++]; // add bias
                hidden[hid] = ActFunc(hidden[hid]); // then squash itr.
            }
            //output nodes --

            for(int out =0; out<params.array[6]; out++){// add hidden neurons to the output nodes
                for(connectPairMatrix it = connections.array; it!= connections.array+connections.size; ++it){
                    thrust::pair<int, int>itr = static_cast<thrust::pair<int, int> >(*it);
                    if(itr.second == out+startOfOutput){
                        outputs[out] += hidden[itr.first-startOfHidden]*weights.array[ind + n++];
                    }
                }
                outputs[out] += 1*weights.array[ind + n++]; // add bias
                outputs[out] = ActFunc(outputs[out]);// then squash itr.
            }

            When[j] += outputs[0]*((2160-hour)-hour)+2160-hour; // nv = ((ov - omin)*(nmax-nmin) / (omax - omin))+nmin
            HowCertain[j] += outputs[1];
            CommunityMag[j] =  outputs[2]; // set the next sets communityMag = output #3.
        }
    }
    for(int j=0; j<numOfSites; j++){ // now lets get the average when and howcertain values.
        When[j] = When[j]/3600*sampleRate;
        HowCertain[j] = HowCertain[j]/3600*sampleRate;
    }
    // calculate performance for this individual - score = 1/(abs(whenGuess-whenReal)*distToQuake), for whenGuess = when[j] where HowCertain is max for set.
    //distToQuake is from the current sites parameters, it emphasizes higher scores for the closest site, a smaller distance is a higher score.
    int maxCertainty=0;
    double whenGuess=0;
    double latSite;
    double lonSite;
    for(int j=0; j<numOfSites; j++){
        if(HowCertain[j] > maxCertainty){
            whenGuess = When[j];
            latSite = siteData.array[j*2];
            lonSite = siteData.array[j*2+1];
        }
    }
    double SiteToQuakeDist = distCalc(latSite, lonSite, answers.array[2], answers.array[3]); // [2] is latitude, [3] is longitude.
    double fitness = 1/(abs(whenGuess - answers.array[1]-hour)*SiteToQuakeDist);//larger is better, negative numbers are impossible.
    weights.array[ind + params.array[2]+2] = fitness; // set the fitness number for the individual.
}

__global__ void reduce_by_block(unifiedArray<double> weights,
                                dataArray<double> per_block_results,
                                dataArray<int> params)
{
    extern __shared__ float sdata[];

    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int fit = idx*params.array[7]+params.array[2]+2;

    // load input into __shared__ memory
    float x = 0;
    if(idx < params.array[8])
    {
        x = weights.array[fit];
    }
    sdata[threadIdx.x] = x;
    __syncthreads();

    // contiguous range pattern
    for(int offset = blockDim.x / 2;
        offset > 0;
        offset >>= 1)
    {
        if(threadIdx.x < offset)
        {
            // add a partial sum upstream to our own
            sdata[threadIdx.x] += sdata[threadIdx.x + offset];
        }
        // wait until all threads in the block have
        // updated their partial sums
        __syncthreads();
    }

    // thread 0 writes the final result
    if(threadIdx.x == 0)
    {
        per_block_results.array[blockIdx.x] = sdata[0];
    }
}

__global__ void swapMemory(unifiedArray<double> device, unifiedArray<double>host, int offset){//swap device and host memory in place.
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    double tmp;
    tmp = device.array[idx];
    device.array[idx] = host.array[idx+offset];
    host.array[idx+offset] = tmp;
}

NetworkGenetic::NetworkGenetic(const int &numInputNodes, const int &numHiddenNeurons, const int &numMemoryNeurons,
                               const int &numOutNeurons, const int &numWeights, std::vector< thrust::pair<int, int> >&connections){
    this->_NNParams.resize(15, 0); // room to grow
    _NNParams[1] = numInputNodes + numHiddenNeurons + numMemoryNeurons*4 + numOutNeurons; //memory neurons each ahve a rmemeber, forget, and push forward gate neuron.
    _NNParams[2] = numWeights;
    _NNParams[3] = numInputNodes;
    _NNParams[4] = numHiddenNeurons;
    _NNParams[5] = numMemoryNeurons;
    _NNParams[6] = numOutNeurons;
    _NNParams[7] = _NNParams[2] + _NNParams[1] + 1 + 1; // plus 1 for fitness, plus 1 for community output composite vector
    _connect = &connections;
}

void NetworkGenetic::initializeWeights(){
    int gridSize; //the blocksize defined by the configurator
    int blockSize = 512; // number of blocks in the grid
    int seedItr = 0;
    int global_offset=0;
    for(int n=0; n<_numOfStreams; n++){
        if(n <_numOfStreams-1){
            long seed = std::clock() + std::clock()*seedItr++;
            gridSize=(_streamSize/_NNParams[7])/(blockSize);
            std::cerr<<"number of blocks is: "<<gridSize<<std::endl;
            std::cerr<<"total num of individuals in this device: "<<gridSize*blockSize<<std::endl;
            std::cerr<<"stream number #"<<n<<std::endl;
            std::cerr<<"seed is:"<<seed<<std::endl;
            std::cerr<<"global offset: "<<global_offset<<std::endl;
            genWeights<<< gridSize, blockSize, 0, _stream[n]>>>(device_genetics, seed, convertToKernel(_NNParams));
            CUDA_SAFE_CALL(hipPeekAtLastError());
            CUDA_SAFE_CALL(hipMemcpyAsync(&host_genetics.array[global_offset], &device_genetics.array[0], _streambytes, hipMemcpyDeviceToHost, _stream[n]));

            global_offset += _streamSize;
        }
        else{//host ram is full, fill the GPU now and then were done.
            long seed = std::clock() + std::clock()*seedItr++;
            gridSize=(_streambytes/_NNParams[7])/(blockSize); // round down isntead of up.
            std::cerr<<"stream number #"<<n<<std::endl;
            std::cerr<<"seed is:"<<seed<<std::endl;
            CUDA_SAFE_CALL(hipDeviceSynchronize());
            genWeights<<< gridSize, blockSize, 0, _stream[n]>>>(device_genetics, seed, convertToKernel(_NNParams));
            CUDA_SAFE_CALL(hipPeekAtLastError());

        }
    }
}


void NetworkGenetic::allocateHostAndGPUObjects( float pMax, size_t deviceRam, size_t hostRam){
    size_t totalHost = hostRam;
    size_t totalWeights = deviceRam; // the number of neurons has precident on the number of devices.
    size_t totalNeurons;
    std::cerr<<"total free device ram : "<<deviceRam<<std::endl;
    std::cerr<<"total free host ram : "<<hostRam<<std::endl;
    totalHost = (totalHost*pMax*_NNParams[7])/(_NNParams[7]);
    totalWeights = (totalWeights*pMax*_NNParams[7])/(_NNParams[7]);
    totalWeights = totalWeights - _NNParams[1]*(totalWeights/_NNParams[7]);
    totalNeurons = (_NNParams[1])*(totalWeights/_NNParams[7]);
    //make each of the memory arguments divisible by 512 (threads per block)
    _streambytes = (totalNeurons+totalWeights);
    _streamSize = _streambytes/sizeof(double);
    assert(_streambytes == (totalNeurons + totalWeights));
    _numOfStreams = ceil((totalNeurons + totalWeights +totalHost)/_streambytes); // number of streams = total array alloc / number of streams.
    assert(_streambytes * _numOfStreams <= totalNeurons+totalWeights+totalHost);
    std::cerr<<"bytes per stream :"<<_streambytes<<std::endl;
    std::cerr<<"number of streams: "<<_numOfStreams<<std::endl;
    device_genetics.size = (totalWeights+totalNeurons)/sizeof(double);
    host_genetics.size = totalHost/sizeof(double);
    std::cerr<<"device ram to allocate: "<<totalWeights<<std::endl;
    std::cerr<<"host ram to allocate: "<<totalHost<<std::endl;
    std::cerr<<"neuron ram to allocate: "<<totalNeurons<<std::endl;
    CUDA_SAFE_CALL(hipDeviceReset());
    CUDA_SAFE_CALL(hipSetDeviceFlags(hipDeviceMapHost));
    CUDA_SAFE_CALL(hipHostAlloc((void**)&host_genetics.array, totalHost, hipHostMallocMapped | hipHostMallocPortable));
    CUDA_SAFE_CALL(hipHostGetDevicePointer((void**)&host_genetics_device.array, (void*)host_genetics.array, 0 ));
    CUDA_SAFE_CALL(hipMalloc((void**) &device_genetics.array, totalWeights + totalNeurons));
    std::cerr<<"all allocated, moving on."<<std::endl;
    _stream.resize(_numOfStreams);
    for(int i=0; i<_numOfStreams; i++){
        CUDA_SAFE_CALL(hipStreamCreate(&_stream.at(i)));
        CUDA_SAFE_CALL( hipStreamQuery(_stream.at(i)));
    }
}
bool NetworkGenetic::init(int sampleRate, int SiteNum, std::vector<double> siteData){
    _sampleRate = sampleRate;
    _numofSites = SiteNum;
    _siteData.resize(siteData.size());
    try{thrust::copy(siteData.begin(), siteData.end(), _siteData.begin());}
    catch(thrust::system_error &e){
        std::cerr<<"Error resizing vector Element: "<<e.what()<<std::endl;
        exit(-1);
    }
    catch(std::bad_alloc &e){
        std::cerr<<"Ran out of space due to : "<<"host"<<std::endl;
        std::cerr<<e.what()<<std::endl;
        exit(-1);
    }
    _istraining = false;
    return true;
}

bool NetworkGenetic::checkForWeights(std::string filepath){
    std::ifstream weightFile(filepath.c_str(), std::ios_base::ate | std::ios_base::binary);
    std::cerr<<"checking for weights.."<<std::endl;
    if(weightFile){
        std::cerr<<"the weightfile exists"<<std::endl;
        std::string line;
        int filesize = weightFile.tellg();
        weightFile.seekg(0, weightFile.beg);
        int itr =0;
        this->allocateHostAndGPUObjects(0.85, GetDeviceRamInBytes(), filesize - GetDeviceRamInBytes());
        for( int n=0; n<_numOfStreams; n++){
            int offset = n*_streambytes/sizeof(double);
            CUDA_SAFE_CALL(hipMemset(&device_genetics.array, 0, _streambytes));
            while(std::getline(weightFile, line) && itr <= device_genetics.size){ // each line
                std::string item;
                std::stringstream ss(line);
                while(std::getline(ss, item, ',') && itr <= device_genetics.size){ // each weight
                    device_genetics.array[itr] = std::atoi(item.c_str());
                }
            }
            CUDA_SAFE_CALL(hipMemcpyAsync(&host_genetics.array[offset], &device_genetics.array[offset], _streambytes, hipMemcpyDeviceToHost, _stream[n]));
            itr = 0;
        }
        weightFile.close();
        return true;
    }
    else{
        std::cerr<<"no weightfile found"<<std::endl;
        weightFile.close();
        return false;
    }
}

void NetworkGenetic::doingTraining(int site, int hour, double lat,
                                   double lon, double mag, double dist){
    _answers.push_back(site);
    _answers.push_back(hour);
    _answers.push_back(lat);
    _answers.push_back(lon);
    _answers.push_back(mag);
    _answers.push_back(dist);
    _istraining = true;
}

void NetworkGenetic::storeWeights(std::string filepath){
    std::ofstream ret;
    ret.open(filepath.c_str(), std::ios_base::out | std::ios_base::trunc);
    for(int i=0; i<device_genetics.size; i++){
        ret << device_genetics.array[i]<<","<<std::endl;
    }
    ret.close();
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    for(int i=0; i<_numOfStreams; i++){
        CUDA_SAFE_CALL(hipStreamDestroy(_stream[i]));
    }
    CUDA_SAFE_CALL(hipFree(device_genetics.array));
    CUDA_SAFE_CALL(hipFree(host_genetics.array));

}

void NetworkGenetic::forecast(std::vector<double> *ret, int &hour, std::vector<int> *data, double &Kp, std::vector<double> *globalQuakes)
{
    //were going to normalize the inputs using v` = v-mean/stdev, so we need mean and stdev for each channel.
    double meanCh1=0, meanCh2=0, meanCh3=0, stdCh1=0, stdCh2=0, stdCh3=0;
    int num=0;
    std::cerr<<"right before mean & std calc"<<std::endl;
    for(int i=0; i<3600*_sampleRate; i++){
        for(int j=0; j < _numofSites; j++){
            meanCh1 += data->at(3600*_sampleRate*j*3 + 0*3600*_sampleRate+i);
            meanCh2 += data->at(3600*_sampleRate*j*3 + 1*3600*_sampleRate+i);
            meanCh3 += data->at(3600*_sampleRate*j*3 + 2*3600*_sampleRate+i);
            num++;
        }
    }
    meanCh1 = meanCh1/num;
    meanCh2 = meanCh2/num;
    meanCh3 = meanCh3/num;
    stdCh1 = sqrt(meanCh1);
    stdCh2 = sqrt(meanCh2);
    stdCh3 = sqrt(meanCh3);
    std::cerr<<"means are: "<<meanCh1<<" "<<meanCh2<<" "<<meanCh3<<std::endl;
    std::cerr<<"stdevs are: "<<stdCh1<<" "<<stdCh2<<" "<<stdCh3<<std::endl;
    std::cerr<<"channels std and mean calculated"<<std::endl;
    //input data from all sites and all channels normalized
    if(_istraining == true){
        thrust::device_vector<int>* input = new thrust::device_vector<int>(data->size());
        thrust::device_vector<double>* retVec = new thrust::device_vector<double>(2160*_numofSites);
        thrust::device_vector<double>* gQuakeAvg = new thrust::device_vector<double>(globalQuakes->size());
        thrust::device_vector<thrust::pair<int, int> >* dConnect = new thrust::device_vector<thrust::pair<int, int> >(_connect->size());
        thrust::copy(data->begin(), data->end(), input->begin());
        thrust::copy(globalQuakes->begin(), globalQuakes->end(), gQuakeAvg->begin());
        thrust::copy(_connect->begin(), _connect->end(), dConnect->begin());

        int gridSize; //the blocksize defined by the configurator
        int blockSize = 512; // the actual grid size needed
        double fitnessAvg=0;
        int fitItr=0;
        gridSize=(_streamSize/_NNParams[7])/(blockSize);
        int host_offset = 0;
        for(int n=0; n<_numOfStreams; n++){

            Net<<<gridSize, blockSize, 0, _stream[n]>>>(device_genetics, convertToKernel(_NNParams),convertToKernel(gQuakeAvg),
                                                        convertToKernel(input),convertToKernel(_siteData),convertToKernel(_answers),
                                                        convertToKernel(dConnect),Kp,_sampleRate,_numofSites, hour,
                                                        meanCh1, meanCh2, meanCh3, stdCh1, stdCh2, stdCh3);

            CUDA_SAFE_CALL(hipPeekAtLastError());
            std::cerr<<"net completed."<<std::endl;
            int gridSize = (_NNParams[8]/blockSize)+((_NNParams[8]%blockSize) ? 1 : 0);
            thrust::device_vector<double> partial_reduce_sums(gridSize+1);
            reduce_by_block<<<gridSize, blockSize, blockSize*sizeof(double), _stream[n]>>>(device_genetics,
                                                                                           convertToKernel(partial_reduce_sums),
                                                                                           convertToKernel(_NNParams));
            std::cerr<<"reduce by block completed."<<std::endl;
            gridSize=_streamSize/blockSize; //swap in place, every double is a job.
            CUDA_SAFE_CALL(hipPeekAtLastError());
            CUDA_SAFE_CALL(hipStreamSynchronize(_stream[n]));
            swapMemory<<<gridSize, blockSize, 0, _stream[n]>>>(device_genetics, host_genetics_device, host_offset);
            std::cerr<<"memory swap completed"<<std::endl;
            CUDA_SAFE_CALL(hipPeekAtLastError());
                    for(thrust::device_vector<double>::iterator it = partial_reduce_sums.begin();
                    it != partial_reduce_sums.end(); ++it){
                fitnessAvg += *it;
                fitItr++;
            }
            host_offset += _streamSize;
        }
        fitnessAvg = fitnessAvg /fitItr;
        std::cerr<<"the average fitness for this round is: "<<fitnessAvg<<std::endl;

        delete dConnect;
        delete input;
        delete gQuakeAvg;
        delete retVec;
    }
    else{
        std::cerr<<"entered not training version.."<<std::endl;
        typedef std::vector<thrust::pair<int, int> > connectPairMatrix;
        //replace this later
        _best.resize(_NNParams[2]);
        for(std::vector<double>::iterator it = _best.begin(); it != _best.end(); ++it){
            std::srand(std::time(NULL)+*it);
            *it = (double)(std::rand())/(RAND_MAX);
        }
        std::cerr<<"example best vector has been set."<<std::endl;
        double CommunityLat = 0;
        double CommunityLon = 0;
        std::vector<double> When(_numofSites, 0);
        std::vector<double> HowCertain(_numofSites,0);
        std::vector<double> CommunityMag(_numofSites, 1); //give all sites equal mag to start, this value is [0,1]
        std::cerr<<"all output vectors created and initialized."<<std::endl;
        for(int step=0; step<3600*_sampleRate; step++){
            std::cerr<<"entering step #"<<step<<std::endl;
            for(int j=0; j<_numofSites; j++){ //sitesWeighted Lat/Lon values are determined based on all previous sites mag output value.
                CommunityLat += _siteData[j*2]*CommunityMag[j];
                CommunityLon += _siteData[j*2+1]*CommunityMag[j];
            }
            CommunityLat = CommunityLat/_numofSites;
            CommunityLon = CommunityLon/_numofSites;

            for(int j=0; j<_numofSites; j++){ // each site is run independently of others, but shares an output from the previous step
                std::cerr<<"entering site #"<<j<<std::endl;
                double latSite = _siteData[j*2];
                double lonSite = _siteData[j*2+1];
                double avgLatGQuake = globalQuakes->at(0);
                double avgLonGQuake = globalQuakes->at(1);
                double GQuakeAvgMag = globalQuakes->at(3);
                double GQuakeAvgdist = distCalc(latSite, lonSite, avgLatGQuake, avgLonGQuake);
                double GQuakeAvgBearing = bearingCalc(latSite, lonSite, avgLatGQuake, avgLonGQuake);
                double CommunityDist = distCalc(latSite, lonSite, CommunityLat, CommunityLon);
                double CommunityBearing = bearingCalc(latSite, lonSite, CommunityLat, CommunityLon);
                std::vector<double> input;
                std::vector<double> hidden, outputs, mem, memGateOut, memGateIn, memGateForget;
                //replace these with real connections, num of inputs, and num of hidden & memory neurons (mem neurons probably accurate)
                input.resize(_NNParams[3], 0); // number of inputs is 9.
                hidden.resize(_NNParams[4], 0); // for practice sake, lets say each input has its own neuron (might be true!)
                mem.resize(_NNParams[5], 0); // stores the input if gate is high
                memGateOut.resize(_NNParams[5], 0); //connects to the input layer and the memN associated with input, if 1 it sends up stream and deletes, if low it keeps.
                memGateIn.resize(_NNParams[5], 0);
                memGateForget.resize(_NNParams[5], 0);
                outputs.resize(_NNParams[6], 0); /* 3 outputs, 1 with an hour in the future when the earthquake will hit,
                    1 with the porbability of that earthquake happening (between [0,1]) and 1 with the sites magnitude (for community feedback) */
                std::cerr<<"all neuron vectors are sized, all pre-net calculations done."<<std::endl;
                int n =0;
                int startOfInput = 0;
                int startOfHidden = startOfInput +_NNParams[3];
                int startOfMem = startOfHidden + _NNParams[4];
                int startOfMemGateIn = startOfMem + _NNParams[5];
                int startOfMemGateOut = startOfMemGateIn + _NNParams[5];
                int startOfMemGateForget = startOfMemGateOut + _NNParams[5];
                int startOfOutput = startOfMemGateForget + _NNParams[5];
                input[0] = shift((double)(data->at(3600*_sampleRate*j*3 + 0*(3600*_sampleRate)+step)), meanCh1, stdCh1);
                input[1] = normalize((double)(data->at(3600*_sampleRate*j*3 + 1*(3600*_sampleRate)+step)), meanCh2, stdCh2);
                input[2] = normalize((double)(data->at(3600*_sampleRate*j*3 + 2*(3600*_sampleRate)+step)), meanCh3, stdCh3);
                input[3] = shift(GQuakeAvgdist, 40075.1, 0);
                input[4] = shift(GQuakeAvgBearing, 360, 0);
                input[5] = shift(GQuakeAvgMag, 9.5, 0);
                input[6] = shift(Kp, 10, 0);
                input[7] = shift(CommunityDist,40075.1/2, 0);
                input[8] = shift(CommunityBearing, 360, 0);
                //lets reset all neuron values for this new timestep (except memory neurons)
                for(int gate=0; gate<_NNParams[5]; gate++){
                    memGateIn.at(gate) = 0;
                    memGateOut.at(gate) = 0;
                    memGateForget.at(gate) = 0;
                }
                for(int hid=0; hid<_NNParams[4]; hid++){
                    hidden[hid] = 0;
                }
                for(int out=0; out<_NNParams[6]; out++){
                    outputs[out] = 0;
                }
                std::cerr<<"memGate, hidden, and output neurons are zeroed."<<std::endl;
                //now that everything that should be zeroed is zeroed, lets start the network.
                //mem gates & LSTM nodes --
                std::cerr<<"preparing to set the values for memoryGates."<<std::endl;
                for(int gate = 0; gate<_NNParams[5]; gate++){//calculate memory gate node values, you can connect inputs & hidden neurons to them.
                    for(connectPairMatrix::iterator it = _connect->begin(); it!= _connect->end(); ++it){//for memGateIn
                        if(it->second == gate+startOfMemGateIn && it->first < startOfHidden){ //for inputs
                            std::cerr<<"weights for memGateIn #"<<gate<<" is: "<<_best[n];
                            memGateIn.at(gate) += input[it->first-startOfInput]*_best[n++]; // memGateIn vect starts at 0
                        }
                        else if(it->second == gate+startOfMemGateIn && it->first >startOfHidden && it->first < startOfMem){//for hidden neurons
                            memGateIn.at(gate) += hidden[it->first-startOfHidden]*_best[n++];
                        }
                    }
                    for(connectPairMatrix::iterator it = _connect->begin(); it!= _connect->end(); ++it){//for memGateOut
                        if(it->second == gate+startOfMemGateOut && it->first < startOfHidden){//for inputs
                            std::cerr<<"weights for memGateOut #"<<gate<<" is: "<<_best[n];
                            memGateOut.at(gate) += input[it->first-startOfInput]*_best[n++];
                        }
                        else if(it->second == gate+startOfMemGateOut && it->first >startOfHidden && it->first <startOfMem){//for hidden neurons
                            memGateOut.at(gate) += hidden[it->first-startOfHidden]*_best[n++];
                        }
                    }
                    for(connectPairMatrix::iterator it = _connect->begin(); it!= _connect->end(); ++it){//for  memGateForget
                        if(it->second == gate+startOfMemGateForget && it->first < startOfHidden){//for inputs
                            std::cerr<<"weights for memGateForget #"<<gate<<" is: "<<_best[n];
                            memGateForget.at(gate) += input[it->first - startOfInput]*_best[n++];
                        }
                        else if(it->second == gate+startOfMemGateForget && it->first >startOfHidden && it->first <startOfMem){//for hidden neurons
                            memGateForget.at(gate) += hidden[it->first-startOfHidden]*_best[n++];
                        }
                    }
                    memGateIn.at(gate) = ActFunc(memGateIn.at(gate));
                    memGateOut.at(gate) = ActFunc(memGateOut.at(gate));
                    memGateForget.at(gate) = ActFunc(memGateForget.at(gate));
                    std::cerr<<"memGateIn val: "<<memGateIn.at(gate)<<std::endl;
                    std::cerr<<"memGateOut val: "<<memGateOut.at(gate)<<std::endl;
                    std::cerr<<"memGateForget val: "<<memGateForget.at(gate)<<std::endl;
                }
                //since we calculated the values for memGateIn and memGateOut, and MemGateForget..
                for (int gate = 0; gate<_NNParams[5]; gate++){ // if memGateIn is greater than 0.3, then let mem = the sum inputs attached to memGateIn
                    if(memGateIn.at(gate) > 0.5){ //gate -startOfMemGateIn = [0, num of mem neurons]
                        for(connectPairMatrix::iterator it = _connect->begin(); it!= _connect->end(); ++it){
                            if(it->second == gate+startOfMemGateIn && it->first < gate+startOfHidden){//only pass inputs
                                mem.at(gate) += input[it->first-startOfInput]; // no weights attached, but the old value stored here is not removed.
                            }
                        }
                    }
                    if(memGateForget.at(gate) > 0.5){// if memGateForget is greater than 0.5, then tell mem to forget
                        mem.at(gate) = 0;
                    }
                    //if memGateForget fires, then memGateOut will output nothing.
                    if(memGateOut.at(gate) > 0.5){//if memGateOut is greater than 0.3, let the nodes mem is connected to recieve mem
                        for(connectPairMatrix::iterator it = _connect->begin(); it!= _connect->end(); ++it){
                            if(it->first == gate+startOfMem){// since mem node: memIn node : memOut node = 1:1:1, we can do this.
                                hidden[it->second-startOfHidden] += mem.at(gate);
                            }
                        }
                    }
                    std::cerr<<"mem val stored is: "<<mem.at(gate)<<std::endl;
                }

                // hidden neuron nodes --
                for(int hid=0; hid<_NNParams[4]; hid++){ // for all hidden neurons at layer 1, lets sum the inputs, the memory values were already added.
                    for(connectPairMatrix::iterator it = _connect->begin(); it!= _connect->end(); ++it){ // Add the inputs to the hidden neurons
                        if(it->second == hid+startOfHidden && it->first < startOfHidden){ // if an input connects with this hidden neuron
                            hidden[hid] += input[it->first]*_best[n++];
                        }
                    }
                    for(connectPairMatrix::iterator it = _connect->begin(); it!= _connect->end(); ++it){//add other hidden neuron inputs to each hidden neuron (if applicable)
                        if(it->second == hid+startOfHidden && it->first < startOfMem && it->first > startOfHidden){
                            hidden[hid] += hidden[it->first-startOfHidden]*_best[n++];
                        }
                    }
                    hidden[hid] += 1*_best[n++]; // add bias
                    hidden[hid] = ActFunc(hidden[hid]); // then squash it.
                    std::cerr<<"hidden nueron values: "<<hidden[hid]<<std::endl;
                }
                //output nodes --

                for(int out =0; out<_NNParams[6]; out++){// add hidden neurons to the output nodes
                    for(connectPairMatrix::iterator it = _connect->begin(); it!= _connect->end(); ++it){
                        if(it->second == out+startOfOutput){
                            outputs[out] += hidden[it->first-startOfHidden]*_best[n++];
                        }
                    }
                    outputs[out] += 1*_best[n++]; // add bias
                    outputs[out] = ActFunc(outputs[out]);// then squash it.
                }


                When[j] += outputs[0]*((2160-hour)-hour)+2160-hour; //return when back to an integer value (adjust to fit within boundaries)
                std::cerr<<"When for site: "<<j<<" and for step: "<<step<< " is: "<<When[j]<<std::endl;
                HowCertain[j] += outputs[1];
                std::cerr<<"howCertain for site: "<<j<<" and for step: "<<step<< " is: "<<HowCertain[j]<<std::endl;
                CommunityMag[j] =  outputs[2]; // set the next sets communityMag = output #3.
                std::cerr<<"ComunityMagnitude for site: "<<j<<" and for step: "<<step<< " is: "<<CommunityMag[j]<<std::endl;
            }
        }
        for(int j=0; j<_numofSites; j++){ // each site has its own when and howcertain vector
            When[j] = When[j]/3600*_sampleRate;
            HowCertain[j] = HowCertain[j]/3600*_sampleRate;
        }
        //all done, lets output the return matrix.
        //since right now were using a point value for when & how certain (only one output per site),
        //we're going to approximate using a normal distribution around when with a sigma of howCertain, over the whole array from T=currentHour [T, 2160]
        for(int h=hour; h<2160; h++){
            for(int j=0; j<_numofSites; j++){
                ret->at(h*_numofSites+j)= 1/(1/HowCertain[j]*sqrt(2*M_PI))*exp(-pow(h-When[j], 2)/(2*pow(1/HowCertain[j], 2))); // normal distribution with a mu of When and a sigma of 1/HowCertain
            }
        }
    }
}
