#include "hip/hip_runtime.h"
#include <kernelDefs.h>

__global__ void reduceFirstKern(kernelArray<double> weights,kernelArray<double> per_block_results, kernelArray<int> params,  int device_offset){
    extern __shared__ float sdata[];

    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int ind = params.array[19] + idx + device_offset;

    // load input into __shared__ memory
    double x = 0;

    x = weights.array[ind];

    sdata[threadIdx.x] = x;
    __syncthreads();

    // contiguous range pattern
    for(int offset = blockDim.x / 2;
        offset > 0;
        offset >>= 1)
    {
        if(threadIdx.x < offset)
        {
            // add a partial sum upstream to our own
            sdata[threadIdx.x] += sdata[threadIdx.x + offset];
        }
        // wait until all threads in the block have
        // updated their partial sums
        __syncthreads();
    }

    // thread 0 writes the final result
    if(threadIdx.x == 0)
    {
        sdata[0] = sdata[0] / blockDim.x;
        per_block_results.array[blockIdx.x] = sdata[0];
    }
}

__global__ void reduceSecondKern(kernelArray<double> per_block_results, kernelArray<int> params, double *result){
    unsigned int idx = threadIdx.x+ blockIdx.x*blockDim.x;
    if(idx ==0){
        *result =0;
        for(int i=0; i<per_block_results.size; i++){
            *result += per_block_results.array[i];

        }
        *result = *result/per_block_results.size;
    }
}
