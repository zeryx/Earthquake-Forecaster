#include "hip/hip_runtime.h"
#include <kernelDefs.h>

__global__ void reduceKern(kernelArray<double> weights,kernelArray<double> per_block_results,kernelArray<int> params, int n, int device_offset, int blockOffset){
    extern __shared__ float sdata[];

    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int ind = idx*params.array[7] + device_offset;

    // load input into __shared__ memory
    float x = 0;
    if(idx < n)
    {
        x = weights.array[ind + params.array[2]];
    }
    sdata[threadIdx.x] = x;
    __syncthreads();

    // contiguous range pattern
    for(int offset = blockDim.x / 2;
        offset > 0;
        offset >>= 1)
    {
        if(threadIdx.x < offset)
        {
            // add a partial sum upstream to our own
            sdata[threadIdx.x] += sdata[threadIdx.x + offset];
        }
        // wait until all threads in the block have
        // updated their partial sums
        __syncthreads();
    }

    // thread 0 writes the final result
    if(threadIdx.x == 0)
    {
        per_block_results.array[blockIdx.x+blockOffset] = sdata[0];
    }
}
