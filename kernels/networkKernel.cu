#include "hip/hip_runtime.h"
#include <kernelDefs.h>
//using
extern __constant__ int inputData[];
extern __constant__ double answers[];
extern __constant__ double globalQuakes[];
extern __constant__ double siteData[];
extern __constant__ double Kp;
extern __constant__ int site_offset[];
extern __constant__ int channel_offset[];
extern __constant__ int trainingsize;
extern __constant__ devicePair<dcon, dcon> connections[];
//endof using

__global__ void NetKern(kernelArray<double> Vec, kernelArray<int> params,int hour, kernelArray<double> meanCh,
                        kernelArray<double> stdCh, size_t device_offset){
//    extern __shared__ std::pair<con, con> connections[];
//    const int tix = threadIdx.x;
//    for(int i=0; i<params.array[26]; i=i+blockDim.x){
//        if((tix+i)<params.array[26]){
//            connections[tix+i].first.first = connections.array[tix+i].first.first;
//            connections[tix+i].first.second = connections.array[tix+i].first.second;
//            connections[tix+i].second.first = connections.array[tix+i].second.first;
//            connections[tix+i].second.second = connections.array[tix+i].second.second;

//        }
//    }
//    __syncthreads();
    const int idx = blockIdx.x * blockDim.x + threadIdx.x; // for each thread is one individual
    typedef devicePair<dcon, dcon>*  connectPairMatrix;
    const int ind = params.array[10]; // number of individuals on device
    const int weightsOffset = params.array[11] + idx + device_offset;
    const int inputOffset = params.array[12] + idx + device_offset; // 6 is the offset to the start of the input neurons
    const int hiddenOffset = params.array[13] + idx + device_offset;
    const int memOffset = params.array[14] + idx + device_offset;
    const int memGateInOffset = params.array[15] + idx + device_offset;
    const int memGateOutOffset = params.array[16] + idx + device_offset;
    const int memGateForgetOffset = params.array[17] + idx + device_offset;
    const int outputOffset = params.array[18] + idx + device_offset;
    const int fitnessOffset = params.array[19] + idx + device_offset;
    const int communityMagOffset = params.array[20] +idx +device_offset;
    const int whenOffset = params.array[21] + idx + device_offset;
    const int howCertainOffset = params.array[22] + idx + device_offset;
    const int ageOffset = params.array[25] + idx + device_offset;


    //reset values from previous individual.
    //community magnitude is not set, as this needs to be continued.
    for(int i=0; i<params.array[23]; i++){
        Vec.array[whenOffset +i*ind] = 0;
        Vec.array[howCertainOffset +i*ind] =0;
    }

    for(int i=0; i<trainingsize; i++){
        float CommunityLat = 0;
        float CommunityLon = 0;
        for(int j=0; j<params.array[23]; j++){//sitesWeighted Lat/Lon values are determined based on all previous zsites mag output value.
            CommunityLat += siteData[j*2]*Vec.array[communityMagOffset+j*ind];
            CommunityLon += siteData[j*2+1]*Vec.array[communityMagOffset+j*ind];
        }
        CommunityLat = CommunityLat/params.array[23];
        CommunityLon = CommunityLon/params.array[23];
        for(int j=0; j<params.array[23]; j++){ //each site is run independently of others, but shares an output from the previous step

            float latSite = siteData[j*2];
            float lonSite = siteData[j*2+1];
            float avgLatGQuake = globalQuakes[0];
            float avgLonGQuake = globalQuakes[1];
            float GQuakeAvgMag = globalQuakes[3];
            float GQuakeAvgdist = distCalc(latSite, lonSite, avgLatGQuake, avgLonGQuake);
            float GQuakeAvgBearing = bearingCalc(latSite, lonSite, avgLatGQuake, avgLonGQuake);
            float CommunityDist = distCalc(latSite, lonSite, CommunityLat, CommunityLon);
            float CommunityBearing = bearingCalc(latSite, lonSite, CommunityLat, CommunityLon);
            /* 3 outputs, 1 with an hour in the future when the earthquake will hit,
                        1 with the porbability of that earthquake happening (between [0,1]) and 1 with the sites magnitude (for community feedback) */
            int n =0; // n is the weight number
            for(int k=0; k<3; k++){
                Vec.array[inputOffset+k*ind] = normalize(inputData[site_offset[j]+channel_offset[k]+i], meanCh.array[k], stdCh.array[k]);//channel 1
            }
            Vec.array[inputOffset+3*ind] = shift(GQuakeAvgdist, 40075.1, 0);
            Vec.array[inputOffset+4*ind] = shift(GQuakeAvgBearing, 360, 0);
            Vec.array[inputOffset+5*ind] = shift(GQuakeAvgMag, 9.5, 0);
            Vec.array[inputOffset+6*ind] = shift(Kp, 10, 0);
            Vec.array[inputOffset+7*ind] = shift(CommunityDist,40075.1, 0);
            Vec.array[inputOffset+8*ind] = shift(CommunityBearing, 360, 0);
            //lets reset all neuron values for this new timestep (except memory neurons)
            for(int gate=0; gate<params.array[5]; gate++){
                Vec.array[memGateInOffset+gate*ind] = 0;
                Vec.array[memGateOutOffset+gate*ind] = 0;
                Vec.array[memGateForgetOffset+gate*ind] = 0;
            }
            for(int hid=0; hid<params.array[4]; hid++){
                Vec.array[hiddenOffset+hid*ind] = 0;
            }
            for(int out=0; out<params.array[9]; out++){
                Vec.array[outputOffset+out*ind] = 0;
            }

            //now that everything that should be zeroed is zeroed, lets start the network.
            //mem gates & LSTM nodes --
            for(int gate = 0; gate<params.array[6]; gate++){//calculate memory gate node values, you can connect inputs & hidden neurons to them.
                for(int pair=0; pair<params.array[26]; pair++){
                    //for memGateIn
                    if(connections[pair].second.first == typeMemGateIn && connections[pair].second.second == gate && connections[pair].first.first ==typeHidden){ //for inputs
                        Vec.array[memGateInOffset+gate*ind] += Vec.array[inputOffset+(connections[pair].first.second)*ind]*Vec.array[weightsOffset+(n++)*ind]; // memGateIn vect starts at 0
                    }
                    else if(connections[pair].second.first == typeMemGateIn && connections[pair].second.second == gate && connections[pair].first.first == typeHidden){//for hidden neurons
                        Vec.array[memGateInOffset+gate*ind] += Vec.array[hiddenOffset+(connections[pair].first.second)*ind]*Vec.array[weightsOffset+(n++)*ind];
                    }
                }
                Vec.array[memGateInOffset+gate*ind] = ActFunc(Vec.array[memGateInOffset+gate*ind]);
            }
            //for memGateOut

            for(int gate = 0; gate<params.array[7]; gate++){//calculate memory gate node values, you can connect inputs & hidden neurons to them.
                for(int pair=0; pair<params.array[26]; pair++){
                    if(connections[pair].second.first == typeMemGateOut && connections[pair].second.second == gate && connections[pair].first.first == typeInput){//for inputs
                        Vec.array[memGateOutOffset+gate*ind] += Vec.array[inputOffset+(connections[pair].first.second)*ind]*Vec.array[weightsOffset+(n++)*ind];
                    }
                    else if(connections[pair].second.first == typeMemGateOut && connections[pair].second.second == gate && connections[pair].first.first == typeHidden){//for hidden neurons
                        Vec.array[memGateOutOffset+gate*ind] += Vec.array[hiddenOffset+(connections[pair].first.second)*ind]*Vec.array[weightsOffset+(n++)*ind];
                    }
                }
                Vec.array[memGateOutOffset+gate*ind] = ActFunc(Vec.array[memGateOutOffset+gate*ind]);

            }
            //for  memGateForget
            for(int gate = 0; gate<params.array[8]; gate++){//calculate memory gate node values, you can connect inputs & hidden neurons to them.
                for(int pair=0; pair<params.array[26]; pair++){
                    if(connections[pair].second.first == typeMemGateForget && connections[pair].second.second == gate && connections[pair].first.first == typeInput){//for inputs
                        Vec.array[memGateForgetOffset+gate*ind] += Vec.array[inputOffset+(connections[pair].first.second)*ind]*Vec.array[weightsOffset+(n++)*ind];
                    }
                    else if(connections[pair].second.first == typeMemGateForget && connections[pair].second.second == gate && connections[pair].first.first == typeHidden){//for hidden neurons
                        Vec.array[memGateForgetOffset+gate*ind] += Vec.array[hiddenOffset+(connections[pair].first.second)*ind]*Vec.array[weightsOffset+(n++)*ind];
                    }
                }
                Vec.array[memGateForgetOffset+gate*ind] = ActFunc(Vec.array[memGateForgetOffset+gate*ind]);
            }
            //since we calculated the values for memGateIn and memGateOut, and MemGateForget..
            for (int gate = 0; gate<params.array[6]; gate++){ // if memGateIn is greater than 0.5, then let mem = the sum inputs attached to memGateIn
                if(Vec.array[memGateInOffset+gate*ind] > 0.5){ //gate -memGateInOffset = [0, num of mem neurons]
                    for(int pair=0; pair<params.array[26]; pair++){
                        if(connections[pair].second.first == typeMemGateIn && connections[pair].second.second == gate && connections[pair].first.first == typeInput){//only pass inputs
                            Vec.array[memOffset+gate*ind] += Vec.array[inputOffset+(connections[pair].first.second)*ind]; // no Vec attached, but the old value stored here is not removed.
                        }
                    }
                }
            }
            for (int gate = 0; gate<params.array[7]; gate++){ // if memGateForget is greater than 0.5, then tell mem to forget
                if(Vec.array[memGateForgetOffset+gate*ind] > 0.5){
                    for(int pair=0; pair<params.array[26]; pair++){
                        if(connections[pair].second.first == typeMemGateForget && connections[pair].second.second == gate && connections[pair].first.first == typeMemory){
                            Vec.array[memOffset+connections[pair].first.second*ind] =0;
                        }
                    }
                }
            }
            //if memGateForget fires, then memGateOut will output nothing.
            for (int gate = 0; gate<params.array[7]; gate++){//if memGateOut is greater than 0.5, let the nodes mem is connected to recieve mem
                if(Vec.array[memGateOutOffset+gate*ind] > 0.5){
                    for(int pair=0; pair<params.array[26]; pair++){
                        if(connections[pair].first.first == typeMemory && connections[pair].first.second == gate && connections[pair].second.first == typeHidden){// since mem node: memIn node : memOut node = 1:1:1, we can do this.
                            Vec.array[hiddenOffset+(connections[pair].second.second)*ind] += Vec.array[memOffset+gate*ind];
                        }
                    }
                }
            }

            // hidden neuron nodes --
            for(int hid=0; hid<params.array[4]; hid++){ // for all hidden neurons at layer 1, lets sum the inputs, the memory values were already added.
                for(int pair=0; pair<params.array[26]; pair++){ // Add the inputs to the hidden neurons
                    if(connections[pair].second.first == typeHidden && connections[pair].second.first == hid && connections[pair].first.first == typeInput){ // if an input connects with this hidden neuron
                        Vec.array[hiddenOffset+hid*ind] += Vec.array[inputOffset+(connections[pair].first.second)*ind]*Vec.array[weightsOffset + (n++)*ind];
                    }
                    else if(connections[pair].second.first == typeHidden && connections[pair].second.second == hid && connections[pair].first.first == typeHidden){
                        Vec.array[hiddenOffset+hid*ind] += Vec.array[hiddenOffset+(connections[pair].first.second)*ind]*Vec.array[weightsOffset + (n++)*ind];
                    }
                }
                Vec.array[hiddenOffset+hid*ind] += 1*Vec.array[weightsOffset + (n++)*ind]; // add bias
                Vec.array[hiddenOffset+hid*ind] = ActFunc(Vec.array[hiddenOffset+hid*ind]); // then squash it.
            }
            //output nodes --
            for(int out =0; out<params.array[9]; out++){// add hidden neurons to the output nodes
                for(int pair=0; pair<params.array[26]; pair++){
                    if(connections[pair].second.first == typeOutput && connections[pair].second.second == out && connections[pair].first.first == typeHidden){
                        Vec.array[outputOffset+out*ind] += Vec.array[hiddenOffset+(connections[pair].first.second)*ind]*Vec.array[weightsOffset + (n++)*ind];
                    }
                }
                Vec.array[outputOffset+out*ind] += 1*Vec.array[weightsOffset + (n++)*ind]; // add bias
                Vec.array[outputOffset+out*ind] = ActFunc(Vec.array[outputOffset+out*ind]);// then squash it.
            }

            Vec.array[whenOffset+j*ind] += Vec.array[outputOffset+0*ind]*((2160-hour)-hour)+2160-hour; // nv = ((ov - omin)*(nmax-nmin) / (omax - omin))+nmin
            Vec.array[howCertainOffset+j*ind] += Vec.array[outputOffset+1*ind];
            Vec.array[communityMagOffset+j*ind] =  Vec.array[outputOffset+2*ind]; // set the next sets communityMag = output #3.
        }
    }
    for(int j=0; j<params.array[23]; j++){ // now lets get the average when and howcertain values.
        Vec.array[whenOffset+j*ind] = Vec.array[whenOffset+j*ind]/trainingsize;
        Vec.array[howCertainOffset+j*ind] = Vec.array[howCertainOffset+j*ind]/trainingsize;
    }
    /*calculate score for this individual during this round, current scoring mechanism is - e^(-(abs(whenGuess-whenAns)+distToCorrectSite)), closer to 1 the better.   */
    float maxCertainty=0;
    float whenGuess=0;
    float guessLat=0;
    float guessLon=0;
    for(int j=0; j<params.array[23]; j++){
        if(Vec.array[howCertainOffset+j*ind] > maxCertainty){
            maxCertainty = Vec.array[howCertainOffset+j*ind];
            whenGuess = Vec.array[whenOffset+j*ind];
            guessLat = siteData[j*2];
            guessLon = siteData[j*2+1];
        }
    }
    float ansLat = siteData[(int)answers[0]*2];
    float ansLon = siteData[(int)answers[0]*2+1];
    int whenAns = (int)answers[1] - hour;
    Vec.array[ageOffset] += 1; //this indvidiual has existed for 1 more iteration.
    double oldFit = Vec.array[fitnessOffset];
    Vec.array[fitnessOffset] = scoreFunc(whenGuess, whenAns, guessLat, guessLon, ansLat, ansLon, oldFit); //we take the average beacuse consistency is more important than being really good at this particular hour.
}
