#include "hip/hip_runtime.h"
#include <kernelDefs.h>
#include <neuroFunc.h>
#include <utilFunc.h>

//using
extern __constant__ int inputData[];
extern __constant__ double answers[];
extern __constant__ double globalQuakes[];
extern __constant__ double siteData[];
extern __constant__ double Kp;
extern __constant__ int site_offset[];
extern __constant__ int channel_offset[];
extern __constant__ int trainingsize;
//endof using

__global__ void NetKern(kernelArray<double> Vec, kernelArray<int> params, Order* commandQueue, int hour, kernelArray<double> meanCh,
                        kernelArray<double> stdCh, size_t device_offset){

    extern __shared__  Order sharedQueue[];
    const int tix = threadIdx.x;
    if(tix < params.array[26]){
        sharedQueue[tix]._first = commandQueue[tix]._first;
        sharedQueue[tix]._second =  commandQueue[tix]._second;
        sharedQueue[tix]._third = commandQueue[tix]._third;
        sharedQueue[tix]._fourth = commandQueue[tix]._fourth;
        sharedQueue[tix]._verb = commandQueue[tix]._verb;
    }
    __syncthreads();
    const int idx = blockIdx.x * blockDim.x + threadIdx.x; // for each thread is one individual
    const int ind = params.array[10];

    const int weightsOffset = params.array[11] + idx + device_offset;
    const int inputOffset = params.array[12] + idx + device_offset;
    const int hiddenOffset = params.array[13] + idx + device_offset;
    const int memOffset = params.array[14] + idx + device_offset;
    const int memGateInOffset = params.array[15] + idx + device_offset;
    const int memGateOutOffset = params.array[16] + idx + device_offset;
    const int memGateForgetOffset = params.array[17] + idx + device_offset;
    const int outputOffset = params.array[18] + idx + device_offset;
    const int fitnessOffset = params.array[19] + idx + device_offset;
    const int communityMagOffset = params.array[20] +idx +device_offset;
    const int whenOffset = params.array[21] + idx + device_offset;
    const int howCertainOffset = params.array[22] + idx + device_offset;

    const double avgLatGQuake = globalQuakes[0];
    const double avgLonGQuake = globalQuakes[1];
    const double GQuakeAvgMag = globalQuakes[3];

    const double ansLat = siteData[(int)answers[0]*2];
    const double ansLon = siteData[(int)answers[0]*2+1];
    const int whenAns = (int)answers[1] - hour;

    //if hour is 0, cut fitness in half.
    if(hour == 0)
        Vec.array[fitnessOffset] /= 250;

    //reset values from previous individual.
    //community magnitude is not set, as this needs to be continued.
    for(int i=0; i<params.array[23]; i++){
        Vec.array[whenOffset +i*ind] = 0;
        Vec.array[howCertainOffset +i*ind] =0;
    }
    for(int i=0; i<trainingsize; i++){ // training size is a constant parameter for the size of each timestep

        double CommunityLat = 0;
        double CommunityLon = 0;

        for(int j=0; j<params.array[23]; j++){//sitesWeighted Lat/Lon values are determined based on all previous zsites mag output value.
            CommunityLat += siteData[j*2]*Vec.array[communityMagOffset+j*ind];
            CommunityLon += siteData[j*2+1]*Vec.array[communityMagOffset+j*ind];
        }

        CommunityLat = CommunityLat/params.array[23];
        CommunityLon = CommunityLon/params.array[23];


        for(int j=0; j<params.array[23]; j++){ //each site is run independently of others, but shares an output from the previous step

            const double latSite = siteData[j*2];
            const double lonSite = siteData[j*2+1];
            const double GQuakeAvgdist = distCalc(latSite, lonSite, avgLatGQuake, avgLonGQuake);
            const double GQuakeAvgBearing = bearingCalc(latSite, lonSite, avgLatGQuake, avgLonGQuake);
            const double CommunityDist = distCalc(latSite, lonSite, CommunityLat, CommunityLon);
            const double CommunityBearing = bearingCalc(latSite, lonSite, CommunityLat, CommunityLon);


            /* 3 outputs, 1 with an hour in the future when the earthquake will hit,
                        1 with the porbability of that earthquake happening (between [0,1]) and 1 with the sites magnitude (for community feedback) */
            int n =0; // n is the weight number

            for(int k=0; k<3; k++){
                Vec.array[inputOffset+k*ind] = normalize(inputData[site_offset[j]+channel_offset[k]+i], meanCh.array[k], stdCh.array[k]);//channels 1-3
            }
            Vec.array[inputOffset+3*ind] = shift(GQuakeAvgdist, 80150.2, 0, 1, 0);
            Vec.array[inputOffset+4*ind] = shift(GQuakeAvgBearing, 360, 0, 1, 0);
            Vec.array[inputOffset+5*ind] = shift(GQuakeAvgMag, 10, 0, 1, 0);
            Vec.array[inputOffset+6*ind] = shift(Kp, 10, 0, 1, 0);
            Vec.array[inputOffset+7*ind] = shift(CommunityDist, 80150.2, 0, 1, 0);
            Vec.array[inputOffset+8*ind] = shift(CommunityBearing, 360, 0, 1, 0);
            //            run the neuroCommand order tree
            for(int itr=0; itr< params.array[26]; itr++){//every order is sequential and run after the previous order to massively simplify the workload in this kernel.
                double tmp;
                //set stuff to zero
                if(sharedQueue[itr]._first.def== nounHidden
                        && sharedQueue[itr]._verb.def == verbZero){

                    neuroZero(Vec.array[hiddenOffset+sharedQueue[itr]._first.id*ind]);

                }

                else if(sharedQueue[itr]._first.def == nounMemGateIn
                        && sharedQueue[itr]._verb.def == verbZero){

                    neuroZero(Vec.array[memGateInOffset+sharedQueue[itr]._first.id*ind]);
                }

                else if(sharedQueue[itr]._first.def == nounMemGateOut
                        && sharedQueue[itr]._verb.def == verbZero)

                    neuroZero(Vec.array[memGateOutOffset+sharedQueue[itr]._first.id*ind]);

                else if(sharedQueue[itr]._first.def == nounMemGateForget
                        && sharedQueue[itr]._verb.def == verbZero)

                    neuroZero(Vec.array[memGateForgetOffset+sharedQueue[itr]._first.id*ind]);

                else if(sharedQueue[itr]._first.def == nounMemory
                        && sharedQueue[itr]._verb.def == verbZero)

                    neuroZero(Vec.array[memOffset+sharedQueue[itr]._first.id*ind]);


                else if(sharedQueue[itr]._first.def == nounOutput
                        && sharedQueue[itr]._verb.def == verbZero)

                    neuroZero(Vec.array[outputOffset+sharedQueue[itr]._first.id*ind]);


                //first->second summations
                else if(sharedQueue[itr]._first.def == nounInput
                        && sharedQueue[itr]._second.def == nounHidden
                        && sharedQueue[itr]._verb.def == verbPush){

                    tmp = Vec.array[inputOffset + sharedQueue[itr]._first.id*ind]*Vec.array[weightsOffset+n++*ind];
                    neuroSum(Vec.array[hiddenOffset + sharedQueue[itr]._second.id*ind], tmp);
                }

                else if(sharedQueue[itr]._first.def == nounInput
                        && sharedQueue[itr]._second.def == nounMemGateIn
                        && sharedQueue[itr]._verb.def == verbPush){

                    tmp = Vec.array[inputOffset + sharedQueue[itr]._first.id*ind]*Vec.array[weightsOffset+n++*ind];
                    neuroSum(Vec.array[memGateInOffset + sharedQueue[itr]._second.id*ind], tmp);
                }

                else if(sharedQueue[itr]._first.def == nounInput
                        && sharedQueue[itr]._second.def == nounMemGateOut
                        && sharedQueue[itr]._verb.def == verbPush){

                    tmp = Vec.array[inputOffset + sharedQueue[itr]._first.id*ind]*Vec.array[weightsOffset+n++*ind];
                    neuroSum(Vec.array[memGateOutOffset + sharedQueue[itr]._second.id*ind], tmp);
                }

                else if(sharedQueue[itr]._first.def == nounInput
                        && sharedQueue[itr]._second.def == nounMemGateForget
                        && sharedQueue[itr]._verb.def == verbPush){

                    tmp = Vec.array[inputOffset + sharedQueue[itr]._first.id*ind]*Vec.array[weightsOffset+n++*ind];
                    neuroSum(Vec.array[memGateForgetOffset + sharedQueue[itr]._second.id*ind], tmp);
                }

                else if(sharedQueue[itr]._first.def == nounHidden
                        && sharedQueue[itr]._second.def == nounHidden
                        && sharedQueue[itr]._verb.def == verbPush){

                    tmp = Vec.array[hiddenOffset + sharedQueue[itr]._first.id*ind]*Vec.array[weightsOffset+n++*ind];
                    neuroSum(Vec.array[hiddenOffset + sharedQueue[itr]._second.id*ind], tmp);
                }

                else if(sharedQueue[itr]._first.def == nounHidden
                        && sharedQueue[itr]._second.def == nounMemGateIn
                        && sharedQueue[itr]._verb.def == verbPush){

                    tmp = Vec.array[hiddenOffset + sharedQueue[itr]._first.id*ind]*Vec.array[weightsOffset+n++*ind];
                    neuroSum(Vec.array[memGateInOffset + sharedQueue[itr]._second.id*ind], tmp);
                }

                else if(sharedQueue[itr]._first.def == nounHidden
                        && sharedQueue[itr]._second.def == nounOutput
                        && sharedQueue[itr]._verb.def == verbPush){

                    tmp = Vec.array[hiddenOffset + sharedQueue[itr]._first.id*ind]*Vec.array[weightsOffset+n++*ind];
                    neuroSum(Vec.array[outputOffset + sharedQueue[itr]._second.id*ind], tmp);
                }


                else if(sharedQueue[itr]._first.def == nounHidden
                        && sharedQueue[itr]._second.def == nounMemGateOut
                        && sharedQueue[itr]._verb.def == verbPush){

                    tmp = Vec.array[hiddenOffset + sharedQueue[itr]._first.id*ind]*Vec.array[weightsOffset+n++*ind];
                    neuroSum(Vec.array[memGateOutOffset + sharedQueue[itr]._second.id*ind], tmp);
                }

                else if(sharedQueue[itr]._first.def == nounHidden
                        && sharedQueue[itr]._second.def == nounMemGateForget
                        && sharedQueue[itr]._verb.def == verbPush){

                    Vec.array[hiddenOffset + sharedQueue[itr]._first.id*ind]*Vec.array[weightsOffset+n++*ind];
                    neuroSum(Vec.array[memGateForgetOffset + sharedQueue[itr]._second.id*ind], tmp);
                }


                //memory gates
                else if(sharedQueue[itr]._first.def == nounInput
                        && sharedQueue[itr]._second.def == nounMemory
                        && sharedQueue[itr]._third.def == nounMemGateIn
                        && sharedQueue[itr]._verb.def == verbMemGate){

                    tmp = Vec.array[inputOffset+sharedQueue[itr]._first.id*ind]; // squash inputs so as to not saturate hidden neurons
                    neuroSquash(tmp);

                    neuroMemGate(Vec.array[memGateInOffset+sharedQueue[itr]._third.id*ind],
                            tmp, Vec.array[memOffset+sharedQueue[itr]._second.id*ind], 0.5);
                }

                else if(sharedQueue[itr]._first.def == nounHidden
                        && sharedQueue[itr]._second.def == nounMemory
                        && sharedQueue[itr]._third.def == nounMemGateIn
                        && sharedQueue[itr]._verb.def == verbMemGate){

                    tmp = Vec.array[hiddenOffset+sharedQueue[itr]._first.id*ind];
                    neuroSquash(tmp);

                    neuroMemGate(Vec.array[memGateInOffset+sharedQueue[itr]._third.id*ind],
                            tmp, Vec.array[memOffset+sharedQueue[itr]._second.id*ind], 0.5);
                }

                else if(sharedQueue[itr]._first.def == nounOutput
                        && sharedQueue[itr]._second.def == nounMemory
                        && sharedQueue[itr]._third.def == nounMemGateIn
                        && sharedQueue[itr]._verb.def == verbMemGate){

                    neuroMemGate(Vec.array[memGateInOffset+sharedQueue[itr]._third.id*ind],
                            Vec.array[outputOffset+sharedQueue[itr]._first.id*ind],
                            Vec.array[memOffset + sharedQueue[itr]._second.id*ind], 0.5);
                }

                else if(sharedQueue[itr]._first.def == nounMemory
                        && sharedQueue[itr]._second.def == nounHidden
                        && sharedQueue[itr]._third.def == nounMemGateOut
                        && sharedQueue[itr]._verb.def == verbMemGate){

                    neuroMemGate(Vec.array[memGateOutOffset+sharedQueue[itr]._third.id*ind],
                            Vec.array[memOffset+sharedQueue[itr]._first.id*ind],
                            Vec.array[hiddenOffset+sharedQueue[itr]._second.id*ind], 0.5);
                }

                else if(sharedQueue[itr]._first.def == nounMemory
                        && sharedQueue[itr]._second.def == nounOutput
                        && sharedQueue[itr]._third.def == nounMemGateOut
                        && sharedQueue[itr]._verb.def == verbMemGate){

                    neuroMemGate(Vec.array[memGateOutOffset+sharedQueue[itr]._third.id*ind],
                            Vec.array[memOffset+sharedQueue[itr]._first.id*ind],
                            Vec.array[outputOffset+sharedQueue[itr]._second.id*ind], 0.5);
                }

                else if(sharedQueue[itr]._first.def == nounMemory
                        && sharedQueue[itr]._second.def == nounMemGateForget
                        && sharedQueue[itr]._verb.def == verbMemGate){

                    neuroMemForget(Vec.array[memGateForgetOffset+sharedQueue[itr]._second.id*ind],
                            Vec.array[memOffset + sharedQueue[itr]._first.id*ind], 0.5);
                }

                //bias
                else if(sharedQueue[itr]._first.def == nounBias
                        && sharedQueue[itr]._second.def == nounHidden
                        && sharedQueue[itr]._verb.def == verbPush){
                    tmp = 1*Vec.array[weightsOffset + n++*ind];
                    neuroSum(Vec.array[hiddenOffset + sharedQueue[itr]._second.id*ind], tmp);
                }

                else if(sharedQueue[itr]._first.def == nounBias
                        && sharedQueue[itr]._second.def == nounMemGateIn
                        && sharedQueue[itr]._verb.def == verbPush){
                    tmp = 1*Vec.array[weightsOffset + n++*ind];
                    neuroSum(Vec.array[memGateInOffset + sharedQueue[itr]._second.id*ind], tmp);
                }

                else if(sharedQueue[itr]._first.def == nounBias
                        && sharedQueue[itr]._second.def == nounMemGateOut
                        && sharedQueue[itr]._verb.def == verbPush){
                    tmp = 1*Vec.array[weightsOffset+n++*ind];
                    neuroSum(Vec.array[memGateOutOffset + sharedQueue[itr]._second.id*ind], tmp);
                }

                else if(sharedQueue[itr]._first.def == nounBias
                        && sharedQueue[itr]._second.def == nounMemGateForget
                        && sharedQueue[itr]._verb.def == verbPush){
                    tmp = 1*Vec.array[weightsOffset+n++*ind];
                    neuroSum(Vec.array[memGateForgetOffset + sharedQueue[itr]._second.id*ind], tmp);
                }

                else if(sharedQueue[itr]._first.def == nounBias
                        && sharedQueue[itr]._second.def == nounOutput
                        && sharedQueue[itr]._verb.def == verbPush){
                    tmp = 1*Vec.array[weightsOffset+n++*ind];
                    neuroSum(Vec.array[outputOffset + sharedQueue[itr]._second.id*ind], tmp);
                }

                //squashing
                else if(sharedQueue[itr]._first.def == nounHidden && sharedQueue[itr]._verb.def == verbSquash){
                    neuroSquash(Vec.array[hiddenOffset + sharedQueue[itr]._first.id*ind]);
                }

                else if(sharedQueue[itr]._first.def == nounMemGateIn && sharedQueue[itr]._verb.def == verbSquash){
                    neuroSquash(Vec.array[memGateInOffset + sharedQueue[itr]._first.id*ind]);
                }

                else if(sharedQueue[itr]._first.def == nounMemGateOut && sharedQueue[itr]._verb.def == verbSquash){
                    neuroSquash(Vec.array[memGateOutOffset + sharedQueue[itr]._first.id*ind]);
                }

                else if(sharedQueue[itr]._first.def == nounMemGateForget && sharedQueue[itr]._verb.def == verbSquash){
                    neuroSquash(Vec.array[memGateForgetOffset + sharedQueue[itr]._first.id*ind]);
                }

                else if(sharedQueue[itr]._first.def == nounOutput && sharedQueue[itr]._verb.def == verbSquash){
                    neuroSquash(Vec.array[outputOffset + sharedQueue[itr]._first.id*ind]);
                }

            }

            Vec.array[whenOffset+j*ind] += shift(isnan(Vec.array[outputOffset+0*ind])? 0 : Vec.array[outputOffset+0*ind], 1, -1, 2160, 0);
            Vec.array[howCertainOffset+j*ind] += shift(isnan(Vec.array[outputOffset+1*ind])? 0 : Vec.array[outputOffset+1*ind], 1, -1, 1, 0);
            Vec.array[communityMagOffset+j*ind] =  shift(isnan(Vec.array[outputOffset+2*ind])? 0 : Vec.array[outputOffset+2*ind], 1, -1, 10, 0); // set the next sets communityMag = output #3.
        }
    }
    for(int j=0; j<params.array[23]; j++){ // now lets get the average when and howcertain values.
        Vec.array[whenOffset+j*ind] = Vec.array[whenOffset+j*ind]/trainingsize;
        Vec.array[howCertainOffset+j*ind] = Vec.array[howCertainOffset+j*ind]/trainingsize;
    }
    /*calculate score for this individual during this round, current scoring mechanism is - e^(-(abs(whenGuess-whenAns)+distToCorrectSite)), closer to 1 the better.   */
    double maxCertainty = 0;
    float avgCertainty = 0;
    float whenGuess=0;
    float guessLat=0;
    float guessLon=0;

    for(int j=0; j<params.array[23]; j++){
        avgCertainty += Vec.array[howCertainOffset+j*ind];
        if(Vec.array[howCertainOffset+j*ind] > maxCertainty){
            maxCertainty = Vec.array[howCertainOffset+j*ind];
            whenGuess = Vec.array[whenOffset+j*ind];
            guessLat = siteData[j*2];
            guessLon = siteData[j*2+1];
        }
    }

    double oldFit = isnan(Vec.array[fitnessOffset]) ? 0 : Vec.array[fitnessOffset];
    Vec.array[fitnessOffset] = scoreFunc(whenGuess, whenAns, guessLat, guessLon, ansLat, ansLon, oldFit, hour); //we take the average beacuse consistency is more important than being really good at this particular hour.
}
