#include "hip/hip_runtime.h"
#include <kernelDefs.h>
#include <neuroFunc.h>
#include <utilFunc.h>
//using
extern __constant__ int inputData[];
extern __constant__ double answers[];
extern __constant__ double globalQuakes[];
extern __constant__ double siteData[];
extern __constant__ double Kp;
extern __constant__ int site_offset[];
extern __constant__ int channel_offset[];
extern __constant__ int trainingsize;
//endof using

__global__ void NetKern(kernelArray<double> Vec, kernelArray<int> params, Order* commandQueue, int hour, kernelArray<double> meanCh,
                        kernelArray<double> stdCh, size_t device_offset){
    extern __shared__  Order sharedQueue[];
    const int tix = threadIdx.x;
    if(tix < params.array[26]){
        sharedQueue[tix].first = commandQueue[tix].first;
        sharedQueue[tix].second = commandQueue[tix].second;
    }
    __syncthreads();
    const int idx = blockIdx.x * blockDim.x + threadIdx.x; // for each thread is one individual
    const int ind = params.array[10];

    const int weightsOffset = params.array[11] + idx + device_offset;
    const int inputOffset = params.array[12] + idx + device_offset;
    const int hiddenOffset = params.array[13] + idx + device_offset;
    const int memOffset = params.array[14] + idx + device_offset;
    const int memGateInOffset = params.array[15] + idx + device_offset;
    const int memGateOutOffset = params.array[16] + idx + device_offset;
    const int memGateForgetOffset = params.array[17] + idx + device_offset;
    const int outputOffset = params.array[18] + idx + device_offset;
    const int fitnessOffset = params.array[19] + idx + device_offset;
    const int communityMagOffset = params.array[20] +idx +device_offset;
    const int whenMinOffset = params.array[21] + idx + device_offset;
    const int howCertainOffset = params.array[22] + idx + device_offset;
    const int whenMaxOffset = params.array[25] + idx + device_offset;

    const double avgLatGQuake = globalQuakes[0];
    const double avgLonGQuake = globalQuakes[1];
    const double GQuakeAvgMag = globalQuakes[3];


    const double ansLat = siteData[(int)answers[0]*2];
    const double ansLon = siteData[(int)answers[0]*2+1];
    const int whenAns = (int)answers[1] - hour;

    //reset values from previous individual.
    //community magnitude is not set, as this needs to be continued.
    for(int i=0; i<params.array[23]; i++){
        Vec.array[whenMinOffset +i*ind] = 0;
        Vec.array[whenMaxOffset +i*ind] =0;
        Vec.array[howCertainOffset +i*ind] =0;
    }
    for(int i=0; i<trainingsize; i++){ // training size is a constant parameter for the size of each timestep

        double CommunityLat = 0;
        double CommunityLon = 0;

        for(int j=0; j<params.array[23]; j++){//sitesWeighted Lat/Lon values are determined based on all previous zsites mag output value.
            CommunityLat += siteData[j*2]*Vec.array[communityMagOffset+j*ind];
            CommunityLon += siteData[j*2+1]*Vec.array[communityMagOffset+j*ind];
        }

        CommunityLat = CommunityLat/params.array[23];
        CommunityLon = CommunityLon/params.array[23];


        for(int j=0; j<params.array[23]; j++){ //each site is run independently of others, but shares an output from the previous step

            const double latSite = siteData[j*2];
            const double lonSite = siteData[j*2+1];
            const double GQuakeAvgdist = distCalc(latSite, lonSite, avgLatGQuake, avgLonGQuake);
            const double GQuakeAvgBearing = bearingCalc(latSite, lonSite, avgLatGQuake, avgLonGQuake);
            const double CommunityDist = distCalc(latSite, lonSite, CommunityLat, CommunityLon);
            const double CommunityBearing = bearingCalc(latSite, lonSite, CommunityLat, CommunityLon);


            /* 3 outputs, 1 with an hour in the future when the earthquake will hit,
                        1 with the porbability of that earthquake happening (between [0,1]) and 1 with the sites magnitude (for community feedback) */
            int n =0; // n is the weight number

            for(int k=0; k<3; k++){
                Vec.array[inputOffset+k*ind] = normalize(inputData[site_offset[j]+channel_offset[k]+i], meanCh.array[k], stdCh.array[k]);//channels 1-3
            }

            Vec.array[inputOffset+3*ind] = shift(GQuakeAvgdist, 40075.1, 0);
            Vec.array[inputOffset+4*ind] = shift(GQuakeAvgBearing, 360, 0);
            Vec.array[inputOffset+5*ind] = shift(GQuakeAvgMag, 9.5, 0);
            Vec.array[inputOffset+6*ind] = shift(Kp, 10, 0);
            Vec.array[inputOffset+7*ind] = shift(CommunityDist,40075.1, 0);
            Vec.array[inputOffset+8*ind] = shift(CommunityBearing, 360, 0);
            //run the neuroCommand order tree
            for(int itr=0; itr< params.array[26]; itr++){//every order is sequential and run after the previous order to massively simplify the workload in this kernel.
                double tmp;
                //set stuff to zero
                if(sharedQueue[itr].first.def == typeHidden && sharedQueue[itr].second.def == typeZero){
                    neuroZero(Vec.array[hiddenOffset+sharedQueue[itr].first.id*ind]);

                }

                else if(sharedQueue[itr].first.def == typeMemGateIn && sharedQueue[itr].second.def == typeZero){
                    neuroZero(Vec.array[memGateInOffset+sharedQueue[itr].first.id*ind]);

                }

                else if(sharedQueue[itr].first.def == typeMemGateOut && sharedQueue[itr].second.def == typeZero){
                    neuroZero(Vec.array[memGateOutOffset+sharedQueue[itr].first.id*ind]);

                }

                else if(sharedQueue[itr].first.def == typeMemGateForget && sharedQueue[itr].second.def == typeZero){
                    neuroZero(Vec.array[memGateForgetOffset+sharedQueue[itr].first.id*ind]);

                }

                else if(sharedQueue[itr].first.def == typeMemory && sharedQueue[itr].second.def == typeZero){
                    neuroZero(Vec.array[memOffset+sharedQueue[itr].first.id*ind]);

                }

                else if(sharedQueue[itr].first.def == typeOutput && sharedQueue[itr].second.def == typeZero){
                    neuroZero(Vec.array[outputOffset+sharedQueue[itr].first.id*ind]);

                }

                //first->second summations
                else if(sharedQueue[itr].first.def == typeInput && sharedQueue[itr].second.def == typeHidden){
                    tmp = Vec.array[inputOffset + sharedQueue[itr].first.id*ind]*Vec.array[weightsOffset+n++*ind];
                    neuroSum(Vec.array[hiddenOffset + sharedQueue[itr].second.id*ind], tmp);

                }

                else if(sharedQueue[itr].first.def == typeInput && sharedQueue[itr].second.def == typeMemGateIn){
                    tmp = Vec.array[inputOffset + sharedQueue[itr].first.id*ind]*Vec.array[weightsOffset+n++*ind];
                    neuroSum(Vec.array[memGateInOffset + sharedQueue[itr].second.id*ind], tmp);

                }

                else if(sharedQueue[itr].first.def == typeInput && sharedQueue[itr].second.def == typeMemGateOut){
                    tmp = Vec.array[inputOffset + sharedQueue[itr].first.id*ind]*Vec.array[weightsOffset+n++*ind];
                    neuroSum(Vec.array[memGateOutOffset + sharedQueue[itr].second.id*ind], tmp);

                }

                else if(sharedQueue[itr].first.def == typeInput && sharedQueue[itr].second.def == typeMemGateForget){
                    tmp = Vec.array[inputOffset + sharedQueue[itr].first.id*ind]*Vec.array[weightsOffset+n++*ind];
                    neuroSum(Vec.array[memGateForgetOffset + sharedQueue[itr].second.id*ind], tmp);

                }

                else if(sharedQueue[itr].first.def == typeHidden && sharedQueue[itr].second.def == typeHidden){
                    tmp = Vec.array[hiddenOffset + sharedQueue[itr].first.id*ind]*Vec.array[weightsOffset+n++*ind];
                    neuroSum(Vec.array[hiddenOffset + sharedQueue[itr].second.id*ind], tmp);

                }

                else if(sharedQueue[itr].first.def == typeHidden && sharedQueue[itr].second.def == typeMemGateIn){
                    tmp = Vec.array[hiddenOffset + sharedQueue[itr].first.id*ind]*Vec.array[weightsOffset+n++*ind];
                    neuroSum(Vec.array[memGateInOffset + sharedQueue[itr].second.id*ind], tmp);

                }

                else if(sharedQueue[itr].first.def == typeHidden && sharedQueue[itr].second.def == typeOutput){
                    tmp = Vec.array[hiddenOffset + sharedQueue[itr].first.id*ind]*Vec.array[weightsOffset+n++*ind];
                    neuroSum(Vec.array[outputOffset + sharedQueue[itr].second.id*ind], tmp);

                }


                else if(sharedQueue[itr].first.def == typeHidden && sharedQueue[itr].second.def == typeMemGateOut){
                    tmp = Vec.array[hiddenOffset + sharedQueue[itr].first.id*ind]*Vec.array[weightsOffset+n++*ind];
                    neuroSum(Vec.array[memGateOutOffset + sharedQueue[itr].second.id*ind], tmp);

                }

                else if(sharedQueue[itr].first.def == typeHidden && sharedQueue[itr].second.def == typeMemGateForget){
                    Vec.array[hiddenOffset + sharedQueue[itr].first.id*ind]*Vec.array[weightsOffset+n++*ind];
                    neuroSum(Vec.array[memGateForgetOffset + sharedQueue[itr].second.id*ind], tmp);

                }


                //memory gates
                else if(sharedQueue[itr].first.def == typeInput && sharedQueue[itr].second.def == typeMemory && sharedQueue[itr].third.def == typeMemGateIn){

                    tmp = Vec.array[inputOffset+sharedQueue[itr].first.id*ind]; // squash inputs so as to not saturate hidden neurons
                    neuroSquash(tmp);

                    neuroMemGate(Vec.array[memGateInOffset+sharedQueue[itr].third.id*ind], tmp, Vec.array[memOffset+sharedQueue[itr].second.id*ind], 0.5);
                }

                else if(sharedQueue[itr].first.def == typeHidden && sharedQueue[itr].second.def == typeMemory && sharedQueue[itr].third.def == typeMemGateIn){

                    tmp = Vec.array[hiddenOffset+sharedQueue[itr].first.id*ind];
                    neuroSquash(tmp);

                    neuroMemGate(Vec.array[memGateInOffset+sharedQueue[itr].third.id*ind], tmp, Vec.array[memOffset+sharedQueue[itr].second.id*ind], 0.5);
                }

                else if(sharedQueue[itr].first.def == typeOutput && sharedQueue[itr].second.def == typeMemory && sharedQueue[itr].third.def == typeMemGateIn){
                    neuroMemGate(Vec.array[memGateInOffset+sharedQueue[itr].third.id*ind],
                            Vec.array[outputOffset+sharedQueue[itr].first.id*ind],
                            Vec.array[memOffset + sharedQueue[itr].second.id*ind], 0.5);
                }

                else if(sharedQueue[itr].first.def == typeMemory && sharedQueue[itr].second.def == typeHidden && sharedQueue[itr].third.def == typeMemGateOut){
                    neuroMemGate(Vec.array[memGateOutOffset+sharedQueue[itr].third.id*ind],
                            Vec.array[memOffset+sharedQueue[itr].first.id*ind],
                            Vec.array[hiddenOffset+sharedQueue[itr].second.id*ind], 0.5);
                }

                else if(sharedQueue[itr].first.def == typeMemory && sharedQueue[itr].second.def == typeOutput && sharedQueue[itr].third.def == typeMemGateOut){
                    neuroMemGate(Vec.array[memGateOutOffset+sharedQueue[itr].third.id*ind],
                            Vec.array[memOffset+sharedQueue[itr].first.id*ind],
                            Vec.array[outputOffset+sharedQueue[itr].second.id*ind], 0.5);
                }

                else if(sharedQueue[itr].first.def == typeMemory && sharedQueue[itr].second.def == typeMemGateForget){
                    neuroMemForget(Vec.array[memGateForgetOffset+sharedQueue[itr].second.id*ind],
                            Vec.array[memOffset + sharedQueue[itr].first.id*ind], 0.5);
                }

                //bias
                else if(sharedQueue[itr].first.def == typeBias && sharedQueue[itr].second.def == typeHidden){
                    tmp = 1*Vec.array[weightsOffset + n++*ind];
                    neuroSum(Vec.array[hiddenOffset + sharedQueue[itr].second.id*ind], tmp);
                }

                else if(sharedQueue[itr].first.def == typeBias && sharedQueue[itr].second.def == typeMemGateIn){
                    tmp = 1*Vec.array[weightsOffset + n++*ind];
                    neuroSum(Vec.array[memGateInOffset + sharedQueue[itr].second.id*ind], tmp);
                }

                else if(sharedQueue[itr].first.def == typeBias && sharedQueue[itr].second.def == typeMemGateOut){
                    tmp = 1*Vec.array[weightsOffset+n++*ind];
                    neuroSum(Vec.array[memGateOutOffset + sharedQueue[itr].second.id*ind], tmp);
                }

                else if(sharedQueue[itr].first.def == typeBias && sharedQueue[itr].second.def == typeMemGateForget){
                    tmp = 1*Vec.array[weightsOffset+n++*ind];
                    neuroSum(Vec.array[memGateForgetOffset + sharedQueue[itr].second.id*ind], tmp);
                }

                else if(sharedQueue[itr].first.def == typeBias && sharedQueue[itr].second.def == typeOutput){
                    tmp = 1*Vec.array[weightsOffset+n++*ind];
                    neuroSum(Vec.array[outputOffset + sharedQueue[itr].second.id*ind], tmp);
                }

                //squashing
                else if(sharedQueue[itr].first.def == typeHidden && sharedQueue[itr].second.def == typeSquash){
                    neuroSquash(Vec.array[hiddenOffset + sharedQueue[itr].first.id*ind]);
                }

                else if(sharedQueue[itr].first.def == typeMemGateIn && sharedQueue[itr].second.def == typeSquash){
                    neuroSquash(Vec.array[memGateInOffset + sharedQueue[itr].first.id*ind]);
                }

                else if(sharedQueue[itr].first.def == typeMemGateOut && sharedQueue[itr].second.def == typeSquash){
                    neuroSquash(Vec.array[memGateOutOffset + sharedQueue[itr].first.id*ind]);
                }

                else if(sharedQueue[itr].first.def == typeMemGateForget && sharedQueue[itr].second.def == typeSquash){
                    neuroSquash(Vec.array[memGateForgetOffset + sharedQueue[itr].first.id*ind]);
                }

                else if(sharedQueue[itr].first.def == typeOutput && sharedQueue[itr].second.def == typeSquash){
                    neuroSquash(Vec.array[outputOffset + sharedQueue[itr].first.id*ind]);
                }

            }

            Vec.array[whenMinOffset+j*ind] += shift(Vec.array[outputOffset+0*ind], 2160, 0); // nv = ((ov - omin)*(nmax-nmin) / (omax - omin))+nmin
            Vec.array[whenMaxOffset+j*ind] += shift(Vec.array[outputOffset+1*ind], 2160, 0);
            Vec.array[howCertainOffset+j*ind] += Vec.array[outputOffset+2*ind];
            Vec.array[communityMagOffset+j*ind] =  Vec.array[outputOffset+3*ind]; // set the next sets communityMag = output #3.
        }
    }
    for(int j=0; j<params.array[23]; j++){ // now lets get the average when and howcertain values.
        Vec.array[whenMinOffset+j*ind] = Vec.array[whenMinOffset+j*ind]/trainingsize;
        Vec.array[whenMaxOffset+j*ind] = Vec.array[whenMaxOffset+j*ind]/trainingsize;
        Vec.array[howCertainOffset+j*ind] = Vec.array[howCertainOffset+j*ind]/trainingsize;
    }
    /*calculate score for this individual during this round, current scoring mechanism is - e^(-(abs(whenGuess-whenAns)+distToCorrectSite)), closer to 1 the better.   */
    double maxCertainty=0;
    double whenMinGuess=0;
    double whenMaxGuess=0;
    double guessLat=0;
    double guessLon=0;

    for(int j=0; j<params.array[23]; j++){
        if(Vec.array[howCertainOffset+j*ind] > maxCertainty){
            maxCertainty = Vec.array[howCertainOffset+j*ind];
            whenMinGuess = Vec.array[whenMinOffset+j*ind];
            whenMaxGuess = Vec.array[whenMaxOffset+j*ind];
            guessLat = siteData[j*2];
            guessLon = siteData[j*2+1];
        }
    }

    Vec.array[fitnessOffset] = scoreFunc(whenMinGuess, whenMaxGuess, whenAns, guessLat, guessLon, ansLat, ansLon); //we take the average beacuse consistency is more important than being really good at this particular hour.
}
