#include "hip/hip_runtime.h"
#include <kernelDefs.h>
//using
extern __constant__ int input[];
extern __constant__ double answers[];
extern __constant__ double globalQuakes[];
extern __constant__ double siteData[];
extern __constant__ double Kp;
extern __constant__ int site_offset[];
extern __constant__ int channel_offset[];
extern __constant__ int trainingsize;
//endof using

__global__ void NetKern(kernelArray<double> Vec, kernelArray<int> params,  kernelArray<std::pair<const int, const int> > connections,
                        int hour, kernelArray<double> meanCh, kernelArray<double> stdCh, size_t device_offset){
    extern __shared__ std::pair<int, int> shdConnect[];
    if(threadIdx.x ==0){
        for(int i=0; i<connections.size; i++){
            shdConnect[i].first = connections.array[i].first;
            shdConnect[i].second = connections.array[i].second;
        }
    }
    __syncthreads();
    const int idx = blockIdx.x * blockDim.x + threadIdx.x; // for each thread is one individual
    typedef std::pair<const int, const int>*  connectPairMatrix;
    const int ind = params.array[10]; // number of individuals on device
    const int weightsOffset = params.array[11] + idx + device_offset;
    const int inputOffset = params.array[12] + idx + device_offset; // 6 is the offset to the start of the input neurons
    const int hiddenOffset = params.array[13] + idx + device_offset;
    const int memOffset = params.array[14] + idx + device_offset;
    const int memGateInOffset = params.array[15] + idx + device_offset;
    const int memGateOutOffset = params.array[16] + idx + device_offset;
    const int memGateForgetOffset = params.array[17] + idx + device_offset;
    const int outputOffset = params.array[18] + idx + device_offset;
    const int fitnessOffset = params.array[19] + idx + device_offset;
    const int communityMagOffset = params.array[20] +idx +device_offset;
    const int whenOffset = params.array[21] + idx + device_offset;
    const int howCertainOffset = params.array[22] + idx + device_offset;

    //for connections --imagined offsets if we strided instead of interleaved
    const int connInputOffset = 0;
    const int connHiddenOffset = params.array[3] + connInputOffset;
    const int connMemOffset = params.array[4] + connHiddenOffset;
    const int connMemInOffset = params.array[5] + connMemOffset;
    const int connMemOutOffset = params.array[6] + connMemInOffset;
    const int connMemForgetOffset = params.array[7] + connMemOutOffset;
    const int connOutputOffset = params.array[8] + connMemForgetOffset;
    //reset values from previous individual.
    //community magnitude is not set, as this needs to be continued.
    for(int i=0; i<params.array[23]; i++){
        Vec.array[whenOffset +i*ind] = 0;
        Vec.array[howCertainOffset +i*ind] =0;
    }

    for(int i=0; i<trainingsize; i++){
        float CommunityLat = 0;
        float CommunityLon = 0;
        for(int j=0; j<params.array[23]; j++){//sitesWeighted Lat/Lon values are determined based on all previous zsites mag output value.
            CommunityLat += siteData[j*2]*Vec.array[communityMagOffset+j*ind];
            CommunityLon += siteData[j*2+1]*Vec.array[communityMagOffset+j*ind];
        }
        CommunityLat = CommunityLat/params.array[23];
        CommunityLon = CommunityLon/params.array[23];
        for(int j=0; j<params.array[23]; j++){ //each site is run independently of others, but shares an output from the previous step

            float latSite = siteData[j*2];
            float lonSite = siteData[j*2+1];
            float avgLatGQuake = globalQuakes[0];
            float avgLonGQuake = globalQuakes[1];
            float GQuakeAvgMag = globalQuakes[3];
            float GQuakeAvgdist = distCalc(latSite, lonSite, avgLatGQuake, avgLonGQuake);
            float GQuakeAvgBearing = bearingCalc(latSite, lonSite, avgLatGQuake, avgLonGQuake);
            float CommunityDist = distCalc(latSite, lonSite, CommunityLat, CommunityLon);
            float CommunityBearing = bearingCalc(latSite, lonSite, CommunityLat, CommunityLon);
            /* 3 outputs, 1 with an hour in the future when the earthquake will hit,
                        1 with the porbability of that earthquake happening (between [0,1]) and 1 with the sites magnitude (for community feedback) */
            int n =0; // n is the weight number
            for(int k=0; k<3; k++){
                Vec.array[inputOffset+k*ind] = normalize(input[site_offset[j]+channel_offset[k]+i], meanCh.array[k], stdCh.array[k]);//channel 1
            }
            Vec.array[inputOffset+3*ind] = shift(GQuakeAvgdist, 40075.1, 0);
            Vec.array[inputOffset+4*ind] = shift(GQuakeAvgBearing, 360, 0);
            Vec.array[inputOffset+5*ind] = shift(GQuakeAvgMag, 9.5, 0);
            Vec.array[inputOffset+6*ind] = shift(Kp, 10, 0);
            Vec.array[inputOffset+7*ind] = shift(CommunityDist,40075.1, 0);
            Vec.array[inputOffset+8*ind] = shift(CommunityBearing, 360, 0);
            //lets reset all neuron values for this new timestep (except memory neurons)
            for(int gate=0; gate<params.array[5]; gate++){
                Vec.array[memGateInOffset+gate*ind] = 0;
                Vec.array[memGateOutOffset+gate*ind] = 0;
                Vec.array[memGateForgetOffset+gate*ind] = 0;
            }
            for(int hid=0; hid<params.array[4]; hid++){
                Vec.array[hiddenOffset+hid*ind] = 0;
            }
            for(int out=0; out<params.array[9]; out++){
                Vec.array[outputOffset+out*ind] = 0;
            }

            //now that everything that should be zeroed is zeroed, lets start the network.
            //mem gates & LSTM nodes --
            for(int gate = 0; gate<params.array[5]; gate++){//calculate memory gate node values, you can connect inputs & hidden neurons to them.
                for(int pair=0; pair<connections.size; pair++){//for memGateIn
                    if(shdConnect[pair].second == gate+connMemInOffset && shdConnect[pair].first < connHiddenOffset){ //for inputs
                        Vec.array[memGateInOffset+gate*ind] += Vec.array[inputOffset+(shdConnect[pair].first)*ind]*Vec.array[weightsOffset + (n++)*ind]; // memGateIn vect starts at 0
                    }
                    else if(shdConnect[pair].second == gate+connMemInOffset && shdConnect[pair].first > connHiddenOffset && shdConnect[pair].first <connMemOffset){//for hidden neurons
                        Vec.array[memGateInOffset+gate*ind] += Vec.array[hiddenOffset+(shdConnect[pair].first)*ind]*Vec.array[weightsOffset + (n++)*ind];
                    }
                }
                for(int pair=0; pair<connections.size; pair++){//for memGateOut
                    if(shdConnect[pair].second == gate+connMemOutOffset && shdConnect[pair].first < connHiddenOffset){//for inputs
                        Vec.array[memGateOutOffset+gate*ind] += Vec.array[inputOffset+(shdConnect[pair].first)*ind]*Vec.array[weightsOffset + (n++)*ind];
                    }
                    else if(shdConnect[pair].second == gate+connMemOutOffset && shdConnect[pair].first > connHiddenOffset && shdConnect[pair].first < connMemOffset){//for hidden neurons
                        Vec.array[memGateOutOffset+gate*ind] += Vec.array[hiddenOffset+(shdConnect[pair].first)*ind]*Vec.array[weightsOffset + (n++)*ind];
                    }
                }
                for(int pair=0; pair<connections.size; pair++){//for  memGateForget
                    if(shdConnect[pair].second == gate+connMemForgetOffset && shdConnect[pair].first < connHiddenOffset){//for inputs
                        Vec.array[memGateForgetOffset+gate*ind] += Vec.array[inputOffset+(shdConnect[pair].first-inputOffset)*ind]*Vec.array[weightsOffset + (n++)*ind];
                    }
                    else if(shdConnect[pair].second == gate+connMemForgetOffset && shdConnect[pair].first >connHiddenOffset && shdConnect[pair].first <connMemOffset){//for hidden neurons
                        Vec.array[memGateForgetOffset+gate*ind] += Vec.array[hiddenOffset+(shdConnect[pair].first)*ind]*Vec.array[weightsOffset + (n++)*ind];
                    }
                }
                Vec.array[memGateInOffset+gate*ind] = ActFunc(Vec.array[memGateInOffset+gate*ind]);
                Vec.array[memGateOutOffset+gate*ind] = ActFunc(Vec.array[memGateOutOffset+gate*ind]);
                Vec.array[memGateForgetOffset+gate*ind] = ActFunc(Vec.array[memGateForgetOffset+gate*ind]);
            }
            //since we calculated the values for memGateIn and memGateOut, and MemGateForget..
            for (int gate = 0; gate<params.array[5]; gate++){ // if memGateIn is greater than 0.3, then let mem = the sum inputs attached to memGateIn
                if(Vec.array[memGateInOffset+gate*ind] > 0.5){ //gate -memGateInOffset = [0, num of mem neurons]
                    for(int pair=0; pair<connections.size; pair++){
                        if(shdConnect[pair].second == gate+connMemInOffset && shdConnect[pair].first < gate+connHiddenOffset){//only pass inputs
                            Vec.array[memOffset+gate*ind] += Vec.array[inputOffset+(shdConnect[pair].first)*ind]; // no Vec attached, but the old value stored here is not removed.
                        }
                    }
                }
                if(Vec.array[memGateForgetOffset+gate*ind] > 0.5){// if memGateForget is greater than 0.5, then tell mem to forget
                    Vec.array[memOffset+gate*ind] = 0;
                }
                //if memGateForget fires, then memGateOut will output nothing.
                if(Vec.array[memGateOutOffset+gate*ind] > 0.5){//if memGateOut is greater than 0.3, let the nodes mem is connected to recieve mem
                    for(int pair=0; pair<connections.size; pair++){
                        if(shdConnect[pair].first == gate+connMemOffset){// since mem node: memIn node : memOut node = 1:1:1, we can do this.
                            Vec.array[hiddenOffset+(shdConnect[pair].second)*ind] += Vec.array[memOffset+gate*ind];
                        }
                    }
                }
            }

            // hidden neuron nodes --
            for(int hid=0; hid<params.array[4]; hid++){ // for all hidden neurons at layer 1, lets sum the inputs, the memory values were already added.
                for(int pair=0; pair<connections.size; pair++){ // Add the inputs to the hidden neurons
                    if(shdConnect[pair].second == hid+connHiddenOffset && shdConnect[pair].first < connHiddenOffset && shdConnect[pair].first >= connInputOffset){ // if an input connects with this hidden neuron
                        Vec.array[hiddenOffset+hid*ind] += Vec.array[inputOffset+(shdConnect[pair].first)*ind]*Vec.array[weightsOffset + (n++)*ind];
                    }
                }
                for(int pair=0; pair<connections.size; pair++){//add other hidden neuron inputs to each hidden neuron (if applicable)
                    if(shdConnect[pair].second == hid+connHiddenOffset && shdConnect[pair].first < connMemOffset && shdConnect[pair].first >= connHiddenOffset){
                        Vec.array[hiddenOffset+hid*ind] += Vec.array[hiddenOffset+(shdConnect[pair].first)*ind]*Vec.array[weightsOffset + (n++)*ind];
                    }
                }
                Vec.array[hiddenOffset+hid*ind] += 1*Vec.array[weightsOffset + (n++)*ind]; // add bias
                Vec.array[hiddenOffset+hid*ind] = ActFunc(Vec.array[hiddenOffset+hid*ind]); // then squash static_cast<std::pair<const int, const int> >(*it).
            }
            //output nodes --

            for(int out =0; out<params.array[9]; out++){// add hidden neurons to the output nodes
                for(int pair=0; pair<connections.size; pair++){
                    if(shdConnect[pair].second == out+connOutputOffset){
                        Vec.array[outputOffset+out*ind] += Vec.array[hiddenOffset+(shdConnect[pair].first)*ind]*Vec.array[weightsOffset + (n++)*ind];
                    }
                }
                Vec.array[outputOffset+out*ind] += 1*Vec.array[weightsOffset + (n++)*ind]; // add bias
                Vec.array[outputOffset+out*ind] = ActFunc(Vec.array[outputOffset+out*ind]);// then squash static_cast<std::pair<const int, const int> >(*it).
            }

            Vec.array[whenOffset+j*ind] += Vec.array[outputOffset+0*ind]*((2160-hour)-hour)+2160-hour; // nv = ((ov - omin)*(nmax-nmin) / (omax - omin))+nmin
            Vec.array[howCertainOffset+j*ind] += Vec.array[outputOffset+1*ind];
            Vec.array[communityMagOffset+j*ind] =  Vec.array[outputOffset+2*ind]; // set the next sets communityMag = output #3.
        }
    }
    for(int j=0; j<params.array[23]; j++){ // now lets get the average when and howcertain values.
        Vec.array[whenOffset+j*ind] = Vec.array[whenOffset+j*ind]/trainingsize;
        Vec.array[howCertainOffset+j*ind] = Vec.array[howCertainOffset+j*ind]/trainingsize;
    }
    /*calculate performance for this individual - score = 1/(abs(whenGuess-whenReal)*distToQuake), for whenGuess = Vec.array[whenOffset+j] where HowCertain is max for set.
    distToQuake is from the current sites parameters, it emphasizes higher scores for the closest site, a smaller distance is a higher score. */
    float maxCertainty=0;
    float whenGuess=0;
    float guessLat=0;
    float guessLon=0;
    for(int j=0; j<params.array[23]; j++){
        if(Vec.array[howCertainOffset+j*ind] > maxCertainty){
            maxCertainty = Vec.array[howCertainOffset+j*ind];
            whenGuess = Vec.array[whenOffset+j*ind];
            guessLat = siteData[j*2];
            guessLon = siteData[j*2+1];
        }
    }
    float ansLat = siteData[(int)answers[0]*2];
    float ansLon = siteData[(int)answers[0]*2+1];
    float whenAns = answers[1];
    Vec.array[fitnessOffset] = scoreFunc(whenGuess, whenAns, hour, guessLat, guessLon, ansLat, ansLon);//larger is better, negative numbers are impossible.
}
