#include "hip/hip_runtime.h"
#include <kernelDefs.h>
__constant__ int input[20*3*40];
__constant__ int site_offset[20];
__constant__ int channel_offset[3];

__global__ void NetKern(kernelArray<double> Vec, kernelArray<int> params, kernelArray<double> globalQuakes,
                        kernelArray<double> siteData, kernelArray<double> answers, kernelArray<std::pair<const int, const int> > connections, double Kp,int numOfSites,
                        int hour, kernelArray<double> meanCh, kernelArray<double> stdCh, size_t device_offset){

    const int idx = blockIdx.x * blockDim.x + threadIdx.x; // for each thread is one individual
    typedef std::pair<const int, const int>*  connectPairMatrix;
    const int ind = params.array[10]; // number of individuals on device
    const int startOfInput = params.array[12] + idx + device_offset; // 6 is the offset to the start of the input neurons
    const int startOfHidden = params.array[13] + idx + device_offset;
    const int startOfMem = params.array[14] + idx+ device_offset;
    const int startOfMemGateIn = params.array[15] + idx + device_offset;
    const int startOfMemGateOut = params.array[16] + idx + device_offset;
    const int startOfMemGateForget = params.array[17] + idx + device_offset;
    const int startOfOutput = params.array[18] + idx + device_offset;
    const int startOfFitness = params.array[19] + idx + device_offset;
    const int startOfCommunityMag = params.array[20] +idx +device_offset;
    const int startOfWhen = params.array[21] + idx + device_offset;
    const int startOfHowCertain = params.array[22] + idx + device_offset;
    for(int i=0; i<numOfSites; i++){
        Vec.array[startOfWhen+i*ind]=0;
        Vec.array[startOfHowCertain+i*ind]=0;
        Vec.array[startOfCommunityMag+i*ind]=1;
    }
    for(int i=0; i<trainingsize; i++){
        float CommunityLat = 0;
        float CommunityLon = 0;
        for(int j=0; j<numOfSites; j++){//sitesWeighted Lat/Lon values are determined based on all previous zsites mag output value.
            CommunityLat += siteData.array[j*2]*Vec.array[startOfCommunityMag+j*ind];
            CommunityLon += siteData.array[j*2+1]*Vec.array[startOfCommunityMag+j*ind];
        }
        CommunityLat = CommunityLat/numOfSites;
        CommunityLon = CommunityLon/numOfSites;
        for(int j=0; j<numOfSites; j++){ //each site is run independently of others, but shares an output from the previous step

            float  latSite = siteData.array[j*2];
            float lonSite = siteData.array[j*2+1];
            float avgLatGQuake = globalQuakes.array[0];
            float avgLonGQuake = globalQuakes.array[1];
            float GQuakeAvgMag = globalQuakes.array[3];
            float GQuakeAvgdist = distCalc(latSite, lonSite, avgLatGQuake, avgLonGQuake);
            float GQuakeAvgBearing = bearingCalc(latSite, lonSite, avgLatGQuake, avgLonGQuake);
            float CommunityDist = distCalc(latSite, lonSite, CommunityLat, CommunityLon);
            float CommunityBearing = bearingCalc(latSite, lonSite, CommunityLat, CommunityLon);
            /* 3 outputs, 1 with an hour in the future when the earthquake will hit,
                        1 with the porbability of that earthquake happening (between [0,1]) and 1 with the sites magnitude (for community feedback) */
            int n =0; // n is the weight number
            for(int k=0; k<3; k++){
                Vec.array[startOfInput+k*ind] = normalize(input[site_offset[j]+channel_offset[k]+i], meanCh.array[k], stdCh.array[k]);//channel 1
            }
            Vec.array[startOfInput+3*ind] = shift(GQuakeAvgdist, 40075.1, 0);
            Vec.array[startOfInput+4*ind] = shift(GQuakeAvgBearing, 360, 0);
            Vec.array[startOfInput+5*ind] = shift(GQuakeAvgMag, 9.5, 0);
            Vec.array[startOfInput+6*ind] = shift(Kp, 10, 0);
            Vec.array[startOfInput+7*ind] = shift(CommunityDist,40075.1, 0);
            Vec.array[startOfInput+8*ind] = shift(CommunityBearing, 360, 0);
            //lets reset all neuron values for this new timestep (except memory neurons)
            for(int gate=0; gate<params.array[5]; gate++){
                Vec.array[startOfMemGateIn+gate*ind] = 0;
                Vec.array[startOfMemGateOut+gate*ind] = 0;
                Vec.array[startOfMemGateForget+gate*ind] = 0;
            }
            for(int hid=0; hid<params.array[4]; hid++){
                Vec.array[startOfHidden+hid*ind] = 0;
            }
            for(int out=0; out<params.array[9]; out++){
                Vec.array[startOfOutput+out*ind] = 0;
            }

            //now that everything that should be zeroed is zeroed, lets start the network.
            //mem gates & LSTM nodes --
            for(int gate = 0; gate<params.array[5]; gate++){//calculate memory gate node values, you can connect inputs & hidden neurons to them.
                for(connectPairMatrix it = connections.array; it!= connections.array + connections.size; ++it){//for memGateIn
                    if(static_cast<std::pair<const int, const int> >(*it).second == gate+startOfMemGateIn && static_cast<std::pair<const int, const int> >(*it).second < startOfHidden){ //for inputs
                        Vec.array[startOfMemGateIn+gate*ind] += Vec.array[startOfInput+(static_cast<std::pair<const int, const int> >(*it).first-startOfInput)*ind]*Vec.array[(n++)*ind]; // memGateIn vect starts at 0
                    }
                    else if(static_cast<std::pair<const int, const int> >(*it).second == gate+startOfMemGateIn && static_cast<std::pair<const int, const int> >(*it).second >startOfHidden && static_cast<std::pair<const int, const int> >(*it).second <startOfMem){//for hidden neurons
                        Vec.array[startOfMemGateIn+gate*ind] += Vec.array[startOfHidden+(static_cast<std::pair<const int, const int> >(*it).first-startOfHidden)*ind]*Vec.array[(n++)*ind];
                    }
                }
                for(connectPairMatrix it = connections.array; it!= connections.array + connections.size; ++it){//for memGateOut
                    if(static_cast<std::pair<const int, const int> >(*it).second == gate+startOfMemGateOut && static_cast<std::pair<const int, const int> >(*it).second < startOfHidden){//for inputs
                        Vec.array[startOfMemGateOut+gate*ind] += Vec.array[startOfInput+(static_cast<std::pair<const int, const int> >(*it).first-startOfInput)*ind]*Vec.array[(n++)*ind];
                    }
                    else if(static_cast<std::pair<const int, const int> >(*it).second == gate+startOfMemGateOut && static_cast<std::pair<const int, const int> >(*it).second >startOfHidden && static_cast<std::pair<const int, const int> >(*it).second <startOfMem){//for hidden neurons
                        Vec.array[startOfMemGateOut+gate*ind] += Vec.array[startOfHidden+(static_cast<std::pair<const int, const int> >(*it).first-startOfHidden)*ind]*Vec.array[(n++)*ind];
                    }
                }
                for(connectPairMatrix it = connections.array; it!= connections.array + connections.size; ++it){//for  memGateForget
                    if(static_cast<std::pair<const int, const int> >(*it).second == gate+startOfMemGateForget && static_cast<std::pair<const int, const int> >(*it).second < startOfHidden){//for inputs
                        Vec.array[startOfMemGateForget+gate*ind] += Vec.array[startOfInput+static_cast<std::pair<const int, const int> >(*it).first - startOfInput]*Vec.array[(n++)*ind];
                    }
                    else if(static_cast<std::pair<const int, const int> >(*it).second == gate+startOfMemGateForget && static_cast<std::pair<const int, const int> >(*it).second >startOfHidden && static_cast<std::pair<const int, const int> >(*it).second <startOfMem){//for hidden neurons
                        Vec.array[startOfMemGateForget+gate*ind] += Vec.array[startOfHidden+(static_cast<std::pair<const int, const int> >(*it).first-startOfHidden)*ind]*Vec.array[(n++)*ind];
                    }
                }
                Vec.array[startOfMemGateIn+gate*ind] = ActFunc(Vec.array[startOfMemGateIn+gate*ind]);
                Vec.array[startOfMemGateOut+gate*ind] = ActFunc(Vec.array[startOfMemGateOut+gate*ind]);
                Vec.array[startOfMemGateForget+gate*ind] = ActFunc(Vec.array[startOfMemGateForget+gate*ind]);
            }
            //since we calculated the values for memGateIn and memGateOut, and MemGateForget..
            for (int gate = 0; gate<params.array[5]; gate++){ // if memGateIn is greater than 0.3, then let mem = the sum inputs attached to memGateIn
                if(Vec.array[startOfMemGateIn+gate*ind] > 0.5){ //gate -startOfMemGateIn = [0, num of mem neurons]
                    for(connectPairMatrix it = connections.array; it!= connections.array + connections.size; ++it){
                        if(static_cast<std::pair<const int, const int> >(*it).second == gate+startOfMemGateIn && static_cast<std::pair<const int, const int> >(*it).first < gate+startOfHidden){//only pass inputs
                            Vec.array[startOfMem+gate*ind] += Vec.array[startOfInput+(static_cast<std::pair<const int, const int> >(*it).first-startOfInput)*ind]; // no Vec attached, but the old value stored here is not removed.
                        }
                    }
                }
                if(Vec.array[startOfMemGateForget+gate*ind] > 0.5){// if memGateForget is greater than 0.5, then tell mem to forget
                    Vec.array[startOfMem+gate*ind] = 0;
                }
                //if memGateForget fires, then memGateOut will output nothing.
                if(Vec.array[startOfMemGateOut+gate*ind] > 0.5){//if memGateOut is greater than 0.3, let the nodes mem is connected to recieve mem
                    for(connectPairMatrix it = connections.array; it!= connections.array + connections.size; ++it){
                        if(static_cast<std::pair<const int, const int> >(*it).first == gate+startOfMem){// since mem node: memIn node : memOut node = 1:1:1, we can do this.
                            Vec.array[startOfHidden+static_cast<std::pair<const int, const int> >(*it).second] += Vec.array[startOfMem+gate*ind];
                        }
                    }
                }
            }

            // hidden neuron nodes --
            for(int hid=0; hid<params.array[4]; hid++){ // for all hidden neurons at layer 1, lets sum the inputs, the memory values were already added.
                for(connectPairMatrix it = connections.array; it!= connections.array + connections.size; ++it){ // Add the inputs to the hidden neurons
                    if(static_cast<std::pair<const int, const int> >(*it).second == hid+startOfHidden && static_cast<std::pair<const int, const int> >(*it).first < startOfHidden && static_cast<std::pair<const int, const int> >(*it).first >= startOfInput){ // if an input connects with this hidden neuron
                        Vec.array[startOfHidden+hid*ind] += Vec.array[startOfInput+static_cast<std::pair<const int, const int> >(*it).first*ind]*Vec.array[(n++)*ind];
                    }
                }
                for(connectPairMatrix it = connections.array; it!= connections.array + connections.size; ++it){//add other hidden neuron inputs to each hidden neuron (if applicable)
                    if(static_cast<std::pair<const int, const int> >(*it).second == hid+startOfHidden && static_cast<std::pair<const int, const int> >(*it).first < startOfMem && static_cast<std::pair<const int, const int> >(*it).first >= startOfHidden){
                        Vec.array[startOfHidden+hid*ind] += Vec.array[startOfHidden+(static_cast<std::pair<const int, const int> >(*it).first-startOfHidden)*ind]*Vec.array[(n++)*ind];
                    }
                }
                Vec.array[startOfHidden+hid*ind] += 1*Vec.array[(n++)*ind]; // add bias
                Vec.array[startOfHidden+hid*ind] = ActFunc(Vec.array[startOfHidden+hid*ind]); // then squash static_cast<std::pair<const int, const int> >(*it).
            }
            //output nodes --

            for(int out =0; out<params.array[9]; out++){// add hidden neurons to the output nodes
                for(connectPairMatrix it = connections.array; it!= connections.array + connections.size; ++it){                    if(static_cast<std::pair<const int, const int> >(*it).second == out+startOfOutput){
                        Vec.array[startOfOutput+out*ind] += Vec.array[startOfHidden+(static_cast<std::pair<const int, const int> >(*it).first-startOfHidden)*ind]*Vec.array[(n++)*ind];
                    }
                }
                Vec.array[startOfOutput+out*ind] += 1*Vec.array[(n++)*ind]; // add bias
                Vec.array[startOfOutput+out*ind] = ActFunc(Vec.array[startOfOutput+out*ind]);// then squash static_cast<std::pair<const int, const int> >(*it).
            }

            Vec.array[startOfWhen+j*ind] += Vec.array[startOfOutput+0*ind]*((2160-hour)-hour)+2160-hour; // nv = ((ov - omin)*(nmax-nmin) / (omax - omin))+nmin
            Vec.array[startOfHowCertain+j*ind] += Vec.array[startOfOutput+1*ind];
            Vec.array[startOfCommunityMag+j*ind] =  Vec.array[startOfOutput+2*ind]; // set the next sets communityMag = output #3.
        }
    }
    for(int j=0; j<numOfSites; j++){ // now lets get the average when and howcertain values.
        Vec.array[startOfWhen+j*ind] = Vec.array[startOfWhen+j*ind]/40;
        Vec.array[startOfHowCertain+j*ind] = Vec.array[startOfHowCertain+j*ind]/40;
    }
    /*calculate performance for this individual - score = 1/(abs(whenGuess-whenReal)*distToQuake), for whenGuess = Vec.array[startOfWhen+j] where HowCertain is max for set.
    distToQuake is from the current sites parameters, it emphasizes higher scores for the closest site, a smaller distance is a higher score. */
    int maxCertainty=0;
    float whenGuess=0;
    float latSite=0;
    float lonSite=0;
    for(int j=0; j<numOfSites; j++){
        if(Vec.array[startOfHowCertain+j*ind] > maxCertainty){
            whenGuess = Vec.array[startOfWhen+j*ind];
            latSite = siteData.array[j*2];
            lonSite = siteData.array[j*2+1];
        }
    }
    float SiteToQuakeDist = distCalc(latSite, lonSite, answers.array[1], answers.array[2]); // [2] is latitude, [3] is longitude.
    Vec.array[startOfFitness] = 1/(fabs(whenGuess - answers.array[0]-hour)*SiteToQuakeDist);//larger is better, negative numbers are impossible.
}
