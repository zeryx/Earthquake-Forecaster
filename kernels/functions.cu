#include "hip/hip_runtime.h"
#include <kernelDefs.h>

__host__ __device__ float bearingCalc(float lat1, float lon1, float lat2, float lon2){

    float y = sin(lon2-lon1) * cos(lat2);
    float x = cos(lat1) * sin(lat2) - sin(lat1) * cos(lat2) * cos(lon2-lon1);

    float brng = atan2(y, x);

    brng = brng*180/M_PI;
    brng += 180;
    while(brng>=360)
        brng -= 360;
    return brng;
}

__host__ __device__ float distCalc(float lat1, float lon1, float lat2, float lon2){
    const float earthRad = 6371.01;
    float dLon = (lon1 - lon2);
    float dlat = (lat1 - lat2);
    lat1 = lat1;
    lat2 = lat2;
    float x = sin(dlat/2) * sin(dlat/2) + cos(lat1) * cos(lat2) * sin(dLon/2) * sin(dLon/2);
    float c = 2*atan2(sqrt(x), sqrt(1-x));

    return earthRad*c;
}

__host__ __device__ float normalize(float x, float mean, float stdev){
    return (fabs(x-mean))/(stdev*2);
}

__host__ __device__ float shift(float x, float max, float min){
    return (x-min)/(max-min);
}

__host__ __device__ float ActFunc(float x){
    return tanh(x);
}

__host__ __device__ double scoreFunc(double whenGuess, double whenAns, int hour, float latGuess, float lonGuess, float latAns, float lonAns){
    return 1/(fabs(whenGuess-whenAns-hour)*distCalc(latGuess, lonGuess, latAns, lonAns));
}

