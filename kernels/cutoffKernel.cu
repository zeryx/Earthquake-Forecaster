#include "hip/hip_runtime.h"
#include <kernelDefs.h>

__global__ void cutoffKern(kernelArray<double>vect, kernelArray<int> params, int *childOffset, int *evoGridSize, double *avgFitness, size_t device_offset){
    const int idx = blockIdx.x * blockDim.x +threadIdx.x;
    const int fitnessval = params.array[19] + idx + device_offset;
    if(vect.array[fitnessval] > 0 && vect.array[fitnessval+1] == 0){
        *childOffset = idx;
        *evoGridSize = (params.array[10]-*childOffset)/512;
    }
}
