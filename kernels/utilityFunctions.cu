#include "hip/hip_runtime.h"
#include <utilFunc.h>
#include <float.h>
#define __AVG 2000
__host__ __device__ float bearingCalc(float lat1, float lon1, float lat2, float lon2){

    float y = sin(lon2-lon1) * cos(lat2);
    float x = cos(lat1) * sin(lat2) - sin(lat1) * cos(lat2) * cos(lon2-lon1);
    float brng = atan2(y, x);

    brng = brng*180/M_PI;
    brng += 180;
    while(brng>=360)
        brng -= 360;
    return brng;
}

__host__ __device__ float distCalc(float lat1, float lon1, float lat2, float lon2){

    const float earthRad = 6371.01;
    float dLon = (lon1 - lon2);
    float dlat = (lat1 - lat2);
    lat1 = lat1;
    lat2 = lat2;
    float x = sin(dlat/2) * sin(dlat/2) + cos(lat1) * cos(lat2) * sin(dLon/2) * sin(dLon/2);
    float c = 2*atan2(sqrt(x), sqrt(1-x));

    return earthRad*c;
}

__host__ __device__ float normalize(float x, float mean, float stdev){
    return (fabs(x-mean))/(stdev*2);
}

__host__ __device__ double shift(const double x, float oldMax, float oldMin, float newMax, float newMin){

    /* shift the value X from one range to a new range */
    return newMin + ((newMax-newMin)/(oldMax-oldMin))*(x-oldMin);
}

__host__ __device__ double ActFunc(double &x){
    return tanh(x);
}

__host__ __device__ double scoreFunc(double guess, float whenAns, double latGuess, double lonGuess,
                                     double latAns, double lonAns, double avgFit, int hour, int daysInScope){

    const double shiftedWhere = shift(distCalc(latGuess, lonGuess, latAns, lonAns), 80150.2, 0, 100, 0);

    double correctedGuess;
    if(hour <= whenAns && hour + daysInScope*24 > whenAns) // if the hour of the quake is within scope, then guess should be big, otherwise small
        correctedGuess = exp(guess); //max value is guess = 1 or correctedGuess = e^1
    else
        correctedGuess = exp(-guess);

    const double newFit = correctedGuess + exp(-(shiftedWhere));

    return  (newFit+avgFit*(__AVG-1))/__AVG; //massively increased the weight towards the average, penalizing being wrong much more severely.
}
