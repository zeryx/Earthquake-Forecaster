#include "hip/hip_runtime.h"
#include <kernelDefs.h>

__global__ void normalizeKern(kernelArray<double> vect, kernelArray<int> params, float avgFitness,  int device_offset){
    const int idx = blockIdx.x * blockDim.x + threadIdx.x; // for each thread is one individual
    const int fitnessval = params.array[19] + idx + device_offset;
    vect.array[fitnessval] = vect.array[fitnessval]/ (avgFitness);
//    if(vect.array[fitnessval] < 1){
//        vect.array[fitnessval] = 0;
//    }
}
