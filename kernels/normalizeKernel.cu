#include "hip/hip_runtime.h"
#include <kernelDefs.h>

__global__ void normalizeKern(kernelArray<double> vect, kernelArray<int> params, double *avgFitness,  size_t device_offset){
    const int idx = blockIdx.x * blockDim.x + threadIdx.x; // for each thread is one individual
    const int fitnessval = params.array[19] + idx + device_offset;
    vect.array[fitnessval] = vect.array[fitnessval]/(*avgFitness);
    if(vect.array[fitnessval] < 1.07){//the value set here dictates "how good" an individual has to be to be eligible to reproduce, 1 better than average.
        vect.array[fitnessval] = 0;
    }
}
