#include <iostream>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>
#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/random.h>

struct GenRand
{
    __device__ float operator () (int idx)
    {
        thrust::default_random_engine randEng;
        thrust::uniform_real_distribution<float> uniDist;
        randEng.discard(idx);
        return uniDist(randEng);
    }
};

int main()
{
//    int *d_a, *d_b; //Device variable Declaration

//        //Allocation of Device Variables
//    hipMalloc((void **)&d_a, sizeof(int));
//    hipMalloc((void **)&d_b, sizeof(int));

//        //Copy Host Memory to Device Memory
//    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
//    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);
thrust::device_vector<float> rvect(1000);
thrust::transform(thrust::make_counting_iterator(0),
                  thrust::make_counting_iterator(1000),
                  rvect.begin(),
                  GenRand());

for(int i=0; i<rvect.size(); i++){
    std::cout<<rvect[i]<<std::endl;
}

        //Launch Kernel
//    AddIntsCUDA << <1, 1 >> >(d_a, d_b);

//        //Copy Device Memory to Host Memory
//    hipMemcpy(&a, d_a, sizeof(int), hipMemcpyDeviceToHost);

//    printf("The answer is %d",a);


//        //Free Device Memory
//        hipFree(d_a);
//    hipFree(d_b);

    return 0;
}
