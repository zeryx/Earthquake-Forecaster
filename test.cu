#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>

__global__ void AddIntsCUDA(int *a, int *b) //Kernel Definition
{
    *a = *a + *b;
}

int main()
{
    int a = 5, b = 9;
    int *d_a, *d_b; //Device variable Declaration

        //Allocation of Device Variables
    hipMalloc((void **)&d_a, sizeof(int));
    hipMalloc((void **)&d_b, sizeof(int));

        //Copy Host Memory to Device Memory
    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);


        //Launch Kernel
    AddIntsCUDA << <1, 1 >> >(d_a, d_b);

        //Copy Device Memory to Host Memory
    hipMemcpy(&a, d_a, sizeof(int), hipMemcpyDeviceToHost);

    printf("The answer is ",a);


        //Free Device Memory
        hipFree(d_a);
    hipFree(d_b);

    return 0;
}
